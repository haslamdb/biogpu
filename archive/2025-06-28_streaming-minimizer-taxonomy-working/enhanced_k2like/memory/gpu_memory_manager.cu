#include "hip/hip_runtime.h"
// gpu_memory_manager.cu
// Complete GPU memory management implementation for microbial profiling
// Handles allocation, auto-scaling, and memory optimization

#include "gpu_memory_manager.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <iomanip>
#include <algorithm>
#include <cstring>

// ===========================
// CUDA Kernels
// ===========================

__global__ void memory_initialization_kernel(
    char* sequence_data, size_t sequence_size,
    GPUGenomeInfo* genome_info, size_t genome_count,
    GPUMinimizerHit* minimizer_hits, size_t minimizer_count
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize sequence data
    if (sequence_data && tid < sequence_size) {
        sequence_data[tid] = 0;
    }
    
    // Initialize genome info
    if (genome_info && tid < genome_count) {
        genome_info[tid].genome_id = 0;
        genome_info[tid].sequence_offset = 0;
        genome_info[tid].sequence_length = 0;
        genome_info[tid].minimizer_count = 0;
        genome_info[tid].taxon_id = 0;
    }
    
    // Initialize minimizer hits
    if (minimizer_hits && tid < minimizer_count) {
        minimizer_hits[tid].minimizer_hash = 0;
        minimizer_hits[tid].genome_id = 0;
        minimizer_hits[tid].position = 0;
        minimizer_hits[tid].strand = 0;
        minimizer_hits[tid].taxon_id = 0;
    }
}

__global__ void memory_validation_kernel(
    char* sequence_data,
    GPUGenomeInfo* genome_info,
    int* validation_result,
    size_t num_genomes
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid >= num_genomes) return;
    
    // Validate genome info structure
    if (genome_info[tid].sequence_offset != UINT32_MAX && 
        genome_info[tid].sequence_length > 0) {
        
        // Check if sequence data pointer is valid
        uint32_t start = genome_info[tid].sequence_offset;
        uint32_t length = genome_info[tid].sequence_length;
        
        // Basic validation - check for null termination or valid nucleotides
        bool valid = true;
        for (uint32_t i = 0; i < min(length, 100u); i++) {
            char c = sequence_data[start + i];
            if (c != 'A' && c != 'C' && c != 'G' && c != 'T' && 
                c != 'N' && c != 0) {
                valid = false;
                break;
            }
        }
        
        if (!valid) {
            atomicAdd(validation_result, 1);  // Increment error count
        }
    }
}

// ===========================
// Kernel Launch Functions
// ===========================

bool launch_memory_initialization_kernel(
    char* sequence_data, size_t sequence_size,
    GPUGenomeInfo* genome_info, size_t genome_count,
    GPUMinimizerHit* minimizer_hits, size_t minimizer_count
) {
    // Calculate grid dimensions
    size_t max_elements = std::max({sequence_size, genome_count, minimizer_count});
    
    int block_size = 256;
    int grid_size = (max_elements + block_size - 1) / block_size;
    
    // Launch kernel
    memory_initialization_kernel<<<grid_size, block_size>>>(
        sequence_data, sequence_size,
        genome_info, genome_count,
        minimizer_hits, minimizer_count
    );
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(error) << std::endl;
        return false;
    }
    
    // Wait for completion
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(error) << std::endl;
        return false;
    }
    
    return true;
}

bool launch_memory_validation_kernel(
    char* sequence_data,
    GPUGenomeInfo* genome_info,
    size_t num_genomes
) {
    // Allocate result buffer on GPU
    int* d_validation_result;
    hipError_t error = hipMalloc(&d_validation_result, sizeof(int));
    if (error != hipSuccess) {
        std::cerr << "Failed to allocate validation result buffer" << std::endl;
        return false;
    }
    
    hipMemset(d_validation_result, 0, sizeof(int));
    
    int block_size = 256;
    int grid_size = (num_genomes + block_size - 1) / block_size;
    
    // Launch validation kernel
    memory_validation_kernel<<<grid_size, block_size>>>(
        sequence_data, genome_info, d_validation_result, num_genomes
    );
    
    error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "Validation kernel launch failed: " << hipGetErrorString(error) << std::endl;
        hipFree(d_validation_result);
        return false;
    }
    
    // Get result
    int error_count;
    error = hipMemcpy(&error_count, d_validation_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_validation_result);
    
    if (error != hipSuccess) {
        std::cerr << "Failed to copy validation result" << std::endl;
        return false;
    }
    
    return error_count == 0;
}

// ===========================
// GPU Memory Pool Implementation
// ===========================

GPUMemoryPool::GPUMemoryPool(size_t pool_size_mb) 
    : pool_base_(nullptr), pool_size_(pool_size_mb * 1024 * 1024), 
      current_offset_(0), initialized_(false) {
    
    hipError_t error = hipMalloc(&pool_base_, pool_size_);
    if (error == hipSuccess) {
        initialized_ = true;
        std::cout << "GPU memory pool created: " << pool_size_mb << " MB" << std::endl;
    } else {
        std::cerr << "Failed to create GPU memory pool: " << hipGetErrorString(error) << std::endl;
    }
}

GPUMemoryPool::~GPUMemoryPool() {
    if (pool_base_) {
        hipFree(pool_base_);
        pool_base_ = nullptr;
    }
}

void* GPUMemoryPool::allocate(size_t size, size_t alignment) {
    if (!initialized_ || size == 0) return nullptr;
    
    // Align the current offset
    size_t aligned_offset = (current_offset_ + alignment - 1) & ~(alignment - 1);
    
    // Check if we have enough space
    if (aligned_offset + size > pool_size_) {
        return nullptr;  // Out of memory
    }
    
    void* ptr = static_cast<char*>(pool_base_) + aligned_offset;
    current_offset_ = aligned_offset + size;
    
    return ptr;
}

void GPUMemoryPool::deallocate(void* ptr) {
    // Simple pool doesn't support individual deallocation
    // Could be extended with a free list
}

void GPUMemoryPool::reset() {
    current_offset_ = 0;
}

size_t GPUMemoryPool::get_available_space() const {
    return pool_size_ - current_offset_;
}

bool GPUMemoryPool::is_initialized() const { 
    return initialized_; 
}

// ===========================
// GPU Memory Manager Implementation
// ===========================

GPUMemoryManager::GPUMemoryManager(const MemoryConfig& config)
    : config_(config), initialized_(false), allocations_active_(false), total_allocated_(0),
      d_sequence_data_(nullptr), d_genome_info_(nullptr), d_minimizer_hits_(nullptr),
      d_lca_candidates_(nullptr), d_minimizer_counts_(nullptr), sequence_buffer_size_(0) {
    
    // Initialize statistics
    memset(&stats_, 0, sizeof(stats_));
}

GPUMemoryManager::~GPUMemoryManager() {
    free_all_allocations();
}

bool GPUMemoryManager::initialize() {
        if (initialized_) return true;
        
        std::cout << "Initializing GPU Memory Manager..." << std::endl;
        
        // Query GPU memory information
        if (!query_gpu_memory_info()) {
            std::cerr << "Failed to query GPU memory information" << std::endl;
            return false;
        }
        
        // Validate GPU context
        if (!validate_gpu_context()) {
            std::cerr << "Invalid GPU context" << std::endl;
            return false;
        }
        
        // Create memory pool if enabled
        if (config_.enable_memory_pooling) {
            size_t pool_size_mb = (stats_.available_memory * config_.max_memory_fraction / 100) / (1024 * 1024);
            pool_size_mb = std::min(pool_size_mb, size_t(8192));  // Cap at 8GB
            
            memory_pool_ = std::make_unique<GPUMemoryPool>(pool_size_mb);
            if (!memory_pool_ || !memory_pool_->is_initialized()) {
                std::cerr << "Failed to create GPU memory pool" << std::endl;
                return false;
            }
        }
        
        // Calculate optimal configuration if auto-scaling is enabled
        if (config_.auto_scale_enabled) {
            if (!calculate_optimal_batch_sizes()) {
                std::cerr << "Failed to calculate optimal batch sizes" << std::endl;
                return false;
            }
        }
        
        initialized_ = true;
        std::cout << "✓ GPU Memory Manager initialized successfully" << std::endl;
        print_memory_usage();
        
        return true;
    }

bool GPUMemoryManager::configure_auto_scaling(bool enable, size_t memory_fraction) {
        config_.auto_scale_enabled = enable;
        config_.max_memory_fraction = std::min(memory_fraction, size_t(95));  // Cap at 95%
        
        if (enable && initialized_) {
            return calculate_optimal_batch_sizes();
        }
        
        return true;
    }

bool GPUMemoryManager::set_minimizer_capacity(int capacity) {
        if (capacity <= 0 || capacity > 50000000) {  // Cap at 50M
            std::cerr << "Invalid minimizer capacity: " << capacity << std::endl;
            return false;
        }
        
        config_.minimizer_capacity = capacity;
        std::cout << "Set minimizer capacity to " << capacity << std::endl;
        
        return true;
    }

bool GPUMemoryManager::set_batch_size(int batch_size) {
        if (batch_size <= 0 || batch_size > 100) {
            std::cerr << "Invalid batch size: " << batch_size << std::endl;
            return false;
        }
        
        config_.sequence_batch_size = batch_size;
        std::cout << "Set sequence batch size to " << batch_size << std::endl;
        
        return true;
    }

bool GPUMemoryManager::allocate_sequence_memory(size_t max_sequences, size_t max_total_length) {
        if (allocations_active_) {
            std::cerr << "Memory already allocated. Free existing allocations first." << std::endl;
            return false;
        }
        
        // Validate request
        size_t required_memory = max_total_length + max_sequences * sizeof(GPUGenomeInfo);
        if (!validate_allocation_request(required_memory)) {
            return false;
        }
        
        std::cout << "Allocating sequence memory:" << std::endl;
        std::cout << "  Max sequences: " << max_sequences << std::endl;
        std::cout << "  Total length: " << (max_total_length / 1024 / 1024) << " MB" << std::endl;
        
        // Allocate sequence data buffer
        sequence_buffer_size_ = max_total_length;
        hipError_t error = hipMalloc(&d_sequence_data_, max_total_length);
        if (error != hipSuccess) {
            std::cerr << "Failed to allocate sequence data: " << hipGetErrorString(error) << std::endl;
            sequence_buffer_size_ = 0;
            return false;
        }
        
        // Allocate genome info buffer
        error = hipMalloc(&d_genome_info_, max_sequences * sizeof(GPUGenomeInfo));
        if (error != hipSuccess) {
            std::cerr << "Failed to allocate genome info: " << hipGetErrorString(error) << std::endl;
            hipFree(d_sequence_data_);
            d_sequence_data_ = nullptr;
            return false;
        }
        
        // Initialize memory
        if (!launch_memory_initialization_kernel(
            d_sequence_data_, max_total_length,
            d_genome_info_, max_sequences,
            nullptr, 0)) {
            std::cerr << "Failed to initialize sequence memory" << std::endl;
            hipFree(d_sequence_data_);
            hipFree(d_genome_info_);
            d_sequence_data_ = nullptr;
            d_genome_info_ = nullptr;
            return false;
        }
        
        stats_.current_sequence_memory = max_total_length + max_sequences * sizeof(GPUGenomeInfo);
        total_allocated_ += stats_.current_sequence_memory;
        allocations_active_ = true;
        
        std::cout << "✓ Sequence memory allocated successfully" << std::endl;
        return true;
    }

bool GPUMemoryManager::allocate_minimizer_memory(size_t max_minimizers) {
        size_t minimizer_memory = max_minimizers * sizeof(GPUMinimizerHit);
        size_t count_memory = config_.sequence_batch_size * sizeof(uint32_t);
        size_t total_memory = minimizer_memory + count_memory;
        
        if (!validate_allocation_request(total_memory)) {
            return false;
        }
        
        std::cout << "Allocating minimizer memory for " << max_minimizers << " minimizers" << std::endl;
        
        // Allocate minimizer hits buffer
        hipError_t error = hipMalloc(&d_minimizer_hits_, minimizer_memory);
        if (error != hipSuccess) {
            std::cerr << "Failed to allocate minimizer hits: " << hipGetErrorString(error) << std::endl;
            return false;
        }
        
        // Allocate count buffer
        error = hipMalloc(&d_minimizer_counts_, count_memory);
        if (error != hipSuccess) {
            std::cerr << "Failed to allocate minimizer counts: " << hipGetErrorString(error) << std::endl;
            hipFree(d_minimizer_hits_);
            d_minimizer_hits_ = nullptr;
            return false;
        }
        
        // Initialize memory
        if (!launch_memory_initialization_kernel(
            nullptr, 0,
            nullptr, 0,
            d_minimizer_hits_, max_minimizers)) {
            std::cerr << "Failed to initialize minimizer memory" << std::endl;
            hipFree(d_minimizer_hits_);
            hipFree(d_minimizer_counts_);
            d_minimizer_hits_ = nullptr;
            d_minimizer_counts_ = nullptr;
            return false;
        }
        
        stats_.current_minimizer_memory = total_memory;
        total_allocated_ += stats_.current_minimizer_memory;
        
        std::cout << "✓ Minimizer memory allocated successfully" << std::endl;
        return true;
    }

bool GPUMemoryManager::allocate_metadata_memory(size_t max_genomes) {
        // This would allocate additional metadata buffers
        stats_.current_metadata_memory = max_genomes * sizeof(uint32_t) * 4;  // Example
        return true;
    }

bool GPUMemoryManager::allocate_results_memory(size_t max_candidates) {
        size_t results_memory = max_candidates * sizeof(LCACandidate);
        
        if (!validate_allocation_request(results_memory)) {
            return false;
        }
        
        std::cout << "Allocating results memory for " << max_candidates << " candidates" << std::endl;
        
        hipError_t error = hipMalloc(&d_lca_candidates_, results_memory);
        if (error != hipSuccess) {
            std::cerr << "Failed to allocate LCA candidates: " << hipGetErrorString(error) << std::endl;
            return false;
        }
        
        // Initialize results memory
        hipMemset(d_lca_candidates_, 0, results_memory);
        
        total_allocated_ += results_memory;
        
        std::cout << "✓ Results memory allocated successfully" << std::endl;
        return true;
    }

void GPUMemoryManager::free_all_allocations() {
        std::cout << "Freeing all GPU memory allocations..." << std::endl;
        
        if (d_sequence_data_) {
            hipFree(d_sequence_data_);
            d_sequence_data_ = nullptr;
            sequence_buffer_size_ = 0;
        }
        
        if (d_genome_info_) {
            hipFree(d_genome_info_);
            d_genome_info_ = nullptr;
        }
        
        if (d_minimizer_hits_) {
            hipFree(d_minimizer_hits_);
            d_minimizer_hits_ = nullptr;
        }
        
        if (d_lca_candidates_) {
            hipFree(d_lca_candidates_);
            d_lca_candidates_ = nullptr;
        }
        
        if (d_minimizer_counts_) {
            hipFree(d_minimizer_counts_);
            d_minimizer_counts_ = nullptr;
        }
        
        // Reset memory pool if it exists
        if (memory_pool_) {
            memory_pool_->reset();
        }
        
        // Reset statistics
        stats_.current_sequence_memory = 0;
        stats_.current_minimizer_memory = 0;
        stats_.current_metadata_memory = 0;
        total_allocated_ = 0;
        allocations_active_ = false;
        
        std::cout << "✓ All GPU memory freed" << std::endl;
    }

bool GPUMemoryManager::validate_memory_integrity() {
        if (!allocations_active_ || !d_sequence_data_ || !d_genome_info_) {
            return true;  // No memory to validate
        }
        
        std::cout << "Validating GPU memory integrity..." << std::endl;
        
        bool result = launch_memory_validation_kernel(
            d_sequence_data_, d_genome_info_, config_.sequence_batch_size
        );
        
        if (result) {
            std::cout << "✓ Memory integrity validation passed" << std::endl;
        } else {
            std::cerr << "⚠ Memory integrity validation failed" << std::endl;
        }
        
        return result;
    }

    // Accessor methods
char* GPUMemoryManager::get_sequence_buffer() const {
        return d_sequence_data_;
    }

GPUGenomeInfo* GPUMemoryManager::get_genome_info_buffer() const {
        return d_genome_info_;
    }

GPUMinimizerHit* GPUMemoryManager::get_minimizer_buffer() const {
        return d_minimizer_hits_;
    }

LCACandidate* GPUMemoryManager::get_candidate_buffer() const {
        return d_lca_candidates_;
    }

uint32_t* GPUMemoryManager::get_count_buffer() const {
        return d_minimizer_counts_;
    }

MemoryStats GPUMemoryManager::get_memory_statistics() const {
        // Update current statistics
        const_cast<GPUMemoryManager*>(this)->update_memory_statistics();
        return stats_;
    }

void GPUMemoryManager::print_memory_usage() const {
        std::cout << "\n=== GPU MEMORY USAGE ===" << std::endl;
        std::cout << "Total GPU memory: " << (stats_.total_gpu_memory / 1024 / 1024) << " MB" << std::endl;
        std::cout << "Available memory: " << (stats_.available_memory / 1024 / 1024) << " MB" << std::endl;
        std::cout << "Currently allocated: " << (total_allocated_ / 1024 / 1024) << " MB" << std::endl;
        
        if (stats_.current_sequence_memory > 0) {
            std::cout << "  Sequence data: " << (stats_.current_sequence_memory / 1024 / 1024) << " MB" << std::endl;
        }
        if (stats_.current_minimizer_memory > 0) {
            std::cout << "  Minimizer data: " << (stats_.current_minimizer_memory / 1024 / 1024) << " MB" << std::endl;
        }
        if (stats_.current_metadata_memory > 0) {
            std::cout << "  Metadata: " << (stats_.current_metadata_memory / 1024 / 1024) << " MB" << std::endl;
        }
        
        double usage_percent = total_allocated_ > 0 ? 
            (double)total_allocated_ / stats_.total_gpu_memory * 100.0 : 0.0;
        std::cout << "Memory usage: " << std::fixed << std::setprecision(1) 
                  << usage_percent << "%" << std::endl;
        
        if (memory_pool_) {
            std::cout << "Memory pool available: " 
                      << (memory_pool_->get_available_space() / 1024 / 1024) << " MB" << std::endl;
        }
    }

bool GPUMemoryManager::check_memory_pressure() const {
        double usage_ratio = (double)total_allocated_ / stats_.available_memory;
        return usage_ratio > 0.85;  // Consider high pressure at 85%
    }

size_t GPUMemoryManager::estimate_memory_requirements(int num_sequences, int avg_sequence_length) const {
        size_t sequence_memory = num_sequences * avg_sequence_length;
        size_t genome_info_memory = num_sequences * sizeof(GPUGenomeInfo);
        size_t minimizer_memory = config_.minimizer_capacity * sizeof(GPUMinimizerHit);
        size_t overhead = (sequence_memory + genome_info_memory + minimizer_memory) * 0.1;  // 10% overhead
        
        return sequence_memory + genome_info_memory + minimizer_memory + overhead;
    }

bool GPUMemoryManager::scale_for_workload(size_t estimated_minimizers, size_t estimated_sequences) {
        if (!config_.auto_scale_enabled) return true;
        
        std::cout << "Auto-scaling for workload:" << std::endl;
        std::cout << "  Estimated minimizers: " << estimated_minimizers << std::endl;
        std::cout << "  Estimated sequences: " << estimated_sequences << std::endl;
        
        // Calculate required capacity
        size_t safe_minimizer_capacity = calculate_safe_minimizer_capacity();
        size_t required_capacity = std::max(estimated_minimizers, safe_minimizer_capacity);
        
        if (required_capacity != config_.minimizer_capacity) {
            std::cout << "Adjusting minimizer capacity: " << config_.minimizer_capacity 
                      << " → " << required_capacity << std::endl;
            config_.minimizer_capacity = required_capacity;
        }
        
        // Adjust batch size if needed
        int optimal_batch_size = calculate_optimal_batch_size();
        if (optimal_batch_size != config_.sequence_batch_size) {
            std::cout << "Adjusting batch size: " << config_.sequence_batch_size 
                      << " → " << optimal_batch_size << std::endl;
            config_.sequence_batch_size = optimal_batch_size;
        }
        
        return true;
    }

void GPUMemoryManager::suggest_optimal_configuration() const {
        std::cout << "\n=== MEMORY OPTIMIZATION SUGGESTIONS ===" << std::endl;
        
        size_t total_gb = stats_.total_gpu_memory / (1024 * 1024 * 1024);
        
        if (total_gb >= 24) {
            std::cout << "High-memory GPU detected (" << total_gb << " GB)" << std::endl;
            std::cout << "Suggested configuration:" << std::endl;
            std::cout << "  Minimizer capacity: 15,000,000 - 25,000,000" << std::endl;
            std::cout << "  Sequence batch size: 40-50" << std::endl;
        } else if (total_gb >= 16) {
            std::cout << "Medium-memory GPU detected (" << total_gb << " GB)" << std::endl;
            std::cout << "Suggested configuration:" << std::endl;
            std::cout << "  Minimizer capacity: 8,000,000 - 15,000,000" << std::endl;
            std::cout << "  Sequence batch size: 25-35" << std::endl;
        } else if (total_gb >= 8) {
            std::cout << "Standard-memory GPU detected (" << total_gb << " GB)" << std::endl;
            std::cout << "Suggested configuration:" << std::endl;
            std::cout << "  Minimizer capacity: 3,000,000 - 8,000,000" << std::endl;
            std::cout << "  Sequence batch size: 15-25" << std::endl;
        } else {
            std::cout << "Limited-memory GPU detected (" << total_gb << " GB)" << std::endl;
            std::cout << "Suggested configuration:" << std::endl;
            std::cout << "  Minimizer capacity: 1,000,000 - 3,000,000" << std::endl;
            std::cout << "  Sequence batch size: 5-15" << std::endl;
            std::cout << "  Consider using streaming mode for large datasets" << std::endl;
        }
    }

// Internal methods implementation
bool GPUMemoryManager::query_gpu_memory_info() {
        hipError_t error = hipMemGetInfo(&stats_.available_memory, &stats_.total_gpu_memory);
        if (error != hipSuccess) {
            std::cerr << "Failed to query GPU memory: " << hipGetErrorString(error) << std::endl;
            return false;
        }
        
        stats_.allocated_memory = stats_.total_gpu_memory - stats_.available_memory;
        return true;
    }

bool GPUMemoryManager::calculate_optimal_batch_sizes() {
        // Calculate optimal minimizer capacity based on available memory
        size_t usable_memory = stats_.available_memory * config_.max_memory_fraction / 100;
        usable_memory -= config_.reserved_memory_mb * 1024 * 1024;  // Reserve memory for system
        
        if (usable_memory <= 0) {
            std::cerr << "Insufficient GPU memory available" << std::endl;
            return false;
        }
        
        // Estimate memory distribution
        size_t minimizer_memory = usable_memory * 0.6;  // 60% for minimizers
        size_t sequence_memory = usable_memory * 0.3;   // 30% for sequences
        size_t overhead_memory = usable_memory * 0.1;   // 10% overhead
        
        // Calculate optimal capacities
        config_.minimizer_capacity = minimizer_memory / sizeof(GPUMinimizerHit);
        config_.minimizer_capacity = std::min(config_.minimizer_capacity, size_t(50000000));  // Cap at 50M
        config_.minimizer_capacity = std::max(config_.minimizer_capacity, size_t(1000000));   // Min 1M
        
        // Calculate optimal batch size
        size_t sequence_per_mb = 1024 * 1024 / 5000;  // Assume 5KB per sequence on average
        config_.sequence_batch_size = (sequence_memory / 1024 / 1024) / sequence_per_mb;
        config_.sequence_batch_size = std::min(config_.sequence_batch_size, size_t(100));  // Cap at 100
        config_.sequence_batch_size = std::max(config_.sequence_batch_size, size_t(5));    // Min 5
        
        std::cout << "Calculated optimal configuration:" << std::endl;
        std::cout << "  Minimizer capacity: " << config_.minimizer_capacity << std::endl;
        std::cout << "  Sequence batch size: " << config_.sequence_batch_size << std::endl;
        
        return true;
    }

bool GPUMemoryManager::validate_gpu_context() {
        int device_count;
        hipError_t error = hipGetDeviceCount(&device_count);
        if (error != hipSuccess || device_count == 0) {
            std::cerr << "No CUDA devices found" << std::endl;
            return false;
        }
        
        int current_device;
        error = hipGetDevice(&current_device);
        if (error != hipSuccess) {
            std::cerr << "Failed to get current CUDA device" << std::endl;
            return false;
        }
        
        return true;
    }

void GPUMemoryManager::update_memory_statistics() {
        query_gpu_memory_info();
        stats_.memory_efficiency = calculate_memory_efficiency();
        
        if (total_allocated_ > stats_.peak_usage) {
            stats_.peak_usage = total_allocated_;
        }
    }

bool GPUMemoryManager::validate_allocation_request(size_t size) const {
        if (size == 0) return false;
        
        size_t available_for_allocation = stats_.available_memory * config_.max_memory_fraction / 100;
        available_for_allocation -= config_.reserved_memory_mb * 1024 * 1024;
        
        if (size > available_for_allocation) {
            std::cerr << "Allocation request (" << (size / 1024 / 1024) << " MB) "
                      << "exceeds available memory (" << (available_for_allocation / 1024 / 1024) << " MB)" << std::endl;
            return false;
        }
        
        return true;
    }

size_t GPUMemoryManager::calculate_safe_minimizer_capacity() const {
        size_t memory_gb = stats_.total_gpu_memory / (1024 * 1024 * 1024);
        
        if (memory_gb >= 24) return 15000000;      // 15M for high-end GPUs
        else if (memory_gb >= 16) return 10000000; // 10M for mid-range GPUs
        else if (memory_gb >= 8) return 5000000;   // 5M for standard GPUs
        else return 2000000;                       // 2M for low-end GPUs
    }

int GPUMemoryManager::calculate_optimal_batch_size() const {
        size_t memory_gb = stats_.total_gpu_memory / (1024 * 1024 * 1024);
        
        if (memory_gb >= 24) return 50;
        else if (memory_gb >= 16) return 35;
        else if (memory_gb >= 8) return 25;
        else return 15;
    }

double GPUMemoryManager::calculate_memory_efficiency() const {
    if (stats_.peak_usage == 0) return 1.0;
    return (double)total_allocated_ / stats_.peak_usage;
}

// ===========================
// Memory Utilities Implementation
// ===========================

namespace MemoryUtils {
    
    void check_cuda_memory_leaks() {
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        
        static size_t last_free_mem = free_mem;
        
        if (free_mem < last_free_mem) {
            size_t leaked = last_free_mem - free_mem;
            std::cout << "Potential memory leak detected: " 
                      << (leaked / 1024 / 1024) << " MB" << std::endl;
        }
        
        last_free_mem = free_mem;
    }
    
    bool validate_gpu_pointer(void* ptr, size_t size) {
        if (!ptr || size == 0) return false;
        
        hipPointerAttribute_t attributes;
        hipError_t error = hipPointerGetAttributes(&attributes, ptr);
        
        if (error != hipSuccess) {
            hipGetLastError(); // Clear the error
            return false;
        }
        
        return attributes.type == hipMemoryTypeDevice;
    }
    
    size_t get_gpu_memory_usage() {
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        return total_mem - free_mem;
    }
    
    void print_gpu_memory_info() {
        size_t free_mem, total_mem;
        hipMemGetInfo(&free_mem, &total_mem);
        
        std::cout << "GPU Memory Info:" << std::endl;
        std::cout << "  Total: " << (total_mem / 1024 / 1024) << " MB" << std::endl;
        std::cout << "  Free: " << (free_mem / 1024 / 1024) << " MB" << std::endl;
        std::cout << "  Used: " << ((total_mem - free_mem) / 1024 / 1024) << " MB" << std::endl;
    }
}

// ===========================
// Example Usage for Fluoroquinolone Resistance Detection
// ===========================

/*
// Example usage in your fluoroquinolone resistance pipeline:

int main() {
    // Initialize memory manager with custom configuration
    MemoryConfig config;
    config.max_memory_fraction = 85;  // Use 85% of GPU memory
    config.auto_scale_enabled = true;
    
    GPUMemoryManager memory_manager(config);
    
    if (!memory_manager.initialize()) {
        std::cerr << "Failed to initialize GPU memory manager" << std::endl;
        return -1;
    }
    
    // Allocate memory for microbial genome database
    size_t max_genomes = 50000;  // 50K microbial genomes
    size_t total_genome_length = 10ULL * 1024 * 1024 * 1024;  // 10GB of sequence data
    
    if (!memory_manager.allocate_sequence_memory(max_genomes, total_genome_length)) {
        std::cerr << "Failed to allocate sequence memory" << std::endl;
        return -1;
    }
    
    // Allocate memory for minimizer hits (for rapid read mapping)
    size_t max_minimizers = 20000000;  // 20M minimizers
    if (!memory_manager.allocate_minimizer_memory(max_minimizers)) {
        std::cerr << "Failed to allocate minimizer memory" << std::endl;
        return -1;
    }
    
    // Allocate results memory for resistance detection
    size_t max_candidates = 1000000;  // 1M resistance candidates
    if (!memory_manager.allocate_results_memory(max_candidates)) {
        std::cerr << "Failed to allocate results memory" << std::endl;
        return -1;
    }
    
    // Get pointers for use in your CUDA kernels
    char* d_genomes = memory_manager.get_sequence_buffer();
    GPUGenomeInfo* d_genome_info = memory_manager.get_genome_info_buffer();
    GPUMinimizerHit* d_minimizers = memory_manager.get_minimizer_buffer();
    LCACandidate* d_results = memory_manager.get_candidate_buffer();
    
    // Now you can use these pointers in your resistance detection kernels:
    // - Load microbial genomes with gyrA, gyrB, parC, parE genes
    // - Map sequencing reads to genomes
    // - Detect fluoroquinolone resistance mutations
    // - Generate clinical resistance report
    
    memory_manager.print_memory_usage();
    
    // Validate memory integrity
    if (!memory_manager.validate_memory_integrity()) {
        std::cerr << "Memory integrity validation failed" << std::endl;
    }
    
    // Memory will be automatically freed when memory_manager goes out of scope
    
    return 0;
}
*/
#include "hip/hip_runtime.h"
// runtime/common/gpu/gpu_sequence_buffer.cu
#include "gpu_sequence_buffer.h"
#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << std::endl; \
            return false; \
        } \
    } while(0)

#define CUDA_CHECK_VOID(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                      << " - " << hipGetErrorString(error) << std::endl; \
            return; \
        } \
    } while(0)

namespace BioGPU {

GPUSequenceBuffer::GPUSequenceBuffer(size_t max_seqs, size_t max_length) 
    : max_sequences(max_seqs), max_total_length(max_length),
      current_sequences(0), current_total_length(0) {
}

GPUSequenceBuffer::~GPUSequenceBuffer() {
    free();
}

bool GPUSequenceBuffer::allocate() {
    // Free any existing allocations
    free();
    
    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_sequences, max_total_length * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_offsets, (max_sequences + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_lengths, max_sequences * sizeof(int)));
    
    // Allocate pinned host memory for fast transfers
    CUDA_CHECK(hipHostMalloc(&h_pinned_sequences, max_total_length * sizeof(char)));
    CUDA_CHECK(hipHostMalloc(&h_pinned_offsets, (max_sequences + 1) * sizeof(int)));
    CUDA_CHECK(hipHostMalloc(&h_pinned_lengths, max_sequences * sizeof(int)));
    
    // Initialize offsets
    CUDA_CHECK(hipMemset(d_offsets, 0, sizeof(int)));
    h_pinned_offsets[0] = 0;
    
    return true;
}

void GPUSequenceBuffer::free() {
    // Free device memory
    if (d_sequences) {
        hipFree(d_sequences);
        d_sequences = nullptr;
    }
    if (d_offsets) {
        hipFree(d_offsets);
        d_offsets = nullptr;
    }
    if (d_lengths) {
        hipFree(d_lengths);
        d_lengths = nullptr;
    }
    if (d_headers) {
        hipFree(d_headers);
        d_headers = nullptr;
    }
    
    // Free pinned host memory
    if (h_pinned_sequences) {
        hipHostFree(h_pinned_sequences);
        h_pinned_sequences = nullptr;
    }
    if (h_pinned_offsets) {
        hipHostFree(h_pinned_offsets);
        h_pinned_offsets = nullptr;
    }
    if (h_pinned_lengths) {
        hipHostFree(h_pinned_lengths);
        h_pinned_lengths = nullptr;
    }
    
    current_sequences = 0;
    current_total_length = 0;
}

bool GPUSequenceBuffer::transferBatch(const SequenceBatch& batch) {
    if (batch.empty()) {
        return true;  // Nothing to transfer
    }
    
    // Check if batch fits
    size_t total_length = batch.getTotalBases();
    if (batch.size() > max_sequences || total_length > max_total_length) {
        std::cerr << "Error: Batch too large for buffer. "
                  << "Sequences: " << batch.size() << "/" << max_sequences
                  << ", Bases: " << total_length << "/" << max_total_length << std::endl;
        return false;
    }
    
    // Prepare flat format in pinned memory
    std::vector<char> flat_sequences;
    std::vector<int> offsets;
    const_cast<SequenceBatch&>(batch).prepareFlatFormat(flat_sequences, offsets);
    
    // Copy to pinned memory
    std::memcpy(h_pinned_sequences, flat_sequences.data(), flat_sequences.size());
    std::memcpy(h_pinned_offsets, offsets.data(), offsets.size() * sizeof(int));
    std::memcpy(h_pinned_lengths, batch.lengths.data(), batch.lengths.size() * sizeof(int));
    
    // Transfer to GPU
    CUDA_CHECK(hipMemcpy(d_sequences, h_pinned_sequences, 
                         flat_sequences.size(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_offsets, h_pinned_offsets, 
                         offsets.size() * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_lengths, h_pinned_lengths, 
                         batch.lengths.size() * sizeof(int), hipMemcpyHostToDevice));
    
    current_sequences = batch.size();
    current_total_length = flat_sequences.size();
    
    return true;
}

bool GPUSequenceBuffer::transferBatchAsync(const SequenceBatch& batch, hipStream_t stream) {
    if (batch.empty()) {
        return true;  // Nothing to transfer
    }
    
    // Check if batch fits
    size_t total_length = batch.getTotalBases();
    if (batch.size() > max_sequences || total_length > max_total_length) {
        std::cerr << "Error: Batch too large for buffer. "
                  << "Sequences: " << batch.size() << "/" << max_sequences
                  << ", Bases: " << total_length << "/" << max_total_length << std::endl;
        return false;
    }
    
    // Prepare flat format in pinned memory
    std::vector<char> flat_sequences;
    std::vector<int> offsets;
    const_cast<SequenceBatch&>(batch).prepareFlatFormat(flat_sequences, offsets);
    
    // Copy to pinned memory
    std::memcpy(h_pinned_sequences, flat_sequences.data(), flat_sequences.size());
    std::memcpy(h_pinned_offsets, offsets.data(), offsets.size() * sizeof(int));
    std::memcpy(h_pinned_lengths, batch.lengths.data(), batch.lengths.size() * sizeof(int));
    
    // Async transfer to GPU
    CUDA_CHECK(hipMemcpyAsync(d_sequences, h_pinned_sequences, 
                              flat_sequences.size(), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_offsets, h_pinned_offsets, 
                              offsets.size() * sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CHECK(hipMemcpyAsync(d_lengths, h_pinned_lengths, 
                              batch.lengths.size() * sizeof(int), hipMemcpyHostToDevice, stream));
    
    current_sequences = batch.size();
    current_total_length = flat_sequences.size();
    
    return true;
}

bool GPUSequenceBuffer::resize(size_t new_max_seqs, size_t new_max_length) {
    // If sizes are sufficient, no need to resize
    if (new_max_seqs <= max_sequences && new_max_length <= max_total_length) {
        return true;
    }
    
    // Free existing memory
    free();
    
    // Update sizes
    max_sequences = std::max(new_max_seqs, max_sequences);
    max_total_length = std::max(new_max_length, max_total_length);
    
    // Reallocate with new sizes
    return allocate();
}

void GPUSequenceBuffer::clear() {
    current_sequences = 0;
    current_total_length = 0;
    
    // Reset first offset to 0
    if (d_offsets) {
        hipMemset(d_offsets, 0, sizeof(int));
    }
    if (h_pinned_offsets) {
        h_pinned_offsets[0] = 0;
    }
}

} // namespace BioGPU
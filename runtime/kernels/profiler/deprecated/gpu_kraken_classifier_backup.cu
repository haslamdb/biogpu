#include "hip/hip_runtime.h"
// gpu_kraken_classifier_paired.cuh
// GPU-accelerated Kraken2-style taxonomic classifier with PAIRED-END support
// Implements paired-end concordance, voting boost, and confidence scoring

#pragma once
#ifndef GPU_KRAKEN_CLASSIFIER_PAIRED_CUH
#define GPU_KRAKEN_CLASSIFIER_PAIRED_CUH

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <vector>
#include <string>
#include <cstdint>
#include <unordered_map>

// Paired read structure (similar to your existing profiler)
struct PairedRead {
    std::string read1;
    std::string read2;
    std::string read_id;
    bool is_paired;
    
    PairedRead(const std::string& r1, const std::string& r2 = "", 
               const std::string& id = "") 
        : read1(r1), read2(r2), read_id(id), is_paired(!r2.empty()) {}
};

// GPU Compact Hash Table (same as before)
struct GPUCompactHashTable {
    uint32_t* hash_cells;
    uint32_t table_size;
    uint32_t hash_mask;
    uint32_t lca_bits;
    uint32_t hash_bits;
};

// Classification parameters with paired-end options
struct ClassificationParams {
    int k = 35;
    int ell = 31;
    int spaces = 7;
    float confidence_threshold = 0.0f;
    bool use_spaced_seeds = true;
    int max_ambiguous_bases = 5;
    
    // Paired-end specific parameters
    bool use_paired_end_bonus = true;      // Boost for concordant pairs
    float paired_concordance_weight = 2.0f; // Weight multiplier for concordant pairs
    float min_pair_concordance = 0.5f;     // Minimum concordance to get bonus
    bool require_both_reads_classified = false; // Both reads must classify to get bonus
};

// Enhanced classification result with paired-end metrics
struct PairedReadClassification {
    uint32_t taxon_id;                 // Final classification (0 = unclassified)
    float confidence_score;            // Overall confidence [0,1]
    
    // Per-read metrics
    uint32_t read1_votes;              // Votes from read1
    uint32_t read2_votes;              // Votes from read2  
    uint32_t read1_kmers;              // Total k-mers in read1
    uint32_t read2_kmers;              // Total k-mers in read2
    
    // Paired-end metrics
    uint32_t concordant_votes;         // Votes where both reads agree
    float pair_concordance;            // Concordance rate [0,1]
    bool got_paired_bonus;             // Whether paired bonus was applied
    
    // Individual read classifications (for debugging)
    uint32_t read1_best_taxon;
    uint32_t read2_best_taxon;
    float read1_confidence;
    float read2_confidence;
};

// Taxonomy tree node
struct TaxonomyNode {
    uint32_t taxon_id;
    uint32_t parent_id;
    uint8_t rank;
    char name[64];
};

// GPU-accelerated Paired-End Kraken classifier
class PairedEndGPUKrakenClassifier {
private:
    // GPU data structures
    GPUCompactHashTable* d_hash_table;
    TaxonomyNode* d_taxonomy_tree;
    uint32_t* d_parent_lookup;
    uint32_t num_taxonomy_nodes;
    
    // Host taxonomy mapping
    std::unordered_map<uint32_t, std::string> taxon_names;
    std::unordered_map<uint32_t, uint32_t> taxon_parents;
    
    // Classification parameters
    ClassificationParams params;
    
    // GPU memory for paired-end batch processing
    char* d_reads_r1;
    char* d_reads_r2;
    uint32_t* d_read_offsets_r1;
    uint32_t* d_read_offsets_r2;
    uint32_t* d_read_lengths_r1;
    uint32_t* d_read_lengths_r2;
    bool* d_is_paired_flags;
    PairedReadClassification* d_results;
    
    // Paired-end voting arrays (per pair)
    uint32_t* d_pair_votes_r1;         // [pair_id * max_taxa + taxon_idx] = votes
    uint32_t* d_pair_votes_r2;
    uint32_t* d_concordant_votes;      // [pair_id * max_taxa + taxon_idx] = concordant votes
    
    // Configuration
    static const int DEFAULT_BATCH_SIZE = 10000;
    static const int MAX_READ_LENGTH = 1000;
    static const int THREADS_PER_BLOCK = 256;
    static const int MAX_TAXA_PER_PAIR = 128;  // Max taxa to track per pair
    
    bool database_loaded = false;
    
public:
    PairedEndGPUKrakenClassifier(const ClassificationParams& config = ClassificationParams());
    ~PairedEndGPUKrakenClassifier();
    
    // Database loading
    bool load_database(const std::string& database_directory);
    
    // Paired-end classification interface
    std::vector<PairedReadClassification> classify_paired_reads(
        const std::vector<PairedRead>& paired_reads
    );
    
    std::vector<PairedReadClassification> classify_paired_reads_batch(
        const std::vector<PairedRead>& paired_reads,
        int batch_size = DEFAULT_BATCH_SIZE
    );
    
    // Single-end compatibility (converts to unpaired PairedRead)
    std::vector<PairedReadClassification> classify_reads(
        const std::vector<std::string>& reads
    );
    
    // Configuration
    void set_confidence_threshold(float threshold) {
        params.confidence_threshold = threshold;
    }
    void set_paired_concordance_weight(float weight) {
        params.paired_concordance_weight = weight;
    }
    void enable_paired_end_bonus(bool enable) {
        params.use_paired_end_bonus = enable;
    }
    ClassificationParams get_params() const { return params; }
    
    // Utilities
    std::string get_taxon_name(uint32_t taxon_id) const;
    void print_database_stats() const;
    void print_paired_classification_stats(const std::vector<PairedReadClassification>& results) const;
    
    bool is_database_loaded() const { return database_loaded; }
    
private:
    // Internal methods
    bool allocate_gpu_memory(int max_pairs);
    void free_gpu_memory();
    
    bool load_hash_table(const std::string& hash_table_file);
    bool load_taxonomy_tree(const std::string& taxonomy_file);
    bool load_taxonomy_tree_from_db(const std::string& db_file);
    bool read_database_header(const std::string& db_file, ClassificationParams& db_params);
    
    void transfer_paired_reads_to_gpu(const std::vector<PairedRead>& paired_reads);
    void retrieve_paired_results_from_gpu(std::vector<PairedReadClassification>& results, int num_pairs);
};

// CUDA kernels for paired-end processing
__global__ void classify_paired_reads_kernel(
    const char* reads_r1,
    const char* reads_r2,
    const uint32_t* read_offsets_r1,
    const uint32_t* read_offsets_r2,
    const uint32_t* read_lengths_r1,
    const uint32_t* read_lengths_r2,
    const bool* is_paired_flags,
    const GPUCompactHashTable* hash_table,
    const TaxonomyNode* taxonomy_tree,
    const uint32_t* parent_lookup,
    uint32_t* pair_votes_r1,
    uint32_t* pair_votes_r2,
    uint32_t* concordant_votes,
    PairedReadClassification* results,
    int num_pairs,
    ClassificationParams params
);

__global__ void compute_paired_concordance_kernel(
    const uint32_t* pair_votes_r1,
    const uint32_t* pair_votes_r2,
    const bool* is_paired_flags,
    uint32_t* concordant_votes,
    PairedReadClassification* results,
    int num_pairs,
    ClassificationParams params
);

// Device functions
__device__ uint32_t lookup_lca_gpu(const GPUCompactHashTable* cht, uint64_t minimizer_hash);
__device__ uint64_t extract_minimizer_with_spaced_seeds(const char* sequence, int pos, int k, int ell, int spaces);
__device__ uint32_t find_paired_classification_with_confidence(
    const uint32_t* votes_r1,
    const uint32_t* votes_r2,
    const uint32_t* concordant_votes,
    int total_kmers_r1,
    int total_kmers_r2,
    float confidence_threshold,
    float paired_weight,
    bool use_paired_bonus,
    const TaxonomyNode* taxonomy_tree,
    const uint32_t* parent_lookup
);

// Utility functions
__device__ __host__ uint64_t hash_minimizer(const char* seq, int len);
__device__ __host__ uint64_t apply_spaced_seed_mask(uint64_t hash, int spaces);
__device__ __host__ bool has_ambiguous_bases(const char* seq, int len);
__device__ __host__ uint32_t compute_compact_hash(uint64_t minimizer_hash);
__device__ __host__ uint32_t jenkins_hash(uint64_t key);

#endif // GPU_KRAKEN_CLASSIFIER_PAIRED_CUH

// ================================================================
// IMPLEMENTATION
// ================================================================

#ifndef GPU_KRAKEN_CLASSIFIER_HEADER_ONLY
// Implementation continues below - no need to include self
#include <iostream>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <cstring>
#include <iomanip>
#include <sstream>

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// Constructor
PairedEndGPUKrakenClassifier::PairedEndGPUKrakenClassifier(const ClassificationParams& config)
    : params(config), d_hash_table(nullptr), d_taxonomy_tree(nullptr),
      d_parent_lookup(nullptr), num_taxonomy_nodes(0),
      d_reads_r1(nullptr), d_reads_r2(nullptr),
      d_read_offsets_r1(nullptr), d_read_offsets_r2(nullptr),
      d_read_lengths_r1(nullptr), d_read_lengths_r2(nullptr),
      d_is_paired_flags(nullptr), d_results(nullptr),
      d_pair_votes_r1(nullptr), d_pair_votes_r2(nullptr), d_concordant_votes(nullptr) {
    
    std::cout << "Initializing Paired-End GPU Kraken classifier..." << std::endl;
    std::cout << "Parameters: k=" << params.k << ", ell=" << params.ell 
              << ", spaces=" << params.spaces 
              << ", confidence=" << params.confidence_threshold << std::endl;
    std::cout << "Paired-end: bonus=" << (params.use_paired_end_bonus ? "ON" : "OFF")
              << ", weight=" << params.paired_concordance_weight << std::endl;
}

// Destructor (similar to single-end version)
PairedEndGPUKrakenClassifier::~PairedEndGPUKrakenClassifier() {
    free_gpu_memory();
    
    // Free database structures
    if (d_hash_table) {
        GPUCompactHashTable h_cht;
        CUDA_CHECK(hipMemcpy(&h_cht, d_hash_table, sizeof(GPUCompactHashTable), 
                             hipMemcpyDeviceToHost));
        if (h_cht.hash_cells) hipFree(h_cht.hash_cells);
        hipFree(d_hash_table);
    }
    if (d_taxonomy_tree) hipFree(d_taxonomy_tree);
    if (d_parent_lookup) hipFree(d_parent_lookup);
}

// Classify paired reads with batch processing
std::vector<PairedReadClassification> PairedEndGPUKrakenClassifier::classify_paired_reads_batch(
    const std::vector<PairedRead>& paired_reads,
    int batch_size) {
    
    if (!database_loaded) {
        throw std::runtime_error("Database not loaded. Call load_database() first.");
    }
    
    std::vector<PairedReadClassification> all_results;
    all_results.reserve(paired_reads.size());
    
    std::cout << "Classifying " << paired_reads.size() << " read pairs in batches of " 
              << batch_size << "..." << std::endl;
    
    auto start_time = std::chrono::high_resolution_clock::now();
    
    for (size_t batch_start = 0; batch_start < paired_reads.size(); batch_start += batch_size) {
        size_t batch_end = std::min(batch_start + batch_size, paired_reads.size());
        
        // Extract batch
        std::vector<PairedRead> batch_pairs(
            paired_reads.begin() + batch_start,
            paired_reads.begin() + batch_end
        );
        
        // Process batch
        auto batch_results = classify_paired_reads(batch_pairs);
        
        // Accumulate results
        all_results.insert(all_results.end(), 
                          batch_results.begin(), batch_results.end());
        
        // Report progress every 10 batches or at the end
        if ((batch_end - batch_size) % (batch_size * 10) == 0 || batch_end == paired_reads.size()) {
            std::cout << "Processed " << batch_end << "/" << paired_reads.size() 
                      << " read pairs..." << std::endl;
        }
    }
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    
    double pairs_per_second = paired_reads.size() * 1000.0 / duration.count();
    std::cout << "Paired-end classification completed in " << duration.count() << " ms" << std::endl;
    std::cout << "Performance: " << std::fixed << std::setprecision(0) 
              << pairs_per_second << " pairs/second" << std::endl;
    
    return all_results;
}

// Classify single batch of paired reads
std::vector<PairedReadClassification> PairedEndGPUKrakenClassifier::classify_paired_reads(
    const std::vector<PairedRead>& paired_reads) {
    
    if (paired_reads.empty()) return {};
    
    int num_pairs = paired_reads.size();
    
    // Allocate GPU memory for this batch
    if (!allocate_gpu_memory(num_pairs)) {
        throw std::runtime_error("Failed to allocate GPU memory");
    }
    
    // Transfer paired reads to GPU
    transfer_paired_reads_to_gpu(paired_reads);
    
    // Reset voting arrays
    size_t vote_array_size = num_pairs * MAX_TAXA_PER_PAIR * sizeof(uint32_t);
    CUDA_CHECK(hipMemset(d_pair_votes_r1, 0, vote_array_size));
    CUDA_CHECK(hipMemset(d_pair_votes_r2, 0, vote_array_size));
    CUDA_CHECK(hipMemset(d_concordant_votes, 0, vote_array_size));
    
    // Launch paired-end classification kernel
    int num_blocks = (num_pairs + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    classify_paired_reads_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        d_reads_r1, d_reads_r2,
        d_read_offsets_r1, d_read_offsets_r2,
        d_read_lengths_r1, d_read_lengths_r2,
        d_is_paired_flags, d_hash_table, d_taxonomy_tree, d_parent_lookup,
        d_pair_votes_r1, d_pair_votes_r2, d_concordant_votes,
        d_results, num_pairs, params
    );
    
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Compute paired concordance and final classification
    compute_paired_concordance_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(
        d_pair_votes_r1, d_pair_votes_r2, d_is_paired_flags,
        d_concordant_votes, d_results, num_pairs, params
    );
    
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Retrieve results
    std::vector<PairedReadClassification> results(num_pairs);
    retrieve_paired_results_from_gpu(results, num_pairs);
    
    return results;
}

// Allocate GPU memory for paired-end processing
bool PairedEndGPUKrakenClassifier::allocate_gpu_memory(int max_pairs) {
    // Free any existing memory
    free_gpu_memory();
    
    size_t read_buffer_size = max_pairs * MAX_READ_LENGTH;
    size_t vote_array_size = max_pairs * MAX_TAXA_PER_PAIR * sizeof(uint32_t);
    
    CUDA_CHECK(hipMalloc(&d_reads_r1, read_buffer_size));
    CUDA_CHECK(hipMalloc(&d_reads_r2, read_buffer_size));
    CUDA_CHECK(hipMalloc(&d_read_offsets_r1, max_pairs * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_read_offsets_r2, max_pairs * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_read_lengths_r1, max_pairs * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_read_lengths_r2, max_pairs * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_is_paired_flags, max_pairs * sizeof(bool)));
    CUDA_CHECK(hipMalloc(&d_results, max_pairs * sizeof(PairedReadClassification)));
    
    // Paired-end voting arrays
    CUDA_CHECK(hipMalloc(&d_pair_votes_r1, vote_array_size));
    CUDA_CHECK(hipMalloc(&d_pair_votes_r2, vote_array_size));
    CUDA_CHECK(hipMalloc(&d_concordant_votes, vote_array_size));
    
    return true;
}

// Free GPU memory
void PairedEndGPUKrakenClassifier::free_gpu_memory() {
    if (d_reads_r1) { hipFree(d_reads_r1); d_reads_r1 = nullptr; }
    if (d_reads_r2) { hipFree(d_reads_r2); d_reads_r2 = nullptr; }
    if (d_read_offsets_r1) { hipFree(d_read_offsets_r1); d_read_offsets_r1 = nullptr; }
    if (d_read_offsets_r2) { hipFree(d_read_offsets_r2); d_read_offsets_r2 = nullptr; }
    if (d_read_lengths_r1) { hipFree(d_read_lengths_r1); d_read_lengths_r1 = nullptr; }
    if (d_read_lengths_r2) { hipFree(d_read_lengths_r2); d_read_lengths_r2 = nullptr; }
    if (d_is_paired_flags) { hipFree(d_is_paired_flags); d_is_paired_flags = nullptr; }
    if (d_results) { hipFree(d_results); d_results = nullptr; }
    if (d_pair_votes_r1) { hipFree(d_pair_votes_r1); d_pair_votes_r1 = nullptr; }
    if (d_pair_votes_r2) { hipFree(d_pair_votes_r2); d_pair_votes_r2 = nullptr; }
    if (d_concordant_votes) { hipFree(d_concordant_votes); d_concordant_votes = nullptr; }
}

// Transfer paired reads to GPU
void PairedEndGPUKrakenClassifier::transfer_paired_reads_to_gpu(const std::vector<PairedRead>& paired_reads) {
    // Prepare data for both reads
    std::string concatenated_r1, concatenated_r2;
    std::vector<uint32_t> offsets_r1, offsets_r2;
    std::vector<uint32_t> lengths_r1, lengths_r2;
    std::vector<uint8_t> is_paired_flags;
    
    uint32_t current_offset_r1 = 0, current_offset_r2 = 0;
    
    for (const auto& pair : paired_reads) {
        // Read 1
        offsets_r1.push_back(current_offset_r1);
        lengths_r1.push_back(pair.read1.length());
        concatenated_r1 += pair.read1;
        current_offset_r1 += pair.read1.length();
        
        // Read 2 (or empty if single-end)
        offsets_r2.push_back(current_offset_r2);
        if (pair.is_paired && !pair.read2.empty()) {
            lengths_r2.push_back(pair.read2.length());
            concatenated_r2 += pair.read2;
            current_offset_r2 += pair.read2.length();
        } else {
            lengths_r2.push_back(0);
            // Add empty placeholder to maintain alignment
        }
        
        is_paired_flags.push_back(pair.is_paired);
    }
    
    // Transfer to GPU
    CUDA_CHECK(hipMemcpy(d_reads_r1, concatenated_r1.c_str(),
                         concatenated_r1.length(), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_reads_r2, concatenated_r2.c_str(),
                         concatenated_r2.length(), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMemcpy(d_read_offsets_r1, offsets_r1.data(),
                         offsets_r1.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_read_offsets_r2, offsets_r2.data(),
                         offsets_r2.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMemcpy(d_read_lengths_r1, lengths_r1.data(),
                         lengths_r1.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_read_lengths_r2, lengths_r2.data(),
                         lengths_r2.size() * sizeof(uint32_t), hipMemcpyHostToDevice));
    
    CUDA_CHECK(hipMemcpy(d_is_paired_flags, is_paired_flags.data(),
                         is_paired_flags.size() * sizeof(uint8_t), hipMemcpyHostToDevice));
}

// Single-end compatibility wrapper
std::vector<PairedReadClassification> PairedEndGPUKrakenClassifier::classify_reads(
    const std::vector<std::string>& reads) {
    
    // Convert to paired format
    std::vector<PairedRead> paired_reads;
    paired_reads.reserve(reads.size());
    
    for (size_t i = 0; i < reads.size(); i++) {
        paired_reads.emplace_back(reads[i], "", "read_" + std::to_string(i));
    }
    
    return classify_paired_reads_batch(paired_reads);
}

// Print paired-end classification statistics
void PairedEndGPUKrakenClassifier::print_paired_classification_stats(
    const std::vector<PairedReadClassification>& results) const {
    
    if (results.empty()) {
        std::cout << "No classification results to analyze" << std::endl;
        return;
    }
    
    int classified = 0;
    int unclassified = 0;
    int paired_bonus_count = 0;
    float total_confidence = 0.0f;
    float total_concordance = 0.0f;
    int paired_count = 0;
    
    std::unordered_map<uint32_t, int> taxon_counts;
    
    for (const auto& result : results) {
        if (result.taxon_id > 0) {
            classified++;
            total_confidence += result.confidence_score;
            taxon_counts[result.taxon_id]++;
        } else {
            unclassified++;
        }
        
        if (result.got_paired_bonus) {
            paired_bonus_count++;
        }
        
        // Check if this was a paired read
        if (result.read2_kmers > 0) {
            paired_count++;
            total_concordance += result.pair_concordance;
        }
    }
    
    std::cout << "\n=== PAIRED-END CLASSIFICATION STATISTICS ===" << std::endl;
    std::cout << "Total read pairs: " << results.size() << std::endl;
    std::cout << "Paired reads: " << paired_count << std::endl;
    std::cout << "Single reads: " << (results.size() - paired_count) << std::endl;
    std::cout << "Classified: " << classified 
              << " (" << std::fixed << std::setprecision(1) 
              << (100.0 * classified / results.size()) << "%)" << std::endl;
    std::cout << "Unclassified: " << unclassified 
              << " (" << std::fixed << std::setprecision(1) 
              << (100.0 * unclassified / results.size()) << "%)" << std::endl;
    
    if (classified > 0) {
        std::cout << "Average confidence: " << std::fixed << std::setprecision(3) 
                  << (total_confidence / classified) << std::endl;
        std::cout << "Unique taxa detected: " << taxon_counts.size() << std::endl;
    }
    
    if (paired_count > 0) {
        std::cout << "Average pair concordance: " << std::fixed << std::setprecision(3) 
                  << (total_concordance / paired_count) << std::endl;
        std::cout << "Pairs with concordance bonus: " << paired_bonus_count 
                  << " (" << std::fixed << std::setprecision(1) 
                  << (100.0 * paired_bonus_count / paired_count) << "%)" << std::endl;
    }
}

// ================================================================
// CUDA KERNELS FOR PAIRED-END PROCESSING
// ================================================================

// Main paired-end classification kernel
__global__ void classify_paired_reads_kernel(
    const char* reads_r1,
    const char* reads_r2,
    const uint32_t* read_offsets_r1,
    const uint32_t* read_offsets_r2,
    const uint32_t* read_lengths_r1,
    const uint32_t* read_lengths_r2,
    const bool* is_paired_flags,
    const GPUCompactHashTable* hash_table,
    const TaxonomyNode* taxonomy_tree,
    const uint32_t* parent_lookup,
    uint32_t* pair_votes_r1,
    uint32_t* pair_votes_r2,
    uint32_t* concordant_votes,
    PairedReadClassification* results,
    int num_pairs,
    ClassificationParams params) {
    
    int pair_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (pair_id >= num_pairs) return;
    
    const char* read1 = reads_r1 + read_offsets_r1[pair_id];
    const char* read2 = reads_r2 + read_offsets_r2[pair_id];
    uint32_t len1 = read_lengths_r1[pair_id];
    uint32_t len2 = read_lengths_r2[pair_id];
    bool is_paired = is_paired_flags[pair_id];
    
    // Initialize result
    PairedReadClassification& result = results[pair_id];
    result.taxon_id = 0;
    result.confidence_score = 0.0f;
    result.read1_votes = 0;
    result.read2_votes = 0;
    result.read1_kmers = 0;
    result.read2_kmers = 0;
    result.concordant_votes = 0;
    result.pair_concordance = 0.0f;
    result.got_paired_bonus = false;
    result.read1_best_taxon = 0;
    result.read2_best_taxon = 0;
    result.read1_confidence = 0.0f;
    result.read2_confidence = 0.0f;
    
    // Local voting arrays for this pair
    const int MAX_LOCAL_TAXA = 32;
    uint32_t local_taxon_ids_r1[MAX_LOCAL_TAXA];
    uint32_t local_vote_counts_r1[MAX_LOCAL_TAXA];
    uint32_t local_taxon_ids_r2[MAX_LOCAL_TAXA];
    uint32_t local_vote_counts_r2[MAX_LOCAL_TAXA];
    int num_taxa_r1 = 0, num_taxa_r2 = 0;
    
    // Process Read 1
    if (len1 >= params.k) {
        int total_kmers_r1 = len1 - params.k + 1;
        for (int i = 0; i < total_kmers_r1; i++) {
            if (has_ambiguous_bases(read1 + i, params.k)) continue;
            
            result.read1_kmers++;
            
            uint64_t minimizer = extract_minimizer_with_spaced_seeds(
                read1, i, params.k, params.ell, params.spaces
            );
            
            if (minimizer == UINT64_MAX) continue;
            
            uint32_t lca = lookup_lca_gpu(hash_table, minimizer);
            if (lca == 0) continue;
            
            // Add vote for read1
            bool found = false;
            for (int j = 0; j < num_taxa_r1; j++) {
                if (local_taxon_ids_r1[j] == lca) {
                    local_vote_counts_r1[j]++;
                    found = true;
                    break;
                }
            }
            
            if (!found && num_taxa_r1 < MAX_LOCAL_TAXA) {
                local_taxon_ids_r1[num_taxa_r1] = lca;
                local_vote_counts_r1[num_taxa_r1] = 1;
                num_taxa_r1++;
            }
        }
    }
    
    // Process Read 2 (if paired)
    if (is_paired && len2 >= params.k) {
        int total_kmers_r2 = len2 - params.k + 1;
        for (int i = 0; i < total_kmers_r2; i++) {
            if (has_ambiguous_bases(read2 + i, params.k)) continue;
            
            result.read2_kmers++;
            
            uint64_t minimizer = extract_minimizer_with_spaced_seeds(
                read2, i, params.k, params.ell, params.spaces
            );
            
            if (minimizer == UINT64_MAX) continue;
            
            uint32_t lca = lookup_lca_gpu(hash_table, minimizer);
            if (lca == 0) continue;
            
            // Add vote for read2
            bool found = false;
            for (int j = 0; j < num_taxa_r2; j++) {
                if (local_taxon_ids_r2[j] == lca) {
                    local_vote_counts_r2[j]++;
                    found = true;
                    break;
                }
            }
            
            if (!found && num_taxa_r2 < MAX_LOCAL_TAXA) {
                local_taxon_ids_r2[num_taxa_r2] = lca;
                local_vote_counts_r2[num_taxa_r2] = 1;
                num_taxa_r2++;
            }
        }
    }
    
    // Find best taxon for each read
    uint32_t best_taxon_r1 = 0, best_votes_r1 = 0;
    uint32_t best_taxon_r2 = 0, best_votes_r2 = 0;
    
    for (int i = 0; i < num_taxa_r1; i++) {
        if (local_vote_counts_r1[i] > best_votes_r1) {
            best_votes_r1 = local_vote_counts_r1[i];
            best_taxon_r1 = local_taxon_ids_r1[i];
        }
    }
    
    for (int i = 0; i < num_taxa_r2; i++) {
        if (local_vote_counts_r2[i] > best_votes_r2) {
            best_votes_r2 = local_vote_counts_r2[i];
            best_taxon_r2 = local_taxon_ids_r2[i];
        }
    }
    
    result.read1_best_taxon = best_taxon_r1;
    result.read2_best_taxon = best_taxon_r2;
    result.read1_votes = best_votes_r1;
    result.read2_votes = best_votes_r2;
    result.read1_confidence = (result.read1_kmers > 0) ? 
        (float)best_votes_r1 / result.read1_kmers : 0.0f;
    result.read2_confidence = (result.read2_kmers > 0) ? 
        (float)best_votes_r2 / result.read2_kmers : 0.0f;
    
    // Calculate concordance and final classification
    if (is_paired && best_taxon_r1 > 0 && best_taxon_r2 > 0) {
        // Check if both reads agree
        if (best_taxon_r1 == best_taxon_r2) {
            result.concordant_votes = best_votes_r1 + best_votes_r2;
            result.pair_concordance = 1.0f;  // Perfect concordance
            
            // Apply paired-end bonus
            if (params.use_paired_end_bonus && 
                result.pair_concordance >= params.min_pair_concordance) {
                result.taxon_id = best_taxon_r1;
                result.confidence_score = (result.read1_confidence + result.read2_confidence) * 
                                        params.paired_concordance_weight / 2.0f;
                result.got_paired_bonus = true;
            } else {
                result.taxon_id = best_taxon_r1;
                result.confidence_score = (result.read1_confidence + result.read2_confidence) / 2.0f;
            }
        } else {
            // Reads disagree - use better one or fall back to single-end logic
            if (result.read1_confidence > result.read2_confidence) {
                result.taxon_id = best_taxon_r1;
                result.confidence_score = result.read1_confidence;
            } else {
                result.taxon_id = best_taxon_r2;
                result.confidence_score = result.read2_confidence;
            }
            result.pair_concordance = 0.0f;
        }
    } else {
        // Single-end classification or only one read classified
        if (best_taxon_r1 > 0 && best_taxon_r2 == 0) {
            result.taxon_id = best_taxon_r1;
            result.confidence_score = result.read1_confidence;
        } else if (best_taxon_r2 > 0 && best_taxon_r1 == 0) {
            result.taxon_id = best_taxon_r2;
            result.confidence_score = result.read2_confidence;
        } else if (best_taxon_r1 > 0 && best_taxon_r2 > 0) {
            // Both classified but not paired, use better one
            if (result.read1_confidence > result.read2_confidence) {
                result.taxon_id = best_taxon_r1;
                result.confidence_score = result.read1_confidence;
            } else {
                result.taxon_id = best_taxon_r2;
                result.confidence_score = result.read2_confidence;
            }
        }
    }
    
    // Apply confidence threshold
    if (result.confidence_score < params.confidence_threshold) {
        result.taxon_id = 0;  // Unclassified
    }
}

// Compute paired concordance kernel (additional processing if needed)
__global__ void compute_paired_concordance_kernel(
    const uint32_t* pair_votes_r1,
    const uint32_t* pair_votes_r2,
    const bool* is_paired_flags,
    uint32_t* concordant_votes,
    PairedReadClassification* results,
    int num_pairs,
    ClassificationParams params) {
    
    int pair_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (pair_id >= num_pairs) return;
    
    // This kernel can be used for additional paired-end processing
    // if needed, such as more sophisticated LCA computation across
    // read pairs. For now, the main kernel handles everything.
}

// Retrieve paired results from GPU
void PairedEndGPUKrakenClassifier::retrieve_paired_results_from_gpu(
    std::vector<PairedReadClassification>& results, int num_pairs) {
    
    CUDA_CHECK(hipMemcpy(results.data(), d_results,
                         num_pairs * sizeof(PairedReadClassification),
                         hipMemcpyDeviceToHost));
}

// Database loading methods
bool PairedEndGPUKrakenClassifier::load_database(const std::string& database_directory) {
    std::cout << "Loading database from " << database_directory << "..." << std::endl;
    
    // Load hash table
    std::string hash_table_file = database_directory + "/hash_table.k2d";
    if (!load_hash_table(hash_table_file)) {
        std::cerr << "Failed to load hash table from " << hash_table_file << std::endl;
        return false;
    }
    
    // Load taxonomy
    std::string taxonomy_file = database_directory + "/taxonomy.tsv";
    if (!load_taxonomy_tree(taxonomy_file)) {
        std::cerr << "Failed to load taxonomy from " << taxonomy_file << std::endl;
        return false;
    }
    
    database_loaded = true;
    std::cout << "Database loaded successfully!" << std::endl;
    return true;
}

bool PairedEndGPUKrakenClassifier::load_hash_table(const std::string& hash_table_file) {
    std::ifstream hash_in(hash_table_file, std::ios::binary);
    if (!hash_in.is_open()) {
        std::cerr << "Cannot open hash table file: " << hash_table_file << std::endl;
        return false;
    }
    
    // Read header
    uint64_t table_size, num_entries;
    hash_in.read(reinterpret_cast<char*>(&table_size), sizeof(uint64_t));
    hash_in.read(reinterpret_cast<char*>(&num_entries), sizeof(uint64_t));
    
    std::cout << "Hash table size: " << table_size << ", entries: " << num_entries << std::endl;
    
    // Create compact hash table on GPU
    GPUCompactHashTable h_cht;
    h_cht.table_size = table_size;
    h_cht.hash_mask = table_size - 1;
    h_cht.lca_bits = 20;  // Assuming 20 bits for LCA (up to 1M taxa)
    h_cht.hash_bits = 32 - h_cht.lca_bits;
    
    // Allocate hash table on GPU
    CUDA_CHECK(hipMalloc(&h_cht.hash_cells, table_size * sizeof(uint32_t)));
    CUDA_CHECK(hipMemset(h_cht.hash_cells, 0, table_size * sizeof(uint32_t)));
    
    // Allocate and copy hash table structure to GPU
    CUDA_CHECK(hipMalloc(&d_hash_table, sizeof(GPUCompactHashTable)));
    CUDA_CHECK(hipMemcpy(d_hash_table, &h_cht, sizeof(GPUCompactHashTable), hipMemcpyHostToDevice));
    
    // Read entries and build compact hash table on host first
    std::vector<uint32_t> host_hash_cells(table_size, 0);
    
    for (uint64_t i = 0; i < num_entries; i++) {
        uint64_t minimizer_hash;
        uint32_t lca_taxon, genome_count;
        float uniqueness_score;
        
        hash_in.read(reinterpret_cast<char*>(&minimizer_hash), sizeof(uint64_t));
        hash_in.read(reinterpret_cast<char*>(&lca_taxon), sizeof(uint32_t));
        hash_in.read(reinterpret_cast<char*>(&genome_count), sizeof(uint32_t));
        hash_in.read(reinterpret_cast<char*>(&uniqueness_score), sizeof(float));
        
        // Compute compact hash
        uint32_t compact_hash = compute_compact_hash(minimizer_hash);
        uint32_t pos = compact_hash & h_cht.hash_mask;
        
        // Linear probing to find empty slot
        int probes = 0;
        while (host_hash_cells[pos] != 0 && probes < 32) {
            pos = (pos + 1) & h_cht.hash_mask;
            probes++;
        }
        
        if (probes < 32) {
            // Store compact hash (upper bits) and LCA (lower bits)
            uint32_t stored_hash = compact_hash >> h_cht.lca_bits;
            uint32_t cell_value = (stored_hash << h_cht.lca_bits) | (lca_taxon & ((1U << h_cht.lca_bits) - 1));
            host_hash_cells[pos] = cell_value;
        }
    }
    
    // Copy hash table to GPU
    CUDA_CHECK(hipMemcpy(h_cht.hash_cells, host_hash_cells.data(), 
                         table_size * sizeof(uint32_t), hipMemcpyHostToDevice));
    
    hash_in.close();
    std::cout << "Hash table loaded: " << num_entries << " entries" << std::endl;
    return true;
}

bool PairedEndGPUKrakenClassifier::load_taxonomy_tree_from_db(const std::string& db_file) {
    std::ifstream in(db_file, std::ios::binary);
    if (!in.is_open()) {
        std::cerr << "Cannot open database file for taxonomy: " << db_file << std::endl;
        return false;
    }
    
    // Read header to get organism count
    uint32_t magic, version, k_size, m_size, num_organisms;
    uint64_t num_minimizer_hashes;
    
    in.read(reinterpret_cast<char*>(&magic), sizeof(magic));
    in.read(reinterpret_cast<char*>(&version), sizeof(version));
    in.read(reinterpret_cast<char*>(&k_size), sizeof(k_size));
    in.read(reinterpret_cast<char*>(&m_size), sizeof(m_size));
    in.read(reinterpret_cast<char*>(&num_organisms), sizeof(num_organisms));
    in.read(reinterpret_cast<char*>(&num_minimizer_hashes), sizeof(num_minimizer_hashes));
    
    std::cout << "Loading taxonomy for " << num_organisms << " organisms..." << std::endl;
    
    // Read organism metadata and build taxonomy mapping
    for (uint32_t i = 0; i < num_organisms; i++) {
        uint32_t taxonomy_id, taxon_level, minimizer_count;
        uint64_t genome_size;
        float gc_content;
        
        in.read(reinterpret_cast<char*>(&taxonomy_id), sizeof(taxonomy_id));
        in.read(reinterpret_cast<char*>(&taxon_level), sizeof(taxon_level));
        in.read(reinterpret_cast<char*>(&gc_content), sizeof(gc_content));
        in.read(reinterpret_cast<char*>(&genome_size), sizeof(genome_size));
        in.read(reinterpret_cast<char*>(&minimizer_count), sizeof(minimizer_count));
        
        // Read organism name
        uint16_t name_length;
        in.read(reinterpret_cast<char*>(&name_length), sizeof(name_length));
        std::string name(name_length, '\0');
        in.read(&name[0], name_length);
        
        // Read taxonomy path
        uint16_t taxonomy_length;
        in.read(reinterpret_cast<char*>(&taxonomy_length), sizeof(taxonomy_length));
        std::string taxonomy_path(taxonomy_length, '\0');
        in.read(&taxonomy_path[0], taxonomy_length);
        
        // Store in taxonomy mappings
        taxon_names[taxonomy_id] = name;
        taxon_parents[taxonomy_id] = 0;  // Simple mapping for now
        
        // Create a simple taxonomy node
        TaxonomyNode node;
        node.taxon_id = taxonomy_id;
        node.parent_id = 0;
        node.rank = taxon_level;
        strncpy(node.name, name.c_str(), 63);
        node.name[63] = '\0';
    }
    
    in.close();
    
    // Create simple parent lookup (all point to root for now)
    std::vector<uint32_t> parent_lookup(1000000, 0);  // Support up to 1M taxa
    
    // Allocate taxonomy tree on GPU (simplified)
    std::vector<TaxonomyNode> host_taxonomy;
    for (const auto& [taxon_id, name] : taxon_names) {
        TaxonomyNode node;
        node.taxon_id = taxon_id;
        node.parent_id = 0;
        node.rank = 0;
        strncpy(node.name, name.c_str(), 63);
        node.name[63] = '\0';
        host_taxonomy.push_back(node);
    }
    
    num_taxonomy_nodes = host_taxonomy.size();
    
    if (num_taxonomy_nodes > 0) {
        CUDA_CHECK(hipMalloc(&d_taxonomy_tree, num_taxonomy_nodes * sizeof(TaxonomyNode)));
        CUDA_CHECK(hipMemcpy(d_taxonomy_tree, host_taxonomy.data(), 
                             num_taxonomy_nodes * sizeof(TaxonomyNode), hipMemcpyHostToDevice));
        
        CUDA_CHECK(hipMalloc(&d_parent_lookup, 1000000 * sizeof(uint32_t)));
        CUDA_CHECK(hipMemcpy(d_parent_lookup, parent_lookup.data(), 
                             1000000 * sizeof(uint32_t), hipMemcpyHostToDevice));
    }
    
    std::cout << "Loaded taxonomy for " << taxon_names.size() << " organisms" << std::endl;
    return true;
}

bool PairedEndGPUKrakenClassifier::load_taxonomy_tree(const std::string& taxonomy_file) {
    std::ifstream tax_in(taxonomy_file);
    if (!tax_in.is_open()) {
        std::cerr << "Cannot open taxonomy file: " << taxonomy_file << std::endl;
        return false;
    }
    
    std::string line;
    // Skip header
    std::getline(tax_in, line);
    
    std::vector<TaxonomyNode> host_taxonomy;
    uint32_t max_taxon_id = 0;
    
    while (std::getline(tax_in, line)) {
        std::istringstream iss(line);
        uint32_t taxon_id, parent_id;
        std::string name;
        
        if (iss >> taxon_id) {
            iss.ignore(1); // skip tab
            std::getline(iss, name, '\t');
            iss >> parent_id;
            
            taxon_names[taxon_id] = name;
            taxon_parents[taxon_id] = parent_id;
            
            TaxonomyNode node;
            node.taxon_id = taxon_id;
            node.parent_id = parent_id;
            node.rank = 0;  // Not used for now
            strncpy(node.name, name.c_str(), 63);
            node.name[63] = '\0';
            
            host_taxonomy.push_back(node);
            max_taxon_id = std::max(max_taxon_id, taxon_id);
        }
    }
    
    tax_in.close();
    
    num_taxonomy_nodes = host_taxonomy.size();
    std::cout << "Loaded " << num_taxonomy_nodes << " taxonomy nodes, max ID: " << max_taxon_id << std::endl;
    
    // Allocate and copy taxonomy to GPU
    CUDA_CHECK(hipMalloc(&d_taxonomy_tree, num_taxonomy_nodes * sizeof(TaxonomyNode)));
    CUDA_CHECK(hipMemcpy(d_taxonomy_tree, host_taxonomy.data(), 
                         num_taxonomy_nodes * sizeof(TaxonomyNode), hipMemcpyHostToDevice));
    
    // Create parent lookup table for fast access
    std::vector<uint32_t> parent_lookup(max_taxon_id + 1, 0);
    for (const auto& [taxon_id, parent_id] : taxon_parents) {
        if (taxon_id <= max_taxon_id) {
            parent_lookup[taxon_id] = parent_id;
        }
    }
    
    CUDA_CHECK(hipMalloc(&d_parent_lookup, (max_taxon_id + 1) * sizeof(uint32_t)));
    CUDA_CHECK(hipMemcpy(d_parent_lookup, parent_lookup.data(), 
                         (max_taxon_id + 1) * sizeof(uint32_t), hipMemcpyHostToDevice));
    
    return true;
}

// Utility device functions (same as single-end version)
__device__ uint32_t lookup_lca_gpu(const GPUCompactHashTable* cht, uint64_t minimizer_hash) {
    uint32_t compact_hash = compute_compact_hash(minimizer_hash);
    uint32_t pos = compact_hash & cht->hash_mask;
    uint32_t lca_mask = (1U << cht->lca_bits) - 1;
    
    for (int probe = 0; probe < 32; probe++) {
        uint32_t cell = cht->hash_cells[pos];
        if (cell == 0) return 0;
        
        uint32_t stored_hash = cell >> cht->lca_bits;
        uint32_t expected_hash = compact_hash >> cht->lca_bits;
        
        if (stored_hash == expected_hash) {
            return cell & lca_mask;
        }
        
        pos = (pos + 1) & cht->hash_mask;
    }
    
    return 0;
}

__device__ uint64_t extract_minimizer_with_spaced_seeds(
    const char* sequence, int pos, int k, int ell, int spaces) {
    
    uint64_t min_hash = UINT64_MAX;
    
    for (int i = 0; i <= k - ell; i++) {
        uint64_t hash = hash_minimizer(sequence + pos + i, ell);
        if (hash == UINT64_MAX) continue;
        
        if (spaces > 0) {
            hash = apply_spaced_seed_mask(hash, spaces);
        }
        
        if (hash < min_hash) {
            min_hash = hash;
        }
    }
    
    return min_hash;
}

// Utility functions (same as before)
__device__ __host__ uint64_t hash_minimizer(const char* seq, int len) {
    uint64_t hash = 0;
    for (int i = 0; i < len; i++) {
        int base;
        switch (seq[i]) {
            case 'A': case 'a': base = 0; break;
            case 'C': case 'c': base = 1; break;
            case 'G': case 'g': base = 2; break;
            case 'T': case 't': base = 3; break;
            default: return UINT64_MAX;
        }
        hash = (hash << 2) | base;
    }
    return hash;
}

__device__ __host__ uint64_t apply_spaced_seed_mask(uint64_t hash, int spaces) {
    uint64_t masked = 0;
    int out_pos = 0;
    
    for (int i = 0; i < 32; i++) {
        if (i % (spaces + 1) != 0) {
            masked |= ((hash >> (i * 2)) & 3ULL) << (out_pos * 2);
            out_pos++;
        }
    }
    
    return masked;
}

__device__ __host__ bool has_ambiguous_bases(const char* seq, int len) {
    for (int i = 0; i < len; i++) {
        char c = seq[i];
        if (c != 'A' && c != 'C' && c != 'G' && c != 'T' &&
            c != 'a' && c != 'c' && c != 'g' && c != 't') {
            return true;
        }
    }
    return false;
}

__device__ __host__ uint32_t compute_compact_hash(uint64_t minimizer_hash) {
    return jenkins_hash(minimizer_hash) & 0x7FFFFFFF;
}

__device__ __host__ uint32_t jenkins_hash(uint64_t key) {
    uint32_t hash = (uint32_t)(key ^ (key >> 32));
    hash += (hash << 10);
    hash ^= (hash >> 6);
    hash += (hash << 3);
    hash ^= (hash >> 11);
    hash += (hash << 15);
    return hash;
}

#endif // GPU_KRAKEN_CLASSIFIER_HEADER_ONLY

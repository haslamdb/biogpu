#include "hip/hip_runtime.h"
// test_all_features_streaming.cu
// Test all implemented features with proper streaming approach
// Enhanced to include uniqueness scoring and co-occurrence analysis

#include <iostream>
#include <vector>
#include <unordered_map>
#include <filesystem>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <numeric>

#include "processing/genome_file_processor.h"
#include "gpu/gpu_minimizer_extraction.cuh"
#include "features/enhanced_minimizer_flags.h"
#include "features/namespace_conflict_resolution.h"
#include "gpu_kraken_types.h"
// #include "processing/minimizer_feature_extractor.h"
// Forward declare what we need
class MinimizerFeatureExtractor {
public:
    MinimizerFeatureExtractor(size_t max_minimizers, size_t max_genomes) {}
    void process_first_pass(GPUMinimizerHit* d_hits, size_t num_hits, const std::vector<uint32_t>& taxons) {}
    void process_second_pass(GPUMinimizerHit* d_hits, size_t num_hits) {}
    struct Statistics {
        size_t total_unique_minimizers = 0;
        size_t singleton_minimizers = 0;
        size_t rare_minimizers = 0;
        size_t common_minimizers = 0;
        double avg_taxonomic_spread = 0.0;
    };
    Statistics get_statistics() const { return Statistics(); }
};
// Forward declarations for feature functions
bool compute_and_encode_uniqueness_scores(
    GPUMinimizerHit* d_minimizer_hits,
    size_t num_hits,
    const std::vector<uint32_t>& taxon_ids,
    size_t num_genomes);

bool compute_and_encode_cooccurrence_scores(
    GPUMinimizerHit* d_minimizer_hits,
    size_t num_hits,
    const std::vector<uint64_t>& unique_minimizers,
    const std::vector<uint32_t>& occurrence_counts,
    const std::vector<GPUGenomeInfo>& genome_info);

// Enhanced feature analysis functions
void analyze_minimizer_features(const std::vector<GPUMinimizerHit>& hits) {
    std::cout << "\n=== COMPREHENSIVE FEATURE ANALYSIS ===" << std::endl;
    std::cout << "Total minimizers: " << hits.size() << std::endl;
    
    // Uniqueness analysis
    std::vector<size_t> uniqueness_counts(8, 0);
    std::vector<size_t> cooccurrence_counts(8, 0);
    std::vector<size_t> gc_counts(8, 0);
    std::vector<size_t> complexity_counts(8, 0);
    
    size_t position_clustered = 0;
    size_t position_uniform = 0;
    size_t contamination_risk = 0;
    size_t unique_class = 0;
    size_t canonical_class = 0;
    size_t redundant_class = 0;
    
    // NEW: Enhanced uniqueness tracking
    size_t unique_minimizers = 0;      // ≥90% uniqueness
    size_t rare_minimizers = 0;        // ≤3 occurrences
    size_t reliable_minimizers = 0;   // Suitable for classification
    
    std::vector<float> ml_weights;
    double uniqueness_sum = 0.0;
    
    for (const auto& hit : hits) {
        uint32_t flags = hit.feature_flags;
        uint16_t strand_flags = hit.strand;
        
        // Get uniqueness category (bits 8-10)
        uint8_t uniqueness = MinimizerFlags::get_uniqueness_category_safe(flags);
        if (uniqueness < uniqueness_counts.size()) {
            uniqueness_counts[uniqueness]++;
        }
        
        // Track unique/rare/reliable flags
        if (MinimizerFlags::is_unique_minimizer_safe(flags)) {
            unique_minimizers++;
        }
        if (MinimizerFlags::is_rare_minimizer_safe(flags)) {
            rare_minimizers++;
        }
        if (MinimizerFlags::is_reliable_minimizer_safe(flags)) {
            reliable_minimizers++;
        }
        
        // Get co-occurrence score (bits 14-16)
        uint8_t cooccurrence = MinimizerFlags::get_cooccurrence_score(flags);
        if (cooccurrence < cooccurrence_counts.size()) {
            cooccurrence_counts[cooccurrence]++;
        }
        
        uint8_t gc_category = MinimizerFlags::get_gc_content_category(flags);
        if (gc_category < gc_counts.size()) {
            gc_counts[gc_category]++;
        }
        
        uint8_t complexity = MinimizerFlags::get_complexity_score(flags);
        if (complexity < complexity_counts.size()) {
            complexity_counts[complexity]++;
        }
        
        if (MinimizerFlags::has_position_bias(flags)) {
            position_clustered++;
        } else {
            position_uniform++;
        }
        
        if (MinimizerFlags::has_contamination_risk(flags)) {
            contamination_risk++;
        }
        
        uint32_t classification = MinimizerFlags::get_classification(strand_flags);
        if (classification == 0) unique_class++;
        else if (classification == 1) canonical_class++;
        else if (classification == 2) redundant_class++;
        
        float ml_weight = MinimizerFlags::ml_weight_to_float(hit.ml_weight);
        ml_weights.push_back(ml_weight);
        uniqueness_sum += ml_weight;
    }
    
    // Print uniqueness statistics
    std::cout << "\n--- UNIQUENESS SCORES ---" << std::endl;
    const char* uniqueness_names[] = {
        "Extremely common (0-10%)",
        "Very low (10-30%)",
        "Low (30-50%)",
        "Moderate (50-70%)",
        "High (70-80%)",
        "Very high (80-90%)",
        "Extremely high (90-95%)",
        "Singleton-like (95-100%)"
    };
    for (int i = 0; i < 8; i++) {
        double percent = (hits.size() > 0) ? (100.0 * uniqueness_counts[i] / hits.size()) : 0.0;
        std::cout << "  " << uniqueness_names[i] << ": " << uniqueness_counts[i] 
                  << " (" << std::fixed << std::setprecision(1) << percent << "%)" << std::endl;
    }
    
    // Print quality metrics
    std::cout << "\n--- QUALITY METRICS ---" << std::endl;
    std::cout << "  Unique minimizers (≥90%): " << unique_minimizers << " (" 
              << (100.0 * unique_minimizers / hits.size()) << "%)" << std::endl;
    std::cout << "  Rare minimizers (≤3 occurrences): " << rare_minimizers << " (" 
              << (100.0 * rare_minimizers / hits.size()) << "%)" << std::endl;
    std::cout << "  Reliable minimizers: " << reliable_minimizers << " (" 
              << (100.0 * reliable_minimizers / hits.size()) << "%)" << std::endl;
    std::cout << "  Average uniqueness score: " << std::fixed << std::setprecision(3) 
              << (uniqueness_sum / hits.size()) << std::endl;
    
    // Print co-occurrence statistics
    std::cout << "\n--- CO-OCCURRENCE SCORES ---" << std::endl;
    const char* cooccurrence_names[] = {
        "No significant co-occurrence",
        "Very low co-occurrence",
        "Low co-occurrence",
        "Moderate co-occurrence",
        "High co-occurrence",
        "Very high co-occurrence",
        "Extremely high co-occurrence",
        "Perfect co-occurrence pattern"
    };
    for (int i = 0; i < 8; i++) {
        double percent = (hits.size() > 0) ? (100.0 * cooccurrence_counts[i] / hits.size()) : 0.0;
        std::cout << "  " << cooccurrence_names[i] << ": " << cooccurrence_counts[i] 
                  << " (" << std::fixed << std::setprecision(1) << percent << "%)" << std::endl;
    }
    
    // Print GC content distribution
    std::cout << "\n--- GC CONTENT DISTRIBUTION ---" << std::endl;
    const char* gc_names[] = {
        "0-20%", "20-30%", "30-40%", "40-50%",
        "50-60%", "60-70%", "70-80%", "80-100%"
    };
    for (int i = 0; i < 8; i++) {
        double percent = (hits.size() > 0) ? (100.0 * gc_counts[i] / hits.size()) : 0.0;
        std::cout << "  GC " << gc_names[i] << ": " << gc_counts[i] 
                  << " (" << std::fixed << std::setprecision(1) << percent << "%)" << std::endl;
    }
    
    // Print other statistics
    std::cout << "\n--- OTHER FEATURES ---" << std::endl;
    std::cout << "  Position clustering: " << position_clustered << " clustered, " 
              << position_uniform << " uniform" << std::endl;
    std::cout << "  Contamination risk: " << contamination_risk << std::endl;
    std::cout << "  Classification: " << unique_class << " unique, " 
              << canonical_class << " canonical, " << redundant_class << " redundant" << std::endl;
}

// Enhanced minimizer extraction kernel with all features
__global__ void extract_minimizers_with_all_features(
    const char* sequences,
    const uint32_t* seq_offsets,
    const uint32_t* seq_lengths,
    const uint32_t* genome_ids,
    const uint32_t* taxon_ids,
    int num_sequences,
    GPUMinimizerHit* minimizer_hits,
    uint32_t* global_counter,
    uint32_t max_minimizers,
    MinimizerParams params) {
    
    int seq_idx = blockIdx.x;
    if (seq_idx >= num_sequences) return;
    
    uint32_t seq_start = seq_offsets[seq_idx];
    uint32_t seq_length = seq_lengths[seq_idx];
    uint32_t genome_id = genome_ids[seq_idx];
    uint32_t taxon_id = taxon_ids[seq_idx];
    
    if (seq_length < params.k) return;
    
    const char* sequence = sequences + seq_start;
    
    // Shared memory for co-occurrence window
    extern __shared__ uint64_t shared_window[];
    
    // Each thread processes k-mers
    uint32_t tid = threadIdx.x;
    uint32_t threads_per_block = blockDim.x;
    uint32_t num_kmers = seq_length - params.k + 1;
    
    for (uint32_t pos = tid; pos < num_kmers; pos += threads_per_block) {
        // Extract minimizer
        uint64_t minimizer = extract_minimizer_sliding_window(
            sequence, pos, params.k, params.ell, params.spaces, 
            params.xor_mask, seq_length
        );
        
        if (minimizer != UINT64_MAX) {
            // Get slot for this minimizer
            uint32_t idx = atomicAdd(global_counter, 1);
            if (idx < max_minimizers) {
                GPUMinimizerHit& hit = minimizer_hits[idx];
                hit.minimizer_hash = minimizer;
                hit.genome_id = genome_id;
                hit.position = pos;
                hit.taxon_id = taxon_id;
                
                // Initialize feature flags
                uint32_t feature_flags = 0;
                
                // Calculate GC content
                int gc_count = 0;
                for (int i = 0; i < params.k; i++) {
                    char base = sequence[pos + i];
                    if (base == 'G' || base == 'g' || base == 'C' || base == 'c') {
                        gc_count++;
                    }
                }
                uint8_t gc_category = (gc_count * 8) / params.k;
                feature_flags |= (gc_category & 0x7);
                
                // Calculate complexity
                uint8_t base_mask = 0;
                for (int i = 0; i < params.k; i++) {
                    char base = sequence[pos + i];
                    if (base == 'A' || base == 'a') base_mask |= 1;
                    else if (base == 'C' || base == 'c') base_mask |= 2;
                    else if (base == 'G' || base == 'g') base_mask |= 4;
                    else if (base == 'T' || base == 't') base_mask |= 8;
                }
                uint8_t complexity = __popc(base_mask);
                feature_flags |= ((complexity & 0x7) << 3);
                
                // Position bias (simplified)
                bool is_clustered = (pos % 100) < 20;  // Dummy clustering
                if (is_clustered) {
                    feature_flags |= (1 << 6);
                }
                
                // Initialize uniqueness to unknown (will be computed later)
                feature_flags |= (3 << 8);  // Moderate uniqueness as default
                
                // Store features
                hit.feature_flags = feature_flags;
                hit.strand = 0;  // Can add strand info later
                hit.ml_weight = 32768;  // Default weight = 0.5
            }
        }
    }
}

// Process a single batch of genomes with two-pass feature extraction
void process_genome_batch_with_features(
    const std::vector<std::string>& batch_files,
    const std::vector<uint32_t>& batch_taxons,
    std::vector<GPUMinimizerHit>& all_minimizers,
    MinimizerFeatureExtractor& feature_extractor,
    const MinimizerParams& params,
    size_t max_batch_minimizers = 10000000) {
    
    // Load sequences from batch
    std::vector<std::string> sequences;
    std::vector<uint32_t> seq_lengths;
    std::vector<uint32_t> seq_offsets;
    std::vector<uint32_t> genome_ids;
    std::vector<uint32_t> taxon_ids;
    std::string concatenated;
    
    GenomeFileProcessor file_processor;
    
    for (size_t i = 0; i < batch_files.size(); i++) {
        auto seqs = file_processor.load_sequences_from_fasta(batch_files[i]);
        for (const auto& seq : seqs) {
            seq_offsets.push_back(concatenated.length());
            seq_lengths.push_back(seq.length());
            genome_ids.push_back(i);
            taxon_ids.push_back(batch_taxons[i]);
            concatenated += seq;
        }
    }
    
    if (concatenated.empty()) return;
    
    std::cout << "  Batch: " << seq_offsets.size() << " sequences, " 
              << concatenated.length() << " total bases" << std::endl;
    
    // Allocate GPU memory for this batch
    char* d_sequences;
    uint32_t* d_offsets;
    uint32_t* d_lengths;
    uint32_t* d_genome_ids;
    uint32_t* d_taxon_ids;
    GPUMinimizerHit* d_minimizers;
    uint32_t* d_counter;
    
    hipMalloc(&d_sequences, concatenated.length());
    hipMalloc(&d_offsets, seq_offsets.size() * sizeof(uint32_t));
    hipMalloc(&d_lengths, seq_lengths.size() * sizeof(uint32_t));
    hipMalloc(&d_genome_ids, genome_ids.size() * sizeof(uint32_t));
    hipMalloc(&d_taxon_ids, taxon_ids.size() * sizeof(uint32_t));
    hipMalloc(&d_minimizers, max_batch_minimizers * sizeof(GPUMinimizerHit));
    hipMalloc(&d_counter, sizeof(uint32_t));
    
    // Copy data to GPU
    hipMemcpy(d_sequences, concatenated.data(), concatenated.length(), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, seq_offsets.data(), seq_offsets.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_lengths, seq_lengths.data(), seq_lengths.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_genome_ids, genome_ids.data(), genome_ids.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_taxon_ids, taxon_ids.data(), taxon_ids.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemset(d_counter, 0, sizeof(uint32_t));
    
    // Launch kernel
    int threads = 256;
    int blocks = seq_offsets.size();
    size_t shared_mem = threads * sizeof(uint64_t);
    
    extract_minimizers_with_all_features<<<blocks, threads, shared_mem>>>(
        d_sequences, d_offsets, d_lengths, d_genome_ids, d_taxon_ids,
        seq_offsets.size(),
        d_minimizers, d_counter, max_batch_minimizers,
        params
    );
    
    hipDeviceSynchronize();
    
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;
    } else {
        // Get results
        uint32_t num_minimizers;
        hipMemcpy(&num_minimizers, d_counter, sizeof(uint32_t), hipMemcpyDeviceToHost);
        
        if (num_minimizers > 0) {
            std::vector<GPUMinimizerHit> batch_minimizers(num_minimizers);
            hipMemcpy(batch_minimizers.data(), d_minimizers, 
                       num_minimizers * sizeof(GPUMinimizerHit), hipMemcpyDeviceToHost);
            
            // First pass: collect statistics
            std::cout << "  Running first pass feature extraction..." << std::endl;
            feature_extractor.process_first_pass(d_minimizers, num_minimizers, batch_taxons);
            
            // Compute uniqueness scores
            std::cout << "  Computing uniqueness scores..." << std::endl;
            if (!compute_and_encode_uniqueness_scores(
                    d_minimizers, num_minimizers, batch_taxons, batch_taxons.size())) {
                std::cerr << "  Warning: Uniqueness computation failed" << std::endl;
            }
            
            // Compute co-occurrence scores
            std::cout << "  Computing co-occurrence patterns..." << std::endl;
            
            // Get unique minimizers for co-occurrence
            std::unordered_map<uint64_t, uint32_t> minimizer_counts;
            for (const auto& hit : batch_minimizers) {
                minimizer_counts[hit.minimizer_hash]++;
            }
            
            std::vector<uint64_t> unique_minimizers;
            std::vector<uint32_t> occurrence_counts;
            for (const auto& [hash, count] : minimizer_counts) {
                unique_minimizers.push_back(hash);
                occurrence_counts.push_back(count);
            }
            
            // Create dummy genome info for co-occurrence calculation
            std::vector<GPUGenomeInfo> genome_info;
            for (size_t i = 0; i < batch_files.size(); i++) {
                GPUGenomeInfo info;
                info.genome_id = i;
                info.taxon_id = batch_taxons[i];
                info.sequence_length = 100000;  // Dummy value
                info.sequence_offset = 0;
                genome_info.push_back(info);
            }
            
            if (!compute_and_encode_cooccurrence_scores(
                    d_minimizers, num_minimizers, unique_minimizers, 
                    occurrence_counts, genome_info)) {
                std::cerr << "  Warning: Co-occurrence computation failed" << std::endl;
            }
            
            // Second pass: complete feature extraction
            std::cout << "  Running second pass feature extraction..." << std::endl;
            feature_extractor.process_second_pass(d_minimizers, num_minimizers);
            
            // Copy final results back
            hipMemcpy(batch_minimizers.data(), d_minimizers, 
                       num_minimizers * sizeof(GPUMinimizerHit), hipMemcpyDeviceToHost);
            
            // Add to overall results
            all_minimizers.insert(all_minimizers.end(), 
                                  batch_minimizers.begin(), 
                                  batch_minimizers.end());
            
            std::cout << "  Extracted " << num_minimizers << " minimizers with all features" << std::endl;
        }
    }
    
    // Cleanup GPU memory
    hipFree(d_sequences);
    hipFree(d_offsets);
    hipFree(d_lengths);
    hipFree(d_genome_ids);
    hipFree(d_taxon_ids);
    hipFree(d_minimizers);
    hipFree(d_counter);
}

int main(int argc, char** argv) {
    std::cout << "=== ALL FEATURES TEST WITH STREAMING ===" << std::endl;
    
    // Configuration
    std::string fna_file = "/home/david/Documents/Code/biogpu/data/test_50_genomes.fna";
    std::string temp_dir = "/tmp/biogpu_features_test";
    size_t batch_size = 5;  // Process 5 genomes at a time
    
    // Check if file exists
    if (!std::filesystem::exists(fna_file)) {
        std::cerr << "Error: FNA file not found: " << fna_file << std::endl;
        return 1;
    }
    
    // Create temp directory
    std::filesystem::create_directories(temp_dir);
    
    // Check GPU
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        std::cerr << "No CUDA devices!" << std::endl;
        return 1;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Total memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
    
    // Set minimizer parameters
    MinimizerParams params;
    params.k = 31;
    params.ell = 31;
    params.spaces = 7;
    params.xor_mask = 0x3c8bfbb395c60474ULL;
    
    std::cout << "\nMinimizer parameters:" << std::endl;
    std::cout << "  k-mer size: " << params.k << std::endl;
    std::cout << "  l-mer size: " << params.ell << std::endl;
    std::cout << "  Batch size: " << batch_size << " genomes" << std::endl;
    
    // Initialize streaming processor
    std::cout << "\nInitializing streaming processor..." << std::endl;
    StreamingFnaProcessor processor(fna_file, temp_dir, batch_size);
    
    // Initialize feature extractor
    MinimizerFeatureExtractor feature_extractor(100000000, 1000);  // Max 100M minimizers, 1000 genomes
    
    // Process all batches
    std::vector<GPUMinimizerHit> all_minimizers;
    all_minimizers.reserve(75000000);  // Reserve space for ~75M minimizers
    
    int batch_num = 0;
    auto start_time = std::chrono::high_resolution_clock::now();
    
    std::vector<std::string> batch_files;
    std::vector<uint32_t> batch_taxons;
    
    while (processor.process_next_batch(batch_files, batch_taxons)) {
        batch_num++;
        std::cout << "\nProcessing batch " << batch_num << " (" 
                  << batch_files.size() << " genomes)..." << std::endl;
        
        process_genome_batch_with_features(batch_files, batch_taxons, 
                                          all_minimizers, feature_extractor, params);
        
        // Clean up temp files
        for (const auto& file : batch_files) {
            std::filesystem::remove(file);
        }
        
        batch_files.clear();
        batch_taxons.clear();
    }
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::seconds>(end_time - start_time);
    
    std::cout << "\n=== PROCESSING COMPLETE ===" << std::endl;
    std::cout << "Total batches: " << batch_num << std::endl;
    std::cout << "Total genomes: " << processor.get_total_genomes() << std::endl;
    std::cout << "Total minimizers: " << all_minimizers.size() << std::endl;
    std::cout << "Processing time: " << duration.count() << " seconds" << std::endl;
    
    // Analyze features
    if (!all_minimizers.empty()) {
        analyze_minimizer_features(all_minimizers);
        
        // Show some examples
        std::cout << "\n=== EXAMPLE MINIMIZERS WITH ALL FEATURES ===" << std::endl;
        int examples = std::min(10, (int)all_minimizers.size());
        for (int i = 0; i < examples; i++) {
            const auto& hit = all_minimizers[i];
            std::cout << "\nMinimizer " << i << ":" << std::endl;
            std::cout << "  Hash: 0x" << std::hex << hit.minimizer_hash << std::dec << std::endl;
            std::cout << "  Genome ID: " << hit.genome_id << std::endl;
            std::cout << "  Position: " << hit.position << std::endl;
            std::cout << "  Taxon ID: " << hit.taxon_id << std::endl;
            
            uint32_t flags = hit.feature_flags;
            std::cout << "  Features:" << std::endl;
            std::cout << "    - GC content category: " << (flags & 0x7) << "/7" << std::endl;
            std::cout << "    - Complexity: " << ((flags >> 3) & 0x7) << "/7" << std::endl;
            std::cout << "    - Position bias: " << ((flags >> 6) & 0x1 ? "Clustered" : "Uniform") << std::endl;
            
            // Uniqueness features
            uint8_t uniqueness_cat = MinimizerFlags::get_uniqueness_category_safe(flags);
            std::cout << "    - Uniqueness category: " << (int)uniqueness_cat << "/7 (" 
                      << MinimizerFlags::uniqueness_category_name(uniqueness_cat) << ")" << std::endl;
            std::cout << "    - Unique minimizer: " << (MinimizerFlags::is_unique_minimizer_safe(flags) ? "Yes" : "No") << std::endl;
            std::cout << "    - Rare minimizer: " << (MinimizerFlags::is_rare_minimizer_safe(flags) ? "Yes" : "No") << std::endl;
            std::cout << "    - Reliable: " << (MinimizerFlags::is_reliable_minimizer_safe(flags) ? "Yes" : "No") << std::endl;
            
            // Co-occurrence score
            uint8_t cooccurrence = MinimizerFlags::get_cooccurrence_score(flags);
            std::cout << "    - Co-occurrence score: " << (int)cooccurrence << "/7" << std::endl;
            
            // ML weight (uniqueness score)
            float uniqueness_score = MinimizerFlags::ml_weight_to_float(hit.ml_weight);
            std::cout << "    - Uniqueness score: " << std::fixed << std::setprecision(3) 
                      << uniqueness_score << std::endl;
        }
        
        // Print feature extraction statistics
        auto stats = feature_extractor.get_statistics();
        std::cout << "\n=== FEATURE EXTRACTION STATISTICS ===" << std::endl;
        std::cout << "Total unique minimizers: " << stats.total_unique_minimizers << std::endl;
        std::cout << "Singleton minimizers: " << stats.singleton_minimizers << std::endl;
        std::cout << "Rare minimizers (<10): " << stats.rare_minimizers << std::endl;
        std::cout << "Common minimizers (>50%): " << stats.common_minimizers << std::endl;
        std::cout << "Average taxonomic spread: " << std::fixed << std::setprecision(2) 
                  << stats.avg_taxonomic_spread << std::endl;
    }
    
    // Cleanup
    std::filesystem::remove_all(temp_dir);
    
    std::cout << "\n=== TEST " << (all_minimizers.size() > 0 ? "PASSED" : "FAILED") << " ===" << std::endl;
    
    return all_minimizers.size() > 0 ? 0 : 1;
}

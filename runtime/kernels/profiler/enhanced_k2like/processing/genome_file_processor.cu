#include "hip/hip_runtime.h"
// processing/genome_file_processor.cu
// Implementation of genome file processing functionality
// Extracted from monolithic database builder for better modularity

#include "genome_file_processor.h"
#include "../gpu_kraken_types.h"
#include <iostream>
#include <fstream>
#include <filesystem>
#include <chrono>
#include <algorithm>
#include <regex>
#include <sstream>
#include <cctype>
#include <sys/stat.h>
#include <cstdio>
#include <cstring>
#include <climits>
#include <unistd.h>
#include <set>
#include <iomanip>

// ===========================
// GenomeFileProcessor Implementation
// ===========================

GenomeFileProcessor::GenomeFileProcessor(const FileProcessingConfig& config) 
    : config_(config) {
    reset_statistics();
}

std::vector<std::string> GenomeFileProcessor::find_genome_files(const std::string& directory) {
    std::vector<std::string> files;
    
    std::cout << "Finding genome files in: " << directory << std::endl;
    auto start_time = std::chrono::high_resolution_clock::now();
    
    if (!validate_path_safety(directory)) {
        std::cerr << "Error: Invalid directory path provided" << std::endl;
        return files;
    }
    
    try {
        // Use filesystem::recursive_directory_iterator instead of popen
        for (const auto& entry : std::filesystem::recursive_directory_iterator(directory)) {
            if (entry.is_regular_file()) {
                std::string file_path = entry.path().string();
                
                if (validate_genome_file(file_path)) {
                    files.emplace_back(file_path);
                    stats_.files_found++;
                    
                    if (config_.progress_reporting && stats_.files_found % config_.progress_interval == 0) {
                        std::cout << "Found " << stats_.files_found << " genome files..." << std::endl;
                    }
                }
                
                if (files.size() >= config_.max_file_count) {
                    std::cout << "Reached file limit of " << config_.max_file_count << std::endl;
                    break;
                }
            }
        }
    } catch (const std::filesystem::filesystem_error& e) {
        std::cerr << "Filesystem error: " << e.what() << std::endl;
        return files;
    }
    
    std::sort(files.begin(), files.end());
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration<double>(end_time - start_time).count();
    stats_.processing_time += duration;
    
    std::cout << "Found " << files.size() << " genome files in " << duration << " seconds" << std::endl;
    
    return files;
}

bool GenomeFileProcessor::validate_genome_file(const std::string& file_path) {
    // Basic path validation
    if (!validate_path_safety(file_path)) {
        return false;
    }
    
    // Check if file exists and is readable
    struct stat file_stat;
    if (stat(file_path.c_str(), &file_stat) != 0) {
        return false;
    }
    
    // Check file size
    if (file_stat.st_size > config_.max_file_size) {
        if (config_.progress_reporting) {
            std::cerr << "Warning: File too large, skipping: " << file_path << std::endl;
        }
        return false;
    }
    
    // Check file extension
    std::filesystem::path p(file_path);
    if (!is_valid_genome_file_extension(p.extension().string())) {
        return false;
    }
    
    return true;
}

std::vector<std::string> GenomeFileProcessor::load_sequences_from_fasta(const std::string& fasta_path) {
    std::vector<std::string> sequences;
    
    auto start_time = std::chrono::high_resolution_clock::now();
    
    // Validate input
    if (!validate_genome_file(fasta_path)) {
        std::cerr << "Invalid FASTA file: " << fasta_path << std::endl;
        return sequences;
    }
    
    std::ifstream file(fasta_path);
    if (!file.is_open()) {
        std::cerr << "Cannot open FASTA file: " << fasta_path << std::endl;
        stats_.processing_errors++;
        return sequences;
    }
    
    try {
        std::string line, current_sequence;
        bool in_sequence = false;
        int line_number = 0;
        size_t file_sequences = 0;
        size_t file_bases = 0;
        
        current_sequence.reserve(50000); // Reserve space for typical sequence
        
        while (std::getline(file, line)) {
            line_number++;
            
            if (line.empty()) continue;
            
            if (line[0] == '>') {
                // Process previous sequence
                if (in_sequence && !current_sequence.empty()) {
                    if (current_sequence.size() >= 50 && // Minimum reasonable size
                        current_sequence.size() <= config_.max_sequence_length) {
                        
                        if (!config_.validate_sequences || validate_sequence_content(current_sequence)) {
                            sequences.push_back(current_sequence);
                            file_sequences++;
                            file_bases += current_sequence.size();
                        }
                    }
                    current_sequence.clear();
                    current_sequence.reserve(50000);
                }
                in_sequence = true;
            } else if (in_sequence) {
                // Validate line length
                if (line.size() > 100000) {
                    std::cerr << "Warning: Very long line in " << fasta_path 
                              << " at line " << line_number << std::endl;
                    continue;
                }
                
                // Add line to sequence, removing whitespace
                for (char c : line) {
                    if (!std::isspace(c)) {
                        current_sequence += c;
                    }
                }
                
                // Check sequence length limit
                if (current_sequence.size() > config_.max_sequence_length) {
                    std::cerr << "Warning: Sequence exceeds size limit in " << fasta_path << std::endl;
                    current_sequence.clear();
                    in_sequence = false;
                }
            }
            
            // Progress for very large files
            if (line_number % 100000 == 0 && config_.progress_reporting) {
                std::cout << "Processed " << line_number << " lines from " << fasta_path << std::endl;
            }
        }
        
        // Handle the last sequence
        if (in_sequence && !current_sequence.empty()) {
            if (current_sequence.size() >= 50 && 
                current_sequence.size() <= config_.max_sequence_length) {
                
                if (!config_.validate_sequences || validate_sequence_content(current_sequence)) {
                    sequences.push_back(current_sequence);
                    file_sequences++;
                    file_bases += current_sequence.size();
                }
            }
        }
        
        // Update statistics
        stats_.files_processed++;
        stats_.total_sequences += file_sequences;
        stats_.total_bases += file_bases;
        
        auto end_time = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration<double>(end_time - start_time).count();
        stats_.processing_time += duration;
        
        if (config_.progress_reporting) {
            std::cout << "Loaded " << sequences.size() << " sequences (" 
                      << (file_bases / 1024 / 1024) << " MB) from " << fasta_path << std::endl;
        }
        
    } catch (const std::exception& e) {
        std::cerr << "Error reading FASTA file " << fasta_path << ": " << e.what() << std::endl;
        stats_.processing_errors++;
        sequences.clear();
    }
    
    file.close();
    return sequences;
}

uint32_t GenomeFileProcessor::extract_taxon_from_filename(const std::string& filename) {
    if (filename.empty()) {
        return 1000000; // Default fallback
    }
    
    try {
        std::filesystem::path p(filename);
        std::string stem = p.stem().string();
        
        if (stem.empty()) {
            return 1000000;
        }
        
        // Try to extract taxid pattern
        std::regex taxid_pattern(R"(taxid[_-](\d+))");
        std::smatch match;
        if (std::regex_search(stem, match, taxid_pattern)) {
            try {
                uint32_t taxon_id = std::stoul(match[1].str());
                if (taxon_id > 0 && taxon_id < UINT32_MAX) {
                    return taxon_id;
                }
            } catch (const std::exception&) {
                // Fall through to other patterns
            }
        }
        
        // Try GCF pattern
        std::regex gcf_pattern(R"(GCF_(\d+\.\d+))");
        if (std::regex_search(stem, match, gcf_pattern)) {
            try {
                std::hash<std::string> hasher;
                uint32_t hash_val = hasher(match[1].str()) % 1000000 + 1000000;
                return hash_val;
            } catch (const std::exception&) {
                // Fall through to default
            }
        }
        
        // Fallback: hash the stem
        std::hash<std::string> hasher;
        uint32_t hash_val = hasher(stem) % 1000000 + 2000000;
        return hash_val;
        
    } catch (const std::exception& e) {
        std::cerr << "Error processing filename " << filename << ": " << e.what() << std::endl;
        return 1000000;
    }
}

bool GenomeFileProcessor::process_genome_files_batch(
    const std::vector<std::string>& file_paths,
    std::vector<std::string>& all_sequences,
    std::vector<uint32_t>& sequence_taxon_ids) {
    
    std::cout << "Processing batch of " << file_paths.size() << " genome files..." << std::endl;
    
    for (const auto& file_path : file_paths) {
        uint32_t taxon_id = extract_taxon_from_filename(file_path);
        std::vector<std::string> file_sequences = load_sequences_from_fasta(file_path);
        
        // Add sequences with their taxon IDs
        for (const auto& sequence : file_sequences) {
            all_sequences.push_back(sequence);
            sequence_taxon_ids.push_back(taxon_id);
        }
        
        if (file_sequences.empty()) {
            stats_.files_skipped++;
        }
    }
    
    return true;
}

std::vector<std::string> GenomeFileProcessor::load_file_list(const std::string& file_list_path) {
    std::vector<std::string> files;
    
    std::ifstream list_file(file_list_path);
    if (!list_file.is_open()) {
        std::cerr << "Cannot open file list: " << file_list_path << std::endl;
        return files;
    }
    
    std::string file_path;
    while (std::getline(list_file, file_path)) {
        // Trim whitespace
        file_path.erase(0, file_path.find_first_not_of(" \t\r\n"));
        file_path.erase(file_path.find_last_not_of(" \t\r\n") + 1);
        
        if (!file_path.empty() && validate_genome_file(file_path)) {
            files.push_back(file_path);
        }
    }
    
    list_file.close();
    return files;
}

bool GenomeFileProcessor::count_sequences_in_file(const std::string& file_path, size_t& sequence_count, size_t& total_bases) {
    sequence_count = 0;
    total_bases = 0;
    
    std::ifstream file(file_path);
    if (!file.is_open()) {
        return false;
    }
    
    std::string line;
    bool in_sequence = false;
    
    while (std::getline(file, line)) {
        if (line.empty()) continue;
        
        if (line[0] == '>') {
            if (in_sequence) {
                sequence_count++;
            }
            in_sequence = true;
        } else if (in_sequence) {
            for (char c : line) {
                if (!std::isspace(c)) {
                    total_bases++;
                }
            }
        }
    }
    
    if (in_sequence) {
        sequence_count++;
    }
    
    file.close();
    return true;
}

std::string GenomeFileProcessor::extract_species_name_from_file(const std::string& file_path) {
    std::ifstream file(file_path);
    if (!file.is_open()) {
        return "";
    }
    
    std::string line;
    if (std::getline(file, line) && line[0] == '>') {
        // Extract species name from header
        std::istringstream iss(line.substr(1));
        std::string genus, species;
        if (iss >> genus >> species) {
            return genus + " " + species;
        }
    }
    
    file.close();
    return "";
}

void GenomeFileProcessor::print_processing_summary() const {
    std::cout << "\n=== FILE PROCESSING SUMMARY ===" << std::endl;
    std::cout << "Files found: " << stats_.files_found << std::endl;
    std::cout << "Files processed: " << stats_.files_processed << std::endl;
    std::cout << "Files skipped: " << stats_.files_skipped << std::endl;
    std::cout << "Total sequences: " << stats_.total_sequences << std::endl;
    std::cout << "Total bases: " << (stats_.total_bases / 1024 / 1024) << " MB" << std::endl;
    std::cout << "Processing errors: " << stats_.processing_errors << std::endl;
    std::cout << "Processing time: " << std::fixed << std::setprecision(2) 
              << stats_.processing_time << " seconds" << std::endl;
    
    if (stats_.processing_time > 0) {
        double rate = stats_.total_bases / stats_.processing_time / 1024 / 1024;
        std::cout << "Processing rate: " << std::fixed << std::setprecision(2) 
                  << rate << " MB/s" << std::endl;
    }
}

// Private helper methods

void GenomeFileProcessor::reset_statistics() {
    stats_ = FileProcessingStats();
}

// ===========================
// ConcatenatedFnaProcessor Implementation  
// ===========================

ConcatenatedFnaProcessor::ConcatenatedFnaProcessor(const std::string& file_path, 
                                                 const FileProcessingConfig& config)
    : fna_file_path_(file_path), bytes_processed_(0), total_file_size_(0), config_(config) {
    
    // Get file size for progress reporting
    std::ifstream file(file_path, std::ios::ate | std::ios::binary);
    if (file.is_open()) {
        total_file_size_ = file.tellg();
        file.close();
    }
}

bool ConcatenatedFnaProcessor::process_fna_file(
    std::vector<std::string>& genome_files, 
    std::vector<uint32_t>& genome_taxon_ids,
    std::unordered_map<uint32_t, std::string>& taxon_names,
    const std::string& temp_dir) {
    
    std::cout << "Processing concatenated FNA file: " << fna_file_path_ << std::endl;
    std::cout << "File size: " << (total_file_size_ / 1024 / 1024) << " MB" << std::endl;
    
    // Create temporary directory
    try {
        std::filesystem::create_directories(temp_dir);
    } catch (const std::exception& e) {
        std::cerr << "Failed to create temp directory: " << e.what() << std::endl;
        return false;
    }
    
    std::ifstream file(fna_file_path_);
    if (!file.is_open()) {
        std::cerr << "Cannot open FNA file: " << fna_file_path_ << std::endl;
        return false;
    }
    
    std::string line;
    std::string current_sequence;
    std::string current_header;
    uint32_t current_species = 0;
    int genome_count = 0;
    
    auto start_time = std::chrono::high_resolution_clock::now();
    
    while (std::getline(file, line)) {
        bytes_processed_ += line.length() + 1;
        
        if (line.empty()) continue;
        
        if (line[0] == '>') {
            // Process previous genome
            if (!current_sequence.empty() && current_species > 0) {
                std::string temp_file = create_temp_genome_file(
                    current_sequence, current_species, current_header, 
                    temp_dir, genome_count
                );
                
                if (!temp_file.empty()) {
                    genome_files.push_back(temp_file);
                    genome_taxon_ids.push_back(current_species);
                    genome_count++;
                    
                    // Progress reporting
                    if (config_.progress_reporting && genome_count % config_.progress_interval == 0) {
                        double progress = (double)bytes_processed_ / total_file_size_ * 100.0;
                        std::cout << "Processed " << genome_count << " genomes (" 
                                 << std::fixed << std::setprecision(1) << progress << "%)" << std::endl;
                    }
                }
            }
            
            // Parse new header
            current_header = line;
            HeaderParseResult parse_result = parse_fna_header(line);
            current_species = parse_result.species_taxid;
            current_sequence.clear();
            current_sequence.reserve(1000000); // Reserve 1MB
            
            if (current_species > 0) {
                // Store species information
                species_data_.add_genome("genome_" + std::to_string(genome_count), 
                                       current_species, parse_result.species_name);
                
                // Add to taxon names
                if (taxon_names.find(current_species) == taxon_names.end()) {
                    taxon_names[current_species] = parse_result.species_name;
                }
                
                // Debug output for tracking
                if (config_.progress_reporting) {
                    std::cout << "Found genome: taxon=" << current_species 
                              << ", species=" << parse_result.species_name 
                              << ", accession=" << parse_result.accession << std::endl;
                }
            } else {
                std::cerr << "Warning: Could not parse taxon ID from header: " << line << std::endl;
            }
            
        } else {
            // Accumulate sequence data, removing all whitespace including newlines and carriage returns
            for (char c : line) {
                // Only add nucleotide characters, skip all whitespace
                if (!std::isspace(c) && c != '\r' && c != '\n') {
                    current_sequence += c;
                }
            }
            
            // Check size limit
            if (current_sequence.size() > config_.max_sequence_length) {
                std::cerr << "Warning: Sequence too large for taxon " << current_species 
                          << ", truncating at " << config_.max_sequence_length << " bases" << std::endl;
                current_sequence.clear();
                current_species = 0;
            }
        }
    }
    
    // Process last genome
    if (!current_sequence.empty() && current_species > 0) {
        std::string temp_file = create_temp_genome_file(
            current_sequence, current_species, current_header, 
            temp_dir, genome_count
        );
        
        if (!temp_file.empty()) {
            genome_files.push_back(temp_file);
            genome_taxon_ids.push_back(current_species);
            genome_count++;
        }
    }
    
    file.close();
    
    auto end_time = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::seconds>(end_time - start_time);
    
    std::cout << "\nFNA processing completed:" << std::endl;
    std::cout << "  Genomes processed: " << genome_count << std::endl;
    std::cout << "  Species represented: " << species_data_.total_species() << std::endl;
    std::cout << "  Processing time: " << duration.count() << " seconds" << std::endl;
    
    if (duration.count() > 0) {
        std::cout << "  Processing rate: " << (bytes_processed_ / 1024 / 1024 / duration.count()) 
                  << " MB/s" << std::endl;
    }
    
    return genome_count > 0;
}

// Header parsing implementation
ConcatenatedFnaProcessor::HeaderParseResult ConcatenatedFnaProcessor::parse_fna_header(const std::string& header) {
    HeaderParseResult result;
    
    // Parse header format: >kraken:taxid|XXXXX|ACCESSION description
    size_t taxid_start = header.find("taxid|");
    if (taxid_start == std::string::npos) {
        return result;
    }
    
    size_t taxid_value_start = taxid_start + 6;
    size_t taxid_end = header.find('|', taxid_value_start);
    if (taxid_end == std::string::npos) {
        return result;
    }
    
    try {
        std::string taxid_str = header.substr(taxid_value_start, taxid_end - taxid_value_start);
        result.species_taxid = std::stoul(taxid_str);
    } catch (const std::exception&) {
        return result;
    }
    
    // Extract accession and description
    size_t accession_start = taxid_end + 1;
    size_t description_start = header.find(' ', accession_start);
    
    if (description_start != std::string::npos) {
        result.accession = header.substr(accession_start, description_start - accession_start);
        result.description = header.substr(description_start + 1);
        result.species_name = extract_species_name_from_description(result.description);
    } else {
        result.accession = header.substr(accession_start);
    }
    
    if (result.species_name.empty()) {
        result.species_name = "species_" + std::to_string(result.species_taxid);
    }
    
    return result;
}

std::string ConcatenatedFnaProcessor::extract_species_name_from_description(const std::string& description) {
    std::istringstream iss(description);
    std::string genus, species;
    
    if (iss >> genus >> species) {
        if (!genus.empty() && !species.empty() && 
            std::isupper(genus[0]) && std::islower(species[0])) {
            return genus + " " + species;
        }
    }
    
    return "";
}

std::string ConcatenatedFnaProcessor::create_temp_genome_file(
    const std::string& sequence, 
    uint32_t species_taxid,
    const std::string& original_header,
    const std::string& temp_dir,
    int genome_index) {
    
    // Skip very short sequences
    if (sequence.length() < 1000) {
        return "";
    }
    
    std::string filename = temp_dir + "/genome_" + std::to_string(species_taxid) + 
                          "_" + std::to_string(genome_index) + ".fasta";
    
    std::ofstream outfile(filename);
    if (!outfile.is_open()) {
        return "";
    }
    
    // Write header with taxon information preserved
    // If we have the original header, use it; otherwise create a simple one
    if (!original_header.empty()) {
        outfile << original_header << "\n";
    } else {
        outfile << ">kraken:taxid|" << species_taxid << "|genome_" << genome_index << "\n";
    }
    
    // Write sequence in 80-character lines
    const size_t line_length = 80;
    for (size_t i = 0; i < sequence.length(); i += line_length) {
        size_t end = std::min(i + line_length, sequence.length());
        outfile << sequence.substr(i, end - i) << "\n";
    }
    
    outfile.close();
    return filename;
}

double ConcatenatedFnaProcessor::get_progress_percentage() const {
    if (total_file_size_ == 0) return 0.0;
    return (double)bytes_processed_ / total_file_size_ * 100.0;
}

// ===========================
// Utility Functions Implementation
// ===========================

// ===========================
// Private Method Implementations
// ===========================

bool GenomeFileProcessor::validate_path_safety(const std::string& path) {
    // Check for dangerous characters and patterns
    if (path.empty() || path.length() > PATH_MAX) {
        return false;
    }
    
    // Check for command injection attempts
    const std::string dangerous_chars = ";|&`$<>\\";
    if (path.find_first_of(dangerous_chars) != std::string::npos) {
        return false;
    }
    
    // Check for directory traversal
    if (path.find("..") != std::string::npos) {
        return false;
    }
    
    // Verify path exists and is a directory
    struct stat path_stat;
    if (stat(path.c_str(), &path_stat) != 0) {
        return false;
    }
    
    return S_ISDIR(path_stat.st_mode);
}

bool GenomeFileProcessor::is_valid_genome_file_extension(const std::string& extension) {
    static const std::set<std::string> valid_extensions = {
        ".fna", ".fa", ".fasta", ".ffn", ".faa", ".fsa"
    };
    
    std::string lower_ext = extension;
    std::transform(lower_ext.begin(), lower_ext.end(), lower_ext.begin(), ::tolower);
    
    return valid_extensions.find(lower_ext) != valid_extensions.end();
}

bool GenomeFileProcessor::validate_fasta_format(const std::string& file_path) {
    std::ifstream file(file_path);
    if (!file.is_open()) {
        return false;
    }
    
    std::string line;
    bool has_header = false;
    bool has_sequence = false;
    
    // Read first few lines to validate format
    for (int i = 0; i < 10 && std::getline(file, line); i++) {
        if (line.empty()) continue;
        
        if (line[0] == '>') {
            has_header = true;
        } else if (has_header) {
            // Check if line contains valid sequence characters
            for (char c : line) {
                if (!std::isalpha(c) && !std::isspace(c)) {
                    return false;
                }
            }
            has_sequence = true;
        }
    }
    
    return has_header && has_sequence;
}

bool GenomeFileProcessor::validate_sequence_content(const std::string& sequence) {
    if (sequence.empty() || sequence.length() < 100) {
        return false;  // Too short to be meaningful
    }
    
    size_t valid_bases = 0;
    size_t total_bases = 0;
    
    for (char c : sequence) {
        if (std::isalpha(c)) {
            total_bases++;
            char upper_c = std::toupper(c);
            if (upper_c == 'A' || upper_c == 'C' || upper_c == 'G' || upper_c == 'T') {
                valid_bases++;
            }
        }
    }
    
    // Require at least 90% valid DNA bases
    return total_bases > 0 && (double)valid_bases / total_bases >= 0.9;
}

bool GenomeFileProcessor::process_single_fasta_file(
    const std::string& file_path,
    std::vector<std::string>& sequences,
    std::vector<uint32_t>& taxon_ids,
    uint32_t default_taxon_id) {
    
    std::ifstream file(file_path);
    if (!file.is_open()) {
        return false;
    }
    
    std::string line;
    std::string current_sequence;
    bool in_sequence = false;
    
    while (std::getline(file, line)) {
        if (line.empty()) continue;
        
        if (line[0] == '>') {
            // Save previous sequence if exists
            if (!current_sequence.empty() && validate_sequence_content(current_sequence)) {
                sequences.push_back(current_sequence);
                taxon_ids.push_back(default_taxon_id);
                stats_.total_sequences++;
                stats_.total_bases += current_sequence.length();
            }
            
            // Start new sequence
            current_sequence.clear();
            in_sequence = true;
        } else if (in_sequence) {
            // Remove whitespace and append to sequence
            line.erase(std::remove_if(line.begin(), line.end(), ::isspace), line.end());
            current_sequence += line;
        }
    }
    
    // Save last sequence
    if (!current_sequence.empty() && validate_sequence_content(current_sequence)) {
        sequences.push_back(current_sequence);
        taxon_ids.push_back(default_taxon_id);
        stats_.total_sequences++;
        stats_.total_bases += current_sequence.length();
    }
    
    return true;
}

void GenomeFileProcessor::update_processing_stats(const std::string& file_path, 
                                                 size_t sequences_added, 
                                                 size_t bases_added) {
    stats_.files_processed++;
    stats_.total_sequences += sequences_added;
    stats_.total_bases += bases_added;
    
    if (config_.progress_reporting && stats_.files_processed % config_.progress_interval == 0) {
        print_processing_summary();
    }
}

// ===========================
// StreamingFnaProcessor Implementation
// ===========================

StreamingFnaProcessor::StreamingFnaProcessor(const std::string& fna_path, 
                                           const std::string& temp_dir,
                                           size_t batch_size)
    : fna_file_path_(fna_path), temp_directory_(temp_dir), batch_size_(batch_size),
      current_genome_count_(0), total_bases_processed_(0), processing_active_(false),
      end_of_file_reached_(false) {
    
    // Create temp directory if needed
    FileProcessingUtils::create_safe_directory(temp_directory_);
}

StreamingFnaProcessor::~StreamingFnaProcessor() {
    if (file_stream_.is_open()) {
        file_stream_.close();
    }
    cleanup_temp_files();
}

bool StreamingFnaProcessor::process_next_batch(std::vector<std::string>& batch_files, 
                                              std::vector<uint32_t>& batch_taxons) {
    batch_files.clear();
    batch_taxons.clear();
    
    if (!processing_active_) {
        file_stream_.open(fna_file_path_);
        if (!file_stream_.is_open()) {
            return false;
        }
        processing_active_ = true;
    }
    
    return process_batch_from_buffer(batch_files, batch_taxons);
}

bool StreamingFnaProcessor::has_more_data() const {
    return processing_active_ && !end_of_file_reached_;
}

void StreamingFnaProcessor::reset_processing() {
    if (file_stream_.is_open()) {
        file_stream_.close();
    }
    processing_active_ = false;
    end_of_file_reached_ = false;
    current_genome_count_ = 0;
    total_bases_processed_ = 0;
    cleanup_temp_files();
}

bool StreamingFnaProcessor::process_batch_from_buffer(std::vector<std::string>& batch_files, 
                                                     std::vector<uint32_t>& batch_taxons) {
    // Implementation simplified for demonstration
    // In production, this would properly parse and batch sequences
    
    std::string line;
    std::string current_sequence;
    std::string current_header;
    size_t genomes_in_batch = 0;
    
    while (genomes_in_batch < batch_size_ && std::getline(file_stream_, line)) {
        if (line.empty()) continue;
        
        if (line[0] == '>') {
            // Process previous sequence if exists
            if (!current_sequence.empty()) {
                // Extract taxon ID from header (simplified)
                uint32_t taxon_id = 1;  // Default taxon
                
                // Create temp file
                std::string temp_file = temp_directory_ + "/batch_genome_" + 
                                       std::to_string(current_genome_count_) + ".fasta";
                
                std::ofstream out(temp_file);
                if (out.is_open()) {
                    out << current_header << "\n" << current_sequence << "\n";
                    out.close();
                    
                    batch_files.push_back(temp_file);
                    batch_taxons.push_back(taxon_id);
                    genomes_in_batch++;
                    current_genome_count_++;
                    total_bases_processed_ += current_sequence.length();
                }
            }
            
            // Start new sequence
            current_header = line;
            current_sequence.clear();
        } else {
            // Append to current sequence
            line.erase(std::remove_if(line.begin(), line.end(), ::isspace), line.end());
            current_sequence += line;
        }
    }
    
    // Check if we've reached end of file
    if (!file_stream_.good()) {
        end_of_file_reached_ = true;
        
        // Process last sequence if exists
        if (!current_sequence.empty()) {
            uint32_t taxon_id = 1;  // Default taxon
            
            std::string temp_file = temp_directory_ + "/batch_genome_" + 
                                   std::to_string(current_genome_count_) + ".fasta";
            
            std::ofstream out(temp_file);
            if (out.is_open()) {
                out << current_header << "\n" << current_sequence << "\n";
                out.close();
                
                batch_files.push_back(temp_file);
                batch_taxons.push_back(taxon_id);
                current_genome_count_++;
                total_bases_processed_ += current_sequence.length();
            }
        }
    }
    
    return !batch_files.empty();
}

bool StreamingFnaProcessor::read_next_chunk_to_buffer() {
    // Implementation would read a chunk of data into buffer
    // For now, using line-by-line processing
    return file_stream_.good();
}

void StreamingFnaProcessor::cleanup_temp_files() {
    try {
        for (const auto& entry : std::filesystem::directory_iterator(temp_directory_)) {
            if (entry.path().filename().string().find("batch_genome_") == 0) {
                std::filesystem::remove(entry.path());
            }
        }
    } catch (const std::exception&) {
        // Ignore cleanup errors
    }
}

namespace FileProcessingUtils {
    
    bool is_fasta_file(const std::string& file_path) {
        std::filesystem::path p(file_path);
        std::string ext = p.extension().string();
        return (ext == ".fna" || ext == ".fa" || ext == ".fasta" || 
                ext == ".ffn" || ext == ".faa");
    }
    
    bool validate_dna_sequence(const std::string& sequence) {
        for (char c : sequence) {
            char upper_c = std::toupper(c);
            if (upper_c != 'A' && upper_c != 'C' && upper_c != 'G' && 
                upper_c != 'T' && upper_c != 'N') {
                return false;
            }
        }
        return true;
    }
    
    bool create_safe_directory(const std::string& directory_path) {
        try {
            std::filesystem::create_directories(directory_path);
            return true;
        } catch (const std::exception&) {
            return false;
        }
    }
    
    std::string format_file_size(size_t bytes) {
        const char* units[] = {"B", "KB", "MB", "GB", "TB"};
        int unit = 0;
        double size = bytes;
        
        while (size >= 1024 && unit < 4) {
            size /= 1024;
            unit++;
        }
        
        std::ostringstream oss;
        oss << std::fixed << std::setprecision(1) << size << " " << units[unit];
        return oss.str();
    }
    
    void print_file_processing_progress(size_t current, size_t total, const std::string& current_file) {
        if (total > 0) {
            double percent = (double)current / total * 100.0;
            std::cout << "Processing file " << current << "/" << total 
                      << " (" << std::fixed << std::setprecision(1) << percent << "%): " 
                      << std::filesystem::path(current_file).filename().string() << std::endl;
        }
    }
    
    bool is_compressed_file(const std::string& file_path) {
        std::filesystem::path p(file_path);
        std::string ext = p.extension().string();
        return (ext == ".gz" || ext == ".bz2" || ext == ".xz" || ext == ".zip");
    }
    
    size_t estimate_uncompressed_size(const std::string& compressed_file) {
        // Simple estimation based on file size
        // In production, would read compression headers
        std::filesystem::path p(compressed_file);
        if (std::filesystem::exists(p)) {
            size_t compressed_size = std::filesystem::file_size(p);
            // Rough estimate: 3x compression ratio
            return compressed_size * 3;
        }
        return 0;
    }
    
    bool has_valid_dna_characters(const std::string& sequence) {
        for (char c : sequence) {
            char upper_c = std::toupper(c);
            if (upper_c != 'A' && upper_c != 'C' && upper_c != 'G' && 
                upper_c != 'T' && upper_c != 'N' && !std::isspace(c)) {
                return false;
            }
        }
        return true;
    }
    
    double calculate_gc_content(const std::string& sequence) {
        if (sequence.empty()) return 0.0;
        
        size_t gc_count = 0;
        for (char c : sequence) {
            char upper_c = std::toupper(c);
            if (upper_c == 'G' || upper_c == 'C') {
                gc_count++;
            }
        }
        
        return (double)gc_count / sequence.length() * 100.0;
    }
    
    bool cleanup_directory(const std::string& directory_path) {
        try {
            std::filesystem::remove_all(directory_path);
            return true;
        } catch (const std::exception&) {
            return false;
        }
    }
    
    std::string generate_temp_filename(const std::string& base_dir, const std::string& prefix, int index) {
        return base_dir + "/" + prefix + "_" + std::to_string(index) + ".tmp";
    }
    
    std::string format_processing_rate(size_t bytes, double seconds) {
        if (seconds <= 0) return "N/A";
        
        double rate = bytes / seconds;
        const char* units[] = {"B/s", "KB/s", "MB/s", "GB/s"};
        int unit = 0;
        
        while (rate >= 1024 && unit < 3) {
            rate /= 1024;
            unit++;
        }
        
        std::ostringstream oss;
        oss << std::fixed << std::setprecision(1) << rate << " " << units[unit];
        return oss.str();
    }
}

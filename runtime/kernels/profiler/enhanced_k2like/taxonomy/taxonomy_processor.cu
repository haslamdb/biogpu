#include "hip/hip_runtime.h"
// taxonomy/taxonomy_processor.cu
// Implementation of taxonomy processing and phylogenetic calculations
// Extracted from monolithic database builder

#include "taxonomy_processor.h"
#include "../gpu_kraken_types.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <set>
#include <iomanip>

// Include compact taxonomy - no conditional compilation needed

// SpeciesTrackingData, PhylogeneticLCACandidate, and ContributingTaxaArrays
// implementations are already defined as inline in gpu_kraken_types.h

// ===========================
// EnhancedNCBITaxonomyProcessor Implementation
// ===========================

EnhancedNCBITaxonomyProcessor::EnhancedNCBITaxonomyProcessor() 
    : taxonomy_loaded(false), max_taxon_id(0) {
}

EnhancedNCBITaxonomyProcessor::~EnhancedNCBITaxonomyProcessor() = default;

bool EnhancedNCBITaxonomyProcessor::load_ncbi_taxonomy(const std::string& nodes_dmp_path, const std::string& names_dmp_path) {
    std::cout << "Loading NCBI taxonomy from DMP files..." << std::endl;
    
    // Use the actual constructor (enable_cache parameter)
    compact_taxonomy = std::make_unique<BioGPU::CompactTaxonomy::CompactGPUTaxonomy>(true);
    
    if (!compact_taxonomy->build_from_ncbi_files(nodes_dmp_path, names_dmp_path)) {
        std::cerr << "Failed to build compact taxonomy from NCBI files" << std::endl;
        return false;
    }
    
    if (!extract_host_lookup_tables()) {
        std::cerr << "Failed to extract lookup tables from compact taxonomy" << std::endl;
        return false;
    }
    
    taxonomy_loaded = true;
    std::cout << "NCBI taxonomy loaded successfully" << std::endl;
    print_taxonomy_statistics();
    
    return true;
}

bool EnhancedNCBITaxonomyProcessor::load_from_compact_file(const std::string& compact_taxonomy_path) {
    std::cout << "Loading from pre-built compact taxonomy: " << compact_taxonomy_path << std::endl;
    
    compact_taxonomy = std::make_unique<BioGPU::CompactTaxonomy::CompactGPUTaxonomy>(true);
    
    if (!compact_taxonomy->load_compact_taxonomy(compact_taxonomy_path)) {
        std::cerr << "Failed to load compact taxonomy file" << std::endl;
        return false;
    }
    
    if (!extract_host_lookup_tables()) {
        std::cerr << "Failed to extract lookup tables from compact taxonomy" << std::endl;
        return false;
    }
    
    taxonomy_loaded = true;
    std::cout << "Compact taxonomy loaded successfully" << std::endl;
    return true;
}

uint32_t EnhancedNCBITaxonomyProcessor::compute_lca_of_species(const std::vector<uint32_t>& species_list) {
    if (!taxonomy_loaded || species_list.empty()) {
        return 1; // Root
    }
    
    if (species_list.size() == 1) {
        return species_list[0];
    }
    
    // Find LCA using proper taxonomy tree traversal
    uint32_t current_lca = species_list[0];
    for (size_t i = 1; i < species_list.size(); i++) {
        current_lca = find_lca_pair(current_lca, species_list[i]);
        if (current_lca == 1) break; // Already at root
    }
    
    return current_lca;
}

uint8_t EnhancedNCBITaxonomyProcessor::calculate_distance_to_lca(uint32_t taxon, uint32_t lca) const {
    if (!taxonomy_loaded || taxon == lca) {
        return 0;
    }
    
    // Use depth lookup if available
    auto taxon_depth_it = depth_lookup.find(taxon);
    auto lca_depth_it = depth_lookup.find(lca);
    
    if (taxon_depth_it != depth_lookup.end() && lca_depth_it != depth_lookup.end()) {
        uint8_t taxon_depth = taxon_depth_it->second;
        uint8_t lca_depth = lca_depth_it->second;
        
        if (taxon_depth >= lca_depth) {
            return taxon_depth - lca_depth;
        }
    }
    
    // Fallback: count steps manually
    uint32_t current = taxon;
    uint8_t distance = 0;
    
    while (current != lca && current != 1 && distance < 50) {
        auto parent_it = parent_lookup.find(current);
        if (parent_it == parent_lookup.end()) break;
        
        current = parent_it->second;
        distance++;
        
        if (current == lca) break;
    }
    
    return (current == lca) ? distance : 255;
}

uint8_t EnhancedNCBITaxonomyProcessor::calculate_phylogenetic_spread(
    const std::vector<uint32_t>& species_list, uint32_t lca) {
    
    if (species_list.size() <= 1) return 0;
    
    std::vector<uint8_t> distances;
    for (uint32_t species : species_list) {
        distances.push_back(calculate_distance_to_lca(species, lca));
    }
    
    uint8_t max_dist = *std::max_element(distances.begin(), distances.end());
    uint8_t min_dist = *std::min_element(distances.begin(), distances.end());
    
    // Enhanced spread calculation using taxonomy ranks
    uint8_t range_spread = max_dist - min_dist;
    uint8_t diversity_factor = std::min((uint8_t)(species_list.size() / 5), (uint8_t)50);
    
    // Weight by taxonomic rank of LCA
    uint8_t rank_weight = get_rank_weight(lca);
    
    return std::min((uint8_t)255, (uint8_t)(range_spread * rank_weight + diversity_factor));
}

std::string EnhancedNCBITaxonomyProcessor::get_scientific_name(uint32_t taxon_id) {
    auto it = name_lookup.find(taxon_id);
    return (it != name_lookup.end()) ? it->second : ("taxon_" + std::to_string(taxon_id));
}

std::string EnhancedNCBITaxonomyProcessor::get_rank(uint32_t taxon_id) {
    auto it = rank_lookup.find(taxon_id);
    return (it != rank_lookup.end()) ? it->second : "no rank";
}

uint32_t EnhancedNCBITaxonomyProcessor::get_parent(uint32_t taxon_id) {
    auto it = parent_lookup.find(taxon_id);
    return (it != parent_lookup.end()) ? it->second : 0;
}

uint8_t EnhancedNCBITaxonomyProcessor::get_depth(uint32_t taxon_id) {
    auto it = depth_lookup.find(taxon_id);
    return (it != depth_lookup.end()) ? it->second : 0;
}

bool EnhancedNCBITaxonomyProcessor::is_loaded() const {
    return taxonomy_loaded;
}

BioGPU::CompactTaxonomy::CompactGPUTaxonomy* EnhancedNCBITaxonomyProcessor::get_compact_taxonomy() {
    return compact_taxonomy.get();
}

size_t EnhancedNCBITaxonomyProcessor::get_taxonomy_size() const {
    return parent_lookup.size();
}

bool EnhancedNCBITaxonomyProcessor::validate_taxon_id(uint32_t taxon_id) const {
    return parent_lookup.find(taxon_id) != parent_lookup.end();
}

void EnhancedNCBITaxonomyProcessor::print_taxonomy_statistics() const {
    std::cout << "\n=== TAXONOMY STATISTICS ===" << std::endl;
    std::cout << "Total taxa: " << parent_lookup.size() << std::endl;
    std::cout << "Named taxa: " << name_lookup.size() << std::endl;
    std::cout << "Ranked taxa: " << rank_lookup.size() << std::endl;
    std::cout << "Max taxon ID: " << max_taxon_id << std::endl;
    
    if (!depth_lookup.empty()) {
        auto max_depth_it = std::max_element(depth_lookup.begin(), depth_lookup.end(),
                                           [](const auto& a, const auto& b) { return a.second < b.second; });
        std::cout << "Maximum depth: " << (int)max_depth_it->second << std::endl;
    }
}

// Private methods
bool EnhancedNCBITaxonomyProcessor::extract_host_lookup_tables() {
    if (!compact_taxonomy) {
        return false;
    }
    
    parent_lookup = compact_taxonomy->get_parent_lookup_map();
    name_lookup = compact_taxonomy->get_name_lookup_map();
    rank_lookup = compact_taxonomy->get_rank_lookup_map();
    depth_lookup = compact_taxonomy->get_depth_lookup_map();
    max_taxon_id = compact_taxonomy->get_max_taxon_id();
    
    std::cout << "Extracted host lookup tables from compact taxonomy" << std::endl;
    return true;
}

uint32_t EnhancedNCBITaxonomyProcessor::find_lca_pair(uint32_t taxon1, uint32_t taxon2) {
    if (taxon1 == taxon2) return taxon1;
    if (taxon1 == 1 || taxon2 == 1) return 1;
    
    // Get paths to root
    std::vector<uint32_t> path1 = get_path_to_root(taxon1);
    std::vector<uint32_t> path2 = get_path_to_root(taxon2);
    
    // Find first common ancestor
    std::set<uint32_t> ancestors1(path1.begin(), path1.end());
    
    for (uint32_t ancestor : path2) {
        if (ancestors1.find(ancestor) != ancestors1.end()) {
            return ancestor;
        }
    }
    
    return 1; // Root fallback
}

std::vector<uint32_t> EnhancedNCBITaxonomyProcessor::get_path_to_root(uint32_t taxon_id) {
    std::vector<uint32_t> path;
    uint32_t current = taxon_id;
    
    while (current != 1 && path.size() < 50) {
        path.push_back(current);
        auto parent_it = parent_lookup.find(current);
        if (parent_it == parent_lookup.end()) break;
        current = parent_it->second;
    }
    
    path.push_back(1); // Add root
    return path;
}

uint8_t EnhancedNCBITaxonomyProcessor::get_rank_weight(uint32_t taxon_id) {
    std::string rank = get_rank(taxon_id);
    
    // Weight factors based on taxonomic rank
    if (rank == "species" || rank == "subspecies") return 1;
    else if (rank == "genus") return 2;
    else if (rank == "family") return 3;
    else if (rank == "order") return 4;
    else if (rank == "class") return 5;
    else if (rank == "phylum") return 6;
    else if (rank == "kingdom" || rank == "superkingdom") return 7;
    else return 3; // Default for "no rank"
}

bool EnhancedNCBITaxonomyProcessor::parse_nodes_dmp(const std::string& nodes_file) {
    std::cout << "Parsing nodes.dmp: " << nodes_file << std::endl;
    
    std::ifstream file(nodes_file);
    if (!file.is_open()) {
        std::cerr << "Cannot open nodes.dmp: " << nodes_file << std::endl;
        return false;
    }
    
    std::string line;
    int nodes_loaded = 0;
    
    while (std::getline(file, line)) {
        if (line.empty()) continue;
        
        std::istringstream iss(line);
        std::string token;
        std::vector<std::string> fields;
        
        while (std::getline(iss, token, '|')) {
            // Trim whitespace
            token.erase(0, token.find_first_not_of(" \t"));
            token.erase(token.find_last_not_of(" \t") + 1);
            fields.push_back(token);
        }
        
        if (fields.size() >= 3) {
            try {
                uint32_t taxon_id = std::stoul(fields[0]);
                uint32_t parent_id = std::stoul(fields[1]);
                std::string rank = fields[2];
                
                parent_lookup[taxon_id] = parent_id;
                rank_lookup[taxon_id] = rank;
                max_taxon_id = std::max(max_taxon_id, taxon_id);
                nodes_loaded++;
                
                // Default name
                name_lookup[taxon_id] = "taxon_" + std::to_string(taxon_id);
                
            } catch (const std::exception& e) {
                continue; // Skip invalid lines
            }
        }
    }
    
    file.close();
    std::cout << "Loaded " << nodes_loaded << " taxonomy nodes" << std::endl;
    return nodes_loaded > 0;
}

bool EnhancedNCBITaxonomyProcessor::parse_names_dmp(const std::string& names_file) {
    std::cout << "Parsing names.dmp: " << names_file << std::endl;
    
    std::ifstream file(names_file);
    if (!file.is_open()) {
        std::cerr << "Cannot open names.dmp: " << names_file << std::endl;
        return false;
    }
    
    std::string line;
    int names_loaded = 0;
    
    while (std::getline(file, line)) {
        if (line.empty()) continue;
        
        std::istringstream iss(line);
        std::string token;
        std::vector<std::string> fields;
        
        while (std::getline(iss, token, '|')) {
            // Trim whitespace
            token.erase(0, token.find_first_not_of(" \t"));
            token.erase(token.find_last_not_of(" \t") + 1);
            fields.push_back(token);
        }
        
        if (fields.size() >= 4) {
            try {
                uint32_t taxon_id = std::stoul(fields[0]);
                std::string name_txt = fields[1];
                std::string name_class = fields[3];
                
                if (name_class == "scientific name" && parent_lookup.find(taxon_id) != parent_lookup.end()) {
                    name_lookup[taxon_id] = name_txt;
                    names_loaded++;
                }
            } catch (const std::exception& e) {
                continue; // Skip invalid lines
            }
        }
    }
    
    file.close();
    std::cout << "Loaded " << names_loaded << " scientific names" << std::endl;
    return true;
}

void EnhancedNCBITaxonomyProcessor::build_depth_lookup() {
    std::cout << "Building depth lookup table..." << std::endl;
    
    depth_lookup.clear();
    depth_lookup[1] = 0; // Root has depth 0
    
    // BFS to assign depths
    std::vector<uint32_t> current_level = {1};
    uint8_t current_depth = 0;
    
    while (!current_level.empty() && current_depth < 50) {
        std::vector<uint32_t> next_level;
        
        for (uint32_t taxon_id : current_level) {
            depth_lookup[taxon_id] = current_depth;
            
            // Find children
            for (const auto& [child_id, parent_id] : parent_lookup) {
                if (parent_id == taxon_id && depth_lookup.find(child_id) == depth_lookup.end()) {
                    next_level.push_back(child_id);
                }
            }
        }
        
        current_level = std::move(next_level);
        current_depth++;
    }
    
    std::cout << "Built depth lookup for " << depth_lookup.size() << " taxa" << std::endl;
}

// ===========================
// SimpleTaxonomyProcessor Implementation
// ===========================

SimpleTaxonomyProcessor::SimpleTaxonomyProcessor() : taxonomy_loaded(false) {
}

bool SimpleTaxonomyProcessor::load_ncbi_files(const std::string& nodes_dmp_path, const std::string& names_dmp_path) {
    std::cout << "Loading NCBI taxonomy with simple processor..." << std::endl;
    
    // Load nodes.dmp
    std::ifstream nodes_file(nodes_dmp_path);
    if (!nodes_file.is_open()) {
        std::cerr << "Cannot open nodes.dmp: " << nodes_dmp_path << std::endl;
        return false;
    }
    
    std::string line;
    while (std::getline(nodes_file, line)) {
        uint32_t taxon_id, parent_id;
        std::string rank;
        
        if (PhylogeneticUtils::parse_taxonomy_line(line, taxon_id, parent_id, taxon_names[taxon_id], rank)) {
            taxon_parents[taxon_id] = parent_id;
            taxon_ranks[taxon_id] = rank;
            taxon_names[taxon_id] = "taxon_" + std::to_string(taxon_id); // Default name
        }
    }
    nodes_file.close();
    
    // Load names.dmp if available
    std::ifstream names_file(names_dmp_path);
    if (names_file.is_open()) {
        while (std::getline(names_file, line)) {
            // Parse names.dmp format (simplified)
            std::istringstream iss(line);
            std::string taxon_str, name, unique_name, name_class;
            
            if (std::getline(iss, taxon_str, '|') &&
                std::getline(iss, name, '|') &&
                std::getline(iss, unique_name, '|') &&
                std::getline(iss, name_class, '|')) {
                
                // Trim whitespace
                name.erase(0, name.find_first_not_of(" \t"));
                name.erase(name.find_last_not_of(" \t") + 1);
                name_class.erase(0, name_class.find_first_not_of(" \t"));
                name_class.erase(name_class.find_last_not_of(" \t") + 1);
                
                if (name_class == "scientific name") {
                    try {
                        uint32_t taxon_id = std::stoul(taxon_str);
                        if (taxon_parents.find(taxon_id) != taxon_parents.end()) {
                            taxon_names[taxon_id] = name;
                        }
                    } catch (const std::exception&) {
                        continue;
                    }
                }
            }
        }
        names_file.close();
    }
    
    taxonomy_loaded = validate_taxonomy_consistency();
    std::cout << "Simple taxonomy loaded: " << taxon_parents.size() << " taxa" << std::endl;
    
    return taxonomy_loaded;
}

bool SimpleTaxonomyProcessor::load_taxonomy_tsv(const std::string& taxonomy_tsv_path) {
    std::cout << "Loading taxonomy from TSV: " << taxonomy_tsv_path << std::endl;
    
    std::ifstream file(taxonomy_tsv_path);
    if (!file.is_open()) {
        std::cerr << "Cannot open taxonomy TSV: " << taxonomy_tsv_path << std::endl;
        return false;
    }
    
    std::string line;
    std::getline(file, line); // Skip header
    
    int taxa_loaded = 0;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string taxon_str, parent_str, name, rank;
        
        if (std::getline(iss, taxon_str, '\t') &&
            std::getline(iss, parent_str, '\t') &&
            std::getline(iss, name, '\t') &&
            std::getline(iss, rank, '\t')) {
            
            try {
                uint32_t taxon_id = std::stoul(taxon_str);
                uint32_t parent_id = std::stoul(parent_str);
                
                taxon_parents[taxon_id] = parent_id;
                taxon_names[taxon_id] = name;
                taxon_ranks[taxon_id] = rank;
                taxa_loaded++;
            } catch (const std::exception&) {
                continue;
            }
        }
    }
    
    file.close();
    taxonomy_loaded = taxa_loaded > 0;
    std::cout << "Loaded " << taxa_loaded << " taxa from TSV" << std::endl;
    
    return taxonomy_loaded;
}

uint32_t SimpleTaxonomyProcessor::compute_simple_lca(uint32_t taxon1, uint32_t taxon2) {
    return LCAAlgorithms::compute_lca_pair(taxon1, taxon2, taxon_parents);
}

uint32_t SimpleTaxonomyProcessor::compute_lca_of_list(const std::vector<uint32_t>& taxon_list) {
    return LCAAlgorithms::compute_lca_multiple(taxon_list, taxon_parents);
}

std::string SimpleTaxonomyProcessor::get_name(uint32_t taxon_id) const {
    auto it = taxon_names.find(taxon_id);
    return (it != taxon_names.end()) ? it->second : ("taxon_" + std::to_string(taxon_id));
}

std::string SimpleTaxonomyProcessor::get_rank(uint32_t taxon_id) const {
    auto it = taxon_ranks.find(taxon_id);
    return (it != taxon_ranks.end()) ? it->second : "no rank";
}

uint32_t SimpleTaxonomyProcessor::get_parent(uint32_t taxon_id) const {
    auto it = taxon_parents.find(taxon_id);
    return (it != taxon_parents.end()) ? it->second : 0;
}

void SimpleTaxonomyProcessor::add_taxon(uint32_t taxon_id, uint32_t parent_id, const std::string& name, const std::string& rank) {
    taxon_parents[taxon_id] = parent_id;
    taxon_names[taxon_id] = name;
    taxon_ranks[taxon_id] = rank;
}

bool SimpleTaxonomyProcessor::save_taxonomy_tsv(const std::string& output_path) const {
    std::ofstream out(output_path);
    if (!out.is_open()) {
        return false;
    }
    
    out << "taxon_id\tparent_id\tname\trank\n";
    for (const auto& [taxon_id, parent_id] : taxon_parents) {
        std::string name = get_name(taxon_id);
        std::string rank = get_rank(taxon_id);
        out << taxon_id << "\t" << parent_id << "\t" << name << "\t" << rank << "\n";
    }
    
    out.close();
    return true;
}

std::vector<uint32_t> SimpleTaxonomyProcessor::get_lineage(uint32_t taxon_id) const {
    return LCAAlgorithms::get_path_to_root(taxon_id, taxon_parents);
}

bool SimpleTaxonomyProcessor::validate_taxonomy_consistency() const {
    // Basic validation: ensure root exists and no cycles
    if (taxon_parents.find(1) == taxon_parents.end()) {
        std::cerr << "Warning: Root taxon (1) not found in taxonomy" << std::endl;
        return false;
    }
    
    // Check for basic consistency
    for (const auto& [taxon_id, parent_id] : taxon_parents) {
        if (taxon_id != 1 && parent_id != 0 && taxon_parents.find(parent_id) == taxon_parents.end()) {
            std::cerr << "Warning: Parent " << parent_id << " not found for taxon " << taxon_id << std::endl;
        }
    }
    
    return true;
}

// ===========================
// Utility Namespace Implementations
// ===========================

namespace PhylogeneticUtils {
    
    bool parse_taxonomy_line(const std::string& line, uint32_t& taxon_id, uint32_t& parent_id, 
                            std::string& name, std::string& rank) {
        std::istringstream iss(line);
        std::string token;
        std::vector<std::string> fields;
        
        while (std::getline(iss, token, '\t')) {
            fields.push_back(token);
        }
        
        if (fields.size() >= 4) {
            try {
                taxon_id = std::stoul(fields[0]);
                parent_id = std::stoul(fields[1]);
                name = fields[2];
                rank = fields[3];
                return true;
            } catch (const std::exception&) {
                return false;
            }
        }
        
        return false;
    }
    
    std::string format_taxonomy_line(uint32_t taxon_id, uint32_t parent_id, 
                                    const std::string& name, const std::string& rank) {
        return std::to_string(taxon_id) + "\t" + std::to_string(parent_id) + "\t" + name + "\t" + rank;
    }
    
    bool validate_species_list(const std::vector<uint32_t>& species_list) {
        if (species_list.empty()) return false;
        
        // Check for valid taxon IDs
        for (uint32_t taxon_id : species_list) {
            if (!is_valid_taxon_id(taxon_id)) {
                return false;
            }
        }
        
        return true;
    }
    
    bool is_valid_taxon_id(uint32_t taxon_id) {
        return taxon_id > 0 && taxon_id < 10000000; // Reasonable range
    }
}

namespace LCAAlgorithms {
    
    uint32_t compute_lca_pair(uint32_t taxon1, uint32_t taxon2, 
                             const std::unordered_map<uint32_t, uint32_t>& parents) {
        if (taxon1 == taxon2) return taxon1;
        if (taxon1 == 1 || taxon2 == 1) return 1;
        
        std::vector<uint32_t> path1 = get_path_to_root(taxon1, parents);
        std::vector<uint32_t> path2 = get_path_to_root(taxon2, parents);
        
        std::set<uint32_t> ancestors1(path1.begin(), path1.end());
        
        for (uint32_t ancestor : path2) {
            if (ancestors1.find(ancestor) != ancestors1.end()) {
                return ancestor;
            }
        }
        
        return 1; // Root fallback
    }
    
    uint32_t compute_lca_multiple(const std::vector<uint32_t>& taxa, 
                                 const std::unordered_map<uint32_t, uint32_t>& parents) {
        if (taxa.empty()) return 1;
        if (taxa.size() == 1) return taxa[0];
        
        uint32_t lca = taxa[0];
        for (size_t i = 1; i < taxa.size(); i++) {
            lca = compute_lca_pair(lca, taxa[i], parents);
            if (lca == 1) break; // Already at root
        }
        
        return lca;
    }
    
    std::vector<uint32_t> get_path_to_root(uint32_t taxon_id, 
                                          const std::unordered_map<uint32_t, uint32_t>& parents) {
        std::vector<uint32_t> path;
        uint32_t current = taxon_id;
        
        while (current != 1 && path.size() < 50) {
            path.push_back(current);
            auto parent_it = parents.find(current);
            if (parent_it == parents.end()) break;
            current = parent_it->second;
        }
        
        path.push_back(1); // Add root
        return path;
    }
    
    uint32_t compute_lca_with_depths(const std::vector<uint32_t>& taxa,
                                    const std::unordered_map<uint32_t, uint32_t>& parents,
                                    const std::unordered_map<uint32_t, uint8_t>& depths) {
        if (taxa.empty()) return 1;
        if (taxa.size() == 1) return taxa[0];
        
        // Use regular LCA computation - depth optimization can be added later
        return compute_lca_multiple(taxa, parents);
    }
    
    uint32_t find_common_ancestor_from_paths(const std::vector<std::vector<uint32_t>>& paths) {
        if (paths.empty()) return 1;
        if (paths.size() == 1) return paths[0].empty() ? 1 : paths[0][0];
        
        // Find the first common element across all paths
        std::set<uint32_t> candidates(paths[0].begin(), paths[0].end());
        
        for (size_t i = 1; i < paths.size(); i++) {
            std::set<uint32_t> current_path(paths[i].begin(), paths[i].end());
            std::set<uint32_t> intersection;
            
            std::set_intersection(candidates.begin(), candidates.end(),
                                current_path.begin(), current_path.end(),
                                std::inserter(intersection, intersection.begin()));
            
            candidates = intersection;
            if (candidates.empty()) break;
        }
        
        return candidates.empty() ? 1 : *candidates.begin();
    }
}

// ADD missing PhylogeneticUtils implementations:
namespace PhylogeneticUtils {
    
    uint8_t calculate_taxonomic_distance(uint32_t taxon1, uint32_t taxon2, 
                                        const std::unordered_map<uint32_t, uint32_t>& parents) {
        if (taxon1 == taxon2) return 0;
        
        std::vector<uint32_t> path1 = LCAAlgorithms::get_path_to_root(taxon1, parents);
        std::vector<uint32_t> path2 = LCAAlgorithms::get_path_to_root(taxon2, parents);
        
        uint32_t lca = LCAAlgorithms::compute_lca_pair(taxon1, taxon2, parents);
        
        uint8_t distance1 = 0, distance2 = 0;
        for (uint32_t taxon : path1) {
            if (taxon == lca) break;
            distance1++;
        }
        
        for (uint32_t taxon : path2) {
            if (taxon == lca) break;
            distance2++;
        }
        
        return distance1 + distance2;
    }
    
    double calculate_phylogenetic_diversity(const std::vector<uint32_t>& species_list,
                                           const EnhancedNCBITaxonomyProcessor& taxonomy) {
        if (species_list.size() <= 1) return 0.0;
        
        // Simple diversity metric based on pairwise distances
        double total_distance = 0.0;
        int pairs = 0;
        
        for (size_t i = 0; i < species_list.size(); i++) {
            for (size_t j = i + 1; j < species_list.size(); j++) {
                uint32_t lca = const_cast<EnhancedNCBITaxonomyProcessor&>(taxonomy)
                    .compute_lca_of_species({species_list[i], species_list[j]});
                
                uint8_t dist1 = taxonomy.calculate_distance_to_lca(species_list[i], lca);
                uint8_t dist2 = taxonomy.calculate_distance_to_lca(species_list[j], lca);
                
                total_distance += (dist1 + dist2);
                pairs++;
            }
        }
        
        return pairs > 0 ? total_distance / pairs : 0.0;
    }
    
    uint8_t calculate_taxonomic_spread(const std::vector<uint32_t>& taxa_list, uint32_t lca,
                                      const std::unordered_map<uint32_t, uint32_t>& parents) {
        if (taxa_list.empty()) return 0;
        
        std::vector<uint8_t> distances;
        for (uint32_t taxon : taxa_list) {
            uint8_t distance = 0;
            uint32_t current = taxon;
            
            while (current != lca && current != 1 && distance < 50) {
                auto parent_it = parents.find(current);
                if (parent_it == parents.end()) break;
                current = parent_it->second;
                distance++;
            }
            
            distances.push_back(distance);
        }
        
        if (distances.empty()) return 0;
        
        uint8_t max_dist = *std::max_element(distances.begin(), distances.end());
        uint8_t min_dist = *std::min_element(distances.begin(), distances.end());
        
        return max_dist - min_dist;
    }
    
    std::vector<uint32_t> extract_species_from_candidates(const std::vector<PhylogeneticLCACandidate>& candidates) {
        std::set<uint32_t> unique_species;
        
        for (const auto& candidate : candidates) {
            for (uint32_t species : candidate.contributing_species) {
                unique_species.insert(species);
            }
        }
        
        return std::vector<uint32_t>(unique_species.begin(), unique_species.end());
    }
    
    std::unordered_map<uint32_t, uint16_t> count_genomes_per_species(const std::vector<PhylogeneticLCACandidate>& candidates) {
        std::unordered_map<uint32_t, uint16_t> species_counts;
        
        for (const auto& candidate : candidates) {
            for (size_t i = 0; i < candidate.contributing_species.size(); i++) {
                uint32_t species = candidate.contributing_species[i];
                uint16_t count = (i < candidate.genome_counts_per_species.size()) ? 
                                candidate.genome_counts_per_species[i] : 1;
                species_counts[species] += count;
            }
        }
        
        return species_counts;
    }
}

// Stub implementation for CompactGPUTaxonomy
namespace BioGPU {
namespace CompactTaxonomy {

class CompactGPUTaxonomy::Impl {
public:
    std::unordered_map<uint32_t, uint32_t> parent_map;
    std::unordered_map<uint32_t, std::string> names_map;
    std::unordered_map<uint32_t, std::string> ranks_map;
    bool enable_cache;
};

CompactGPUTaxonomy::CompactGPUTaxonomy(bool enable_cache) 
    : pImpl(std::make_unique<Impl>()) {
    pImpl->enable_cache = enable_cache;
}

CompactGPUTaxonomy::~CompactGPUTaxonomy() = default;

bool CompactGPUTaxonomy::build_from_ncbi_files(const std::string& nodes_file, const std::string& names_file) {
    // Stub implementation - just return true for now
    std::cout << "CompactGPUTaxonomy: Loading taxonomy files (stub implementation)" << std::endl;
    return true;
}

bool CompactGPUTaxonomy::load_compact_taxonomy(const std::string& compact_file) {
    // Stub implementation
    return true;
}

bool CompactGPUTaxonomy::save_compact_taxonomy(const std::string& compact_file) const {
    // Stub implementation
    return true;
}

uint32_t CompactGPUTaxonomy::get_parent(uint32_t taxon_id) const {
    auto it = pImpl->parent_map.find(taxon_id);
    return (it != pImpl->parent_map.end()) ? it->second : 0;
}

std::string CompactGPUTaxonomy::get_name(uint32_t taxon_id) const {
    auto it = pImpl->names_map.find(taxon_id);
    return (it != pImpl->names_map.end()) ? it->second : "";
}

std::string CompactGPUTaxonomy::get_rank(uint32_t taxon_id) const {
    auto it = pImpl->ranks_map.find(taxon_id);
    return (it != pImpl->ranks_map.end()) ? it->second : "";
}

void CompactGPUTaxonomy::get_all_taxon_ids(std::vector<uint32_t>& taxon_ids) const {
    taxon_ids.clear();
    for (const auto& pair : pImpl->parent_map) {
        taxon_ids.push_back(pair.first);
    }
}

bool CompactGPUTaxonomy::build_gpu_structures() {
    // Stub implementation
    return true;
}

void CompactGPUTaxonomy::free_gpu_memory() {
    // Stub implementation
}

} // namespace CompactTaxonomy
} // namespace BioGPU

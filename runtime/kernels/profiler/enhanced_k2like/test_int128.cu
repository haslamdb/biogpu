#include <hip/hip_runtime.h>
#include <iostream>

__global__ void test_int128_kernel(bool* result) {
    // Test basic 128-bit arithmetic
    unsigned __int128 a = 1;
    a = a << 70;  // Shift by 70 bits (requires 128-bit)
    
    unsigned __int128 b = 1;
    b = b << 69;
    
    // Test comparison
    *result = (a > b);
}

int main() {
    bool* d_result;
    bool h_result = false;
    
    hipMalloc(&d_result, sizeof(bool));
    
    test_int128_kernel<<<1, 1>>>(d_result);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    
    hipMemcpy(&h_result, d_result, sizeof(bool), hipMemcpyDeviceToHost);
    
    std::cout << "128-bit integer test " << (h_result ? "PASSED" : "FAILED") << std::endl;
    std::cout << "Result: " << h_result << " (expected: 1)" << std::endl;
    
    hipFree(d_result);
    
    return h_result ? 0 : 1;
}
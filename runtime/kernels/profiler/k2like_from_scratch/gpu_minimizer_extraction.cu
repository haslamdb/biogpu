#include "hip/hip_runtime.h"
// gpu_minimizer_extraction.cu
// FIXED: Optimized GPU kernel for Kraken2-style minimizer extraction
// Implements proper sliding window minimizer algorithm with deduplication

#ifndef GPU_MINIMIZER_EXTRACTION_CU
#define GPU_MINIMIZER_EXTRACTION_CU

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <hipcub/hipcub.hpp>
#include "../../../include/biogpu/minimizer_extraction.h"

// Configuration constants
#define MAX_KMER_LENGTH 64
#define MAX_THREADS_PER_BLOCK 256
#define SHARED_MEMORY_SIZE 48 * 1024  // 48KB shared memory per block

// Minimizer hit structure (same as your original)
struct GPUMinimizerHit {
    uint64_t minimizer_hash;
    uint32_t taxon_id;
    uint32_t position;
    uint32_t genome_id;
};

// Genome info structure
struct GPUGenomeInfo {
    uint32_t taxon_id;
    uint32_t sequence_offset;
    uint32_t sequence_length;
    uint32_t genome_id;
};

// MinimizerParams is defined in minimizer_extraction.h

// Device functions for minimizer computation
__device__ uint64_t encode_base(char base) {
    switch (base) {
        case 'A': case 'a': return 0;
        case 'C': case 'c': return 1;
        case 'G': case 'g': return 2;
        case 'T': case 't': return 3;
        default: return 4;  // Invalid base marker
    }
}

__device__ uint64_t hash_lmer(const char* sequence, int pos, int ell) {
    uint64_t hash = 0;
    for (int i = 0; i < ell; i++) {
        uint64_t base = encode_base(sequence[pos + i]);
        if (base == 4) return UINT64_MAX;  // Invalid sequence
        hash = (hash << 2) | base;
    }
    return hash;
}

__device__ uint64_t apply_spaced_seed_mask(uint64_t hash, int spaces, int ell) {
    if (spaces == 0) return hash;
    
    uint64_t masked_hash = 0;
    int out_pos = 0;
    
    // Apply spaced seed pattern: keep every (spaces+1)th position
    for (int i = 0; i < ell; i++) {
        if (i % (spaces + 1) == 0) {
            uint64_t base = (hash >> (2 * (ell - 1 - i))) & 3;
            masked_hash = (masked_hash << 2) | base;
            out_pos++;
        }
    }
    
    return masked_hash;
}

__device__ uint64_t compute_canonical_minimizer(uint64_t hash, uint64_t xor_mask) {
    // Apply XOR shuffling to avoid bias toward low-complexity sequences
    return hash ^ xor_mask;
}

__device__ bool is_valid_sequence(const char* seq, int len) {
    for (int i = 0; i < len; i++) {
        char c = seq[i];
        if (c != 'A' && c != 'C' && c != 'G' && c != 'T' &&
            c != 'a' && c != 'c' && c != 'g' && c != 't') {
            return false;
        }
    }
    return true;
}

// Sliding window minimizer extraction for a single k-mer
__device__ uint64_t extract_minimizer_sliding_window(
    const char* sequence, 
    int kmer_pos, 
    int k, 
    int ell, 
    int spaces,
    uint64_t xor_mask) {
    
    uint64_t min_hash = UINT64_MAX;
    
    // Slide window of size ell across the k-mer
    for (int i = 0; i <= k - ell; i++) {
        if (!is_valid_sequence(sequence + kmer_pos + i, ell)) {
            continue;
        }
        
        uint64_t lmer_hash = hash_lmer(sequence, kmer_pos + i, ell);
        if (lmer_hash == UINT64_MAX) continue;
        
        // Apply spaced seed mask
        uint64_t masked_hash = apply_spaced_seed_mask(lmer_hash, spaces, ell);
        
        // Apply XOR shuffling
        uint64_t canonical_hash = compute_canonical_minimizer(masked_hash, xor_mask);
        
        if (canonical_hash < min_hash) {
            min_hash = canonical_hash;
        }
    }
    
    return min_hash;
}

// FIXED: Proper Kraken2-style sliding window minimizer extraction
__global__ void extract_minimizers_optimized_kernel(
    const char* sequence_data,
    const GPUGenomeInfo* genome_info,
    int num_genomes,
    GPUMinimizerHit* minimizer_hits,
    uint32_t* hit_counts_per_genome,
    uint32_t* global_hit_counter,
    MinimizerParams params,
    int max_minimizers) {
    
    int genome_id = blockIdx.x;
    int thread_id = threadIdx.x;
    
    if (genome_id >= num_genomes) return;
    
    const GPUGenomeInfo& genome = genome_info[genome_id];
    const char* sequence = sequence_data + genome.sequence_offset;
    uint32_t seq_length = genome.sequence_length;
    
    if (seq_length < params.k) {
        if (thread_id == 0) hit_counts_per_genome[genome_id] = 0;
        return;
    }
    
    // FIXED: Only thread 0 per block to ensure proper sequential processing
    if (thread_id != 0) return;
    
    uint32_t local_minimizer_count = 0;
    uint64_t last_minimizer = UINT64_MAX;  // KEY FIX: Track last minimizer
    uint32_t total_kmers = seq_length - params.k + 1;
    
    // FIXED: Process k-mers sequentially with proper deduplication
    for (uint32_t kmer_idx = 0; kmer_idx < total_kmers; kmer_idx++) {
        
        // Extract minimizer for this k-mer
        uint64_t minimizer = extract_minimizer_sliding_window(
            sequence, kmer_idx, params.k, params.ell, params.spaces, params.xor_mask
        );
        
        if (minimizer != UINT64_MAX) {
            // KEY FIX: Only store if different from last minimizer
            if (minimizer != last_minimizer) {
                // Check if we still have space before incrementing
                uint32_t current_count = atomicAdd(global_hit_counter, 0); // Read without modifying
                if (current_count >= max_minimizers) {
                    // Buffer is full, stop processing
                    break;
                }
                
                uint32_t global_pos = atomicAdd(global_hit_counter, 1);
                
                if (global_pos < max_minimizers) {
                    GPUMinimizerHit hit;
                    hit.minimizer_hash = minimizer;
                    hit.taxon_id = genome.taxon_id;
                    hit.position = kmer_idx;
                    hit.genome_id = genome.genome_id;
                    
                    minimizer_hits[global_pos] = hit;
                    local_minimizer_count++;
                } else {
                    // We exceeded the limit, stop processing
                    break;
                }
                
                last_minimizer = minimizer;  // Update last seen minimizer
            }
            // If minimizer == last_minimizer, skip it (this creates compression!)
        }
    }
    
    // Store final count for this genome
    hit_counts_per_genome[genome_id] = local_minimizer_count;
}

// Alternative kernel: multi-threaded version with cooperation (for very large genomes)
__global__ void extract_minimizers_cooperative_kernel(
    const char* sequence_data,
    const GPUGenomeInfo* genome_info,
    int num_genomes,
    GPUMinimizerHit* minimizer_hits,
    uint32_t* hit_counts_per_genome,
    uint32_t* global_hit_counter,
    MinimizerParams params,
    int max_minimizers) {
    
    int genome_id = blockIdx.x;
    int thread_id = threadIdx.x;
    int block_size = blockDim.x;
    
    if (genome_id >= num_genomes) return;
    
    const GPUGenomeInfo& genome = genome_info[genome_id];
    const char* sequence = sequence_data + genome.sequence_offset;
    uint32_t seq_length = genome.sequence_length;
    
    if (seq_length < params.k) {
        if (thread_id == 0) hit_counts_per_genome[genome_id] = 0;
        return;
    }
    
    // Shared memory for temporary storage
    extern __shared__ uint64_t shared_minimizers[];
    uint32_t* shared_positions = (uint32_t*)(shared_minimizers + block_size);
    
    uint32_t total_kmers = seq_length - params.k + 1;
    uint32_t local_count = 0;
    
    // Divide work among threads, but process results sequentially
    uint32_t kmers_per_thread = (total_kmers + block_size - 1) / block_size;
    uint32_t start_kmer = thread_id * kmers_per_thread;
    uint32_t end_kmer = min(start_kmer + kmers_per_thread, total_kmers);
    
    // Each thread extracts minimizers for its assigned k-mers
    uint32_t thread_minimizer_count = 0;
    for (uint32_t kmer_idx = start_kmer; kmer_idx < end_kmer; kmer_idx++) {
        uint64_t minimizer = extract_minimizer_sliding_window(
            sequence, kmer_idx, params.k, params.ell, params.spaces, params.xor_mask
        );
        
        if (minimizer != UINT64_MAX && thread_minimizer_count < block_size) {
            shared_minimizers[thread_id] = minimizer;
            shared_positions[thread_id] = kmer_idx;
            thread_minimizer_count++;
            break; // Only store first valid minimizer per thread for now
        }
    }
    
    __syncthreads();
    
    // Thread 0 processes all minimizers sequentially to maintain order
    if (thread_id == 0) {
        uint64_t last_minimizer = UINT64_MAX;
        
        for (uint32_t kmer_idx = 0; kmer_idx < total_kmers; kmer_idx++) {
            uint64_t minimizer = extract_minimizer_sliding_window(
                sequence, kmer_idx, params.k, params.ell, params.spaces, params.xor_mask
            );
            
            if (minimizer != UINT64_MAX && minimizer != last_minimizer) {
                uint32_t global_pos = atomicAdd(global_hit_counter, 1);
                
                if (global_pos < max_minimizers) {
                    GPUMinimizerHit hit;
                    hit.minimizer_hash = minimizer;
                    hit.taxon_id = genome.taxon_id;
                    hit.position = kmer_idx;
                    hit.genome_id = genome.genome_id;
                    
                    minimizer_hits[global_pos] = hit;
                    local_count++;
                }
                
                last_minimizer = minimizer;
            }
        }
        
        hit_counts_per_genome[genome_id] = local_count;
    }
}

// Keep your working kernel as fallback
__global__ void extract_minimizers_sliding_window_kernel(
    const char* sequence_data,
    const GPUGenomeInfo* genome_info,
    int num_genomes,
    GPUMinimizerHit* minimizer_hits,
    uint32_t* global_hit_counter,
    MinimizerParams params,
    int max_minimizers) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_genomes) return;
    
    const GPUGenomeInfo& genome = genome_info[idx];
    const char* sequence = sequence_data + genome.sequence_offset;
    uint32_t seq_length = genome.sequence_length;
    
    if (seq_length < params.k) return;
    
    // This kernel already has the correct logic!
    uint64_t prev_minimizer = UINT64_MAX;
    uint32_t total_kmers = seq_length - params.k + 1;
    
    for (uint32_t pos = 0; pos < total_kmers; pos++) {
        uint64_t current_minimizer = extract_minimizer_sliding_window(
            sequence, pos, params.k, params.ell, params.spaces, params.xor_mask
        );
        
        // Only output if minimizer changed (proper deduplication)
        if (current_minimizer != UINT64_MAX && current_minimizer != prev_minimizer) {
            uint32_t global_pos = atomicAdd(global_hit_counter, 1);
            
            if (global_pos < max_minimizers) {
                GPUMinimizerHit hit;
                hit.minimizer_hash = current_minimizer;
                hit.taxon_id = genome.taxon_id;
                hit.position = pos;
                hit.genome_id = genome.genome_id;
                
                minimizer_hits[global_pos] = hit;
            }
            
            prev_minimizer = current_minimizer;
        }
    }
}

// FIXED: Host function to launch optimized minimizer extraction
bool extract_minimizers_gpu_optimized(
    const char* d_sequence_data,
    const GPUGenomeInfo* d_genome_info,
    int num_genomes,
    GPUMinimizerHit* d_minimizer_hits,
    uint32_t* d_hit_counts,
    uint32_t* total_hits,
    MinimizerParams params,
    int max_minimizers) {
    
    // Reset global counter
    uint32_t zero = 0;
    uint32_t* d_global_counter;
    hipMalloc(&d_global_counter, sizeof(uint32_t));
    hipMemcpy(d_global_counter, &zero, sizeof(uint32_t), hipMemcpyHostToDevice);
    
    // Clear hit counts
    hipMemset(d_hit_counts, 0, num_genomes * sizeof(uint32_t));
    
    // FIXED: Always use the corrected kernel with proper deduplication
    // Use 1 thread per block for proper sequential processing
    extract_minimizers_optimized_kernel<<<num_genomes, 1>>>(
        d_sequence_data, d_genome_info, num_genomes,
        d_minimizer_hits, d_hit_counts, d_global_counter,
        params, max_minimizers
    );
    
    hipDeviceSynchronize();
    
    // Check for kernel errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error in minimizer extraction: %s\n", hipGetErrorString(error));
        hipFree(d_global_counter);
        return false;
    }
    
    // Get total hit count
    hipMemcpy(total_hits, d_global_counter, sizeof(uint32_t), hipMemcpyDeviceToHost);
    
    // CRITICAL: Clamp total_hits to max_minimizers to prevent overflow
    if (*total_hits > max_minimizers) {
        printf("WARNING: Minimizer extraction hit limit. Clamping %u to %d\n", 
               *total_hits, max_minimizers);
        *total_hits = max_minimizers;
    }
    
    hipFree(d_global_counter);
    
    return true;
}

// Host function for post-processing deduplication using Thrust
bool deduplicate_minimizers_gpu(
    GPUMinimizerHit* d_minimizer_hits,
    uint32_t num_hits,
    uint32_t* final_count,
    uint32_t max_allocated_hits) {
    
    if (num_hits == 0) {
        *final_count = 0;
        return true;
    }
    
    // CRITICAL: Check bounds before processing
    if (num_hits > max_allocated_hits) {
        printf("ERROR: num_hits (%u) exceeds allocated memory (%u)!\n", 
               num_hits, max_allocated_hits);
        return false;
    }
    
    // Sort by minimizer hash
    thrust::device_ptr<GPUMinimizerHit> hits_ptr(d_minimizer_hits);
    
    try {
        // Add explicit CUDA error checking before sort
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error before sort: %s\n", hipGetErrorString(err));
            return false;
        }
        
        thrust::sort(hits_ptr, hits_ptr + num_hits, 
            [] __device__ (const GPUMinimizerHit& a, const GPUMinimizerHit& b) {
                return a.minimizer_hash < b.minimizer_hash;
            });
        
        // Check for errors after sort
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error after sort: %s\n", hipGetErrorString(err));
            return false;
        }
        
        // Remove duplicates (keep first occurrence of each hash)
        auto new_end = thrust::unique(hits_ptr, hits_ptr + num_hits,
            [] __device__ (const GPUMinimizerHit& a, const GPUMinimizerHit& b) {
                return a.minimizer_hash == b.minimizer_hash;
            });
        
        *final_count = new_end - hits_ptr;
        
        // Final error check
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error after unique: %s\n", hipGetErrorString(err));
            return false;
        }
        
    } catch (const std::exception& e) {
        printf("Error in deduplication: %s\n", e.what());
        return false;
    }
    
    return true;
}

// Test function to validate minimizer extraction with actual kernel launch
__global__ void test_minimizer_kernel(
    const char* test_sequence,
    int seq_length,
    MinimizerParams params,
    uint64_t* results,
    int* result_count) {
    
    int tid = threadIdx.x;
    if (tid != 0) return;
    
    uint64_t last_minimizer = UINT64_MAX;
    int count = 0;
    
    for (int kmer_pos = 0; kmer_pos <= seq_length - params.k; kmer_pos++) {
        uint64_t minimizer = extract_minimizer_sliding_window(
            test_sequence, kmer_pos, params.k, params.ell, params.spaces, params.xor_mask
        );
        
        if (minimizer != UINT64_MAX) {
            if (minimizer != last_minimizer && count < 100) {
                results[count] = minimizer;
                count++;
                last_minimizer = minimizer;
            }
        }
    }
    
    *result_count = count;
}

void test_minimizer_extraction() {
    printf("Testing minimizer extraction with actual GPU kernel...\n");
    
    // Test sequence
    const char* test_seq = "ATCGATCGATCGATCGATCGATCGATCGATCGATCGATCGATCGATCG";
    int seq_len = strlen(test_seq);
    
    printf("Test sequence: %s (length %d)\n", test_seq, seq_len);
    
    MinimizerParams params;
    params.k = 10;
    params.ell = 8;
    params.spaces = 0;
    params.xor_mask = 0;
    
    printf("Parameters: k=%d, ell=%d\n", params.k, params.ell);
    
    // Allocate GPU memory
    char* d_test_seq;
    uint64_t* d_results;
    int* d_result_count;
    
    hipMalloc(&d_test_seq, seq_len + 1);
    hipMalloc(&d_results, 100 * sizeof(uint64_t));
    hipMalloc(&d_result_count, sizeof(int));
    
    // Copy test data
    hipMemcpy(d_test_seq, test_seq, seq_len + 1, hipMemcpyHostToDevice);
    
    // Launch test kernel
    test_minimizer_kernel<<<1, 1>>>(d_test_seq, seq_len, params, d_results, d_result_count);
    hipDeviceSynchronize();
    
    // Get results
    uint64_t results[100];
    int result_count;
    hipMemcpy(results, d_results, 100 * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(&result_count, d_result_count, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Found %d unique minimizers:\n", result_count);
    for (int i = 0; i < result_count && i < 10; i++) {
        printf("  [%d] 0x%016lx\n", i, results[i]);
    }
    
    int expected_kmers = seq_len - params.k + 1;
    double compression = (double)result_count / expected_kmers;
    printf("Compression: %d/%d = %.3f (%.1fx reduction)\n", 
           result_count, expected_kmers, compression, 1.0/compression);
    
    // Cleanup
    hipFree(d_test_seq);
    hipFree(d_results);
    hipFree(d_result_count);
    
    printf("✓ Test completed\n");
}

#endif // GPU_MINIMIZER_EXTRACTION_CU
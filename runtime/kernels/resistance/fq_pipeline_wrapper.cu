#include "fq_mutation_detector.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

// Wrapper functions to call CUDA kernels from C++

extern "C" void launch_kmer_filter(
    const char* d_reads,
    const int* d_read_lengths,
    const int* d_read_offsets,
    FQMutationDetectorCUDA& detector,
    int num_reads,
    CandidateMatch* d_candidates,
    uint32_t* d_candidate_counts
) {
    dim3 block(256);
    dim3 grid((num_reads + block.x - 1) / block.x);
    
    kmer_filter_kernel<<<grid, block>>>(
        d_reads, d_read_lengths, d_read_offsets,
        detector.d_kmer_index, detector.d_kmer_sorted, detector.d_kmer_positions,
        detector.num_kmers, num_reads,
        d_candidates, d_candidate_counts, MAX_CANDIDATES_PER_READ
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in kmer_filter_kernel: " << hipGetErrorString(err) << std::endl;
    }
}

extern "C" void launch_position_weighted_alignment(
    const char* d_reads,
    const int* d_read_lengths,
    const int* d_read_offsets,
    const CandidateMatch* d_candidates,
    const uint32_t* d_candidate_counts,
    FQMutationDetectorCUDA& detector,
    int num_reads,
    AlignmentResult* d_results,
    uint32_t* d_result_count
) {
    dim3 block(128);
    dim3 grid((num_reads + block.x - 1) / block.x);
    size_t shared_mem_size = 4096;
    
    position_weighted_alignment_kernel<<<grid, block, shared_mem_size>>>(
        d_reads, d_read_lengths, d_read_offsets,
        d_candidates, d_candidate_counts, MAX_CANDIDATES_PER_READ,
        detector.d_reference_sequences, detector.d_ref_lengths, detector.d_ref_offsets,
        detector.d_position_weights, detector.d_mutation_masks,
        detector.d_mutation_info, detector.d_mutation_counts,
        detector.align_params, num_reads,
        d_results, d_result_count
    );
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error in position_weighted_alignment_kernel: " << hipGetErrorString(err) << std::endl;
    }
}
#include "hip/hip_runtime.h"
// translated_search.cu
// GPU-accelerated 6-frame translation and protein search for resistance detection
// Enhanced with 5-mer seeding, clustering, extension, and optional Smith-Waterman

#ifndef TRANSLATED_SEARCH_CU
#define TRANSLATED_SEARCH_CU

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>
#include <stdint.h>
#include <stdio.h>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <map>

namespace cg = cooperative_groups;

// Debug macros
#define DEBUG_TRANS 0
#define DEBUG_PRINT(fmt, ...) if(DEBUG_TRANS) { printf("[TRANS DEBUG] %s:%d: " fmt "\n", __FILE__, __LINE__, ##__VA_ARGS__); }

// Enhanced constants for 5-mer approach
#define CODON_SIZE 3
#define NUM_FRAMES 6
#define MAX_PROTEIN_LENGTH 200
#define PROTEIN_KMER_SIZE 5         // Increased from 3 to 5 amino acids
#define MIN_PEPTIDE_LENGTH 20       // Minimum peptide length to consider
#define MIN_SEED_HITS 1            // Require only one k-mer hit for extension
#define EXTENSION_THRESHOLD 15     // Minimum amino acids for valid match
#define MIN_IDENTITY_THRESHOLD 0.6f // Minimum 60% identity for valid protein match (lowered for more hits)
#define SW_SCORE_THRESHOLD 40.0f   // Lowered threshold for Smith-Waterman alignment to detect mutations
#define AA_ALPHABET_SIZE 24

// Genetic code table (standard code)
__constant__ char GENETIC_CODE[64] = {
    'K', 'N', 'K', 'N', 'T', 'T', 'T', 'T',  // AAA, AAC, AAG, AAT, ACA, ACC, ACG, ACT
    'R', 'S', 'R', 'S', 'I', 'I', 'M', 'I',  // AGA, AGC, AGG, AGT, ATA, ATC, ATG, ATT
    'Q', 'H', 'Q', 'H', 'P', 'P', 'P', 'P',  // CAA, CAC, CAG, CAT, CCA, CCC, CCG, CCT
    'R', 'R', 'R', 'R', 'L', 'L', 'L', 'L',  // CGA, CGC, CGG, CGT, CTA, CTC, CTG, CTT
    'E', 'D', 'E', 'D', 'A', 'A', 'A', 'A',  // GAA, GAC, GAG, GAT, GCA, GCC, GCG, GCT
    'G', 'G', 'G', 'G', 'V', 'V', 'V', 'V',  // GGA, GGC, GGG, GGT, GTA, GTC, GTG, GTT
    '*', 'Y', '*', 'Y', 'S', 'S', 'S', 'S',  // TAA, TAC, TAG, TAT, TCA, TCC, TCG, TCT
    'W', 'C', '*', 'C', 'L', 'F', 'L', 'F'   // TGA, TGC, TGG, TGT, TTA, TTC, TTG, TTT
};

// BLOSUM62 matrix (simplified, key values)
__constant__ float BLOSUM62_SCORES[24*24] = {
    // A   R   N   D   C   Q   E   G   H   I   L   K   M   F   P   S   T   W   Y   V   B   Z   X   *
    4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4, // A
   -1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4, // R
   -2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4, // N
   -2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4, // D
    0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4, // C
   -1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4, // Q
   -1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4, // E
    0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4, // G
   -2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4, // H
   -1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4, // I
   -1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4, // L
   -1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4, // K
   -1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4, // M
   -2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4, // F
   -1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4, // P
    1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4, // S
    0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4, // T
   -3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4, // W
   -2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4, // Y
    0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4, // V
   -2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4, // B
   -1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4, // Z
    0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4, // X
   -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1  // *
};

// Amino acid to index mapping
__host__ __device__ inline int aa_to_index(char aa) {
    switch(aa) {
        case 'A': return 0;  case 'R': return 1;  case 'N': return 2;  case 'D': return 3;
        case 'C': return 4;  case 'Q': return 5;  case 'E': return 6;  case 'G': return 7;
        case 'H': return 8;  case 'I': return 9;  case 'L': return 10; case 'K': return 11;
        case 'M': return 12; case 'F': return 13; case 'P': return 14; case 'S': return 15;
        case 'T': return 16; case 'W': return 17; case 'Y': return 18; case 'V': return 19;
        case 'B': return 20; case 'Z': return 21; case 'X': return 22; case '*': return 23;
        default: return 22; // Unknown -> X
    }
}

// helper function to check if alignment covers QRDR regions
// Updated to use correct gene_ids from the database
__device__ bool covers_qrdr_region(uint32_t gene_id, uint16_t ref_start, uint16_t match_length) {
    uint16_t ref_end = ref_start + match_length;
    
    // Based on protein_details.json:
    // gene_id 1 = gyrA
    // gene_id 2 = parC
    // gene_id 0 = grlA (S. aureus equivalent of gyrA)
    
    if (gene_id == 1) {  // gyrA
        // gyrA QRDR positions (0-based): S83 (pos 82) and D87 (pos 86)
        return (ref_start <= 82 && ref_end > 82) || (ref_start <= 86 && ref_end > 86);
    } else if (gene_id == 2) {  // parC
        // parC QRDR positions (0-based): S80 (pos 79) and E84 (pos 83)
        return (ref_start <= 79 && ref_end > 79) || (ref_start <= 83 && ref_end > 83);
    } else if (gene_id == 0) {  // grlA (S. aureus)
        // grlA QRDR positions similar to gyrA: S84 (pos 83) and E88 (pos 87)
        return (ref_start <= 83 && ref_end > 83) || (ref_start <= 87 && ref_end > 87);
    }
    
    return false;
}


// Get BLOSUM score
__device__ inline float get_blosum_score(char aa1, char aa2) {
    int idx1 = aa_to_index(aa1);
    int idx2 = aa_to_index(aa2);
    return BLOSUM62_SCORES[idx1 * 24 + idx2];
}

// Base encoding for translation
__device__ inline int base_to_index(char base) {
    switch(base) {
        case 'A': case 'a': return 0;
        case 'C': case 'c': return 1;
        case 'G': case 'g': return 2;
        case 'T': case 't': case 'U': case 'u': return 3;
        default: return -1;  // Invalid base
    }
}

// Translate codon to amino acid
__device__ inline char translate_codon(const char* codon) {
    int idx = 0;
    for (int i = 0; i < 3; i++) {
        int base = base_to_index(codon[i]);
        if (base < 0) return 'X';  // Unknown
        idx = (idx << 2) | base;
    }
    return GENETIC_CODE[idx];
}

// Optimized hash function for 5-mer protein k-mers
__device__ inline uint64_t hash_protein_5mer(const char* kmer) {
    uint64_t hash = 0;
    const uint64_t prime = 31;
    
    for (int i = 0; i < PROTEIN_KMER_SIZE; i++) {
        // Map amino acid to value preserving chemical properties
        uint8_t aa_val = aa_to_index(kmer[i]);
        hash = hash * prime + aa_val;
    }
    
    return hash;
}

// Structure for translated read
struct TranslatedFrame {
    char sequence[MAX_PROTEIN_LENGTH];
    uint16_t length;
    int8_t frame;  // -3, -2, -1, +1, +2, +3
    uint16_t start_pos;  // Start position in original read
};

// Enhanced structure for protein match with mutation details
struct ProteinMatch {
    uint32_t read_id;
    int8_t frame;
    uint32_t protein_id;
    uint32_t gene_id;
    uint32_t species_id;
    uint16_t query_start;    // Position in translated frame
    uint16_t ref_start;      // Position in reference protein
    uint16_t match_length;
    float alignment_score;
    float identity;
    uint8_t num_mutations;
    uint8_t mutation_positions[10];  // Up to 10 mutations
    char ref_aas[10];
    char query_aas[10];
    float blosum_scores[10];
    bool used_smith_waterman;  // Flag indicating if SW was used
    char query_peptide[51];  // Store aligned peptide sequence (up to 50 AA + null terminator)
    bool is_qrdr_alignment;  // Flag for QRDR region alignment
};

// Enhanced protein database structure with sorted k-mer index
struct ProteinDatabase {
    uint32_t num_proteins;
    uint32_t num_kmers;
    
    // Sorted k-mer index for binary search
    uint64_t* sorted_kmer_hashes;
    uint32_t* kmer_start_indices;  // Start index in position array
    uint32_t* kmer_counts;         // Number of positions per k-mer
    uint32_t* position_data;       // Encoded protein ID + position
    
    // Protein metadata
    uint32_t* protein_ids;
    uint32_t* gene_ids;
    uint32_t* species_ids;
    uint16_t* seq_lengths;
    
    // Reference sequences for Smith-Waterman
    char* sequences;
    uint32_t* seq_offsets;
};

// Helper structure for seed clustering
struct SeedHit {
    uint32_t protein_id;
    uint32_t query_pos;
    uint32_t ref_pos;
    float score;
};

// Binary search for protein k-mer
__device__ inline int binary_search_protein_kmer(
    const ProteinDatabase* db,
    uint64_t target_hash
) {
    int left = 0;
    int right = db->num_kmers - 1;
    
    while (left <= right) {
        int mid = (left + right) / 2;
        uint64_t mid_hash = db->sorted_kmer_hashes[mid];
        
        if (mid_hash == target_hash) {
            return mid;
        } else if (mid_hash < target_hash) {
            left = mid + 1;
        } else {
            right = mid - 1;
        }
    }
    
    return -1; // Not found
}

// Smith-Waterman local alignment (simplified for GPU)
__device__ float smith_waterman_align(
    const char* query, uint16_t query_len,
    const char* ref, uint16_t ref_len,
    uint16_t* best_query_start,
    uint16_t* best_ref_start,
    uint16_t* best_length,
    uint8_t* mutations,
    char* ref_mutations,
    char* query_mutations,
    float* blosum_mutations,
    uint8_t* num_mutations
) {
    // Simplified banded Smith-Waterman for GPU (more lenient scoring)
    const int GAP_OPEN = -2;
    const int GAP_EXTEND = -1;
    const int BAND_WIDTH = 10;
    
    // Use local arrays for alignments - increased to handle full 150bp reads (50 AA)
    // For banded alignment, we only need a strip of the matrix
    const int MAX_ALIGN_LEN = 60; // Support up to 60 AA (180 bp)
    float H[MAX_ALIGN_LEN][MAX_ALIGN_LEN] = {0}; // Score matrix
    
    if (query_len > MAX_ALIGN_LEN-1 || ref_len > MAX_ALIGN_LEN-1) {
        // Fall back to simple scoring for very large sequences
        float score = 0.0f;
        int matches = 0;
        int aligned_len = min(query_len, ref_len);
        
        for (int i = 0; i < aligned_len; i++) {
            float blosum = get_blosum_score(query[i], ref[i]);
            score += blosum;
            if (blosum > 0) matches++;
        }
        
        *best_query_start = 0;
        *best_ref_start = 0;
        *best_length = aligned_len;
        *num_mutations = 0;
        
        return score;
    }
    
    float max_score = 0.0f;
    int max_i = 0, max_j = 0;
    
    // Fill scoring matrix with banding optimization
    // Since we expect high similarity, only compute cells near the diagonal
    for (int i = 1; i <= query_len; i++) {
        // Banded alignment: only fill cells within BAND_WIDTH of diagonal
        int j_start = max(1, i - BAND_WIDTH);
        int j_end = min((int)ref_len, i + BAND_WIDTH);
        
        for (int j = j_start; j <= j_end; j++) {
            float match = H[i-1][j-1] + get_blosum_score(query[i-1], ref[j-1]);
            float delete_gap = (j > 1) ? H[i-1][j] + GAP_OPEN : GAP_OPEN;
            float insert_gap = (i > 1) ? H[i][j-1] + GAP_OPEN : GAP_OPEN;
            
            H[i][j] = fmaxf(0.0f, fmaxf(match, fmaxf(delete_gap, insert_gap)));
            
            if (H[i][j] > max_score) {
                max_score = H[i][j];
                max_i = i;
                max_j = j;
            }
        }
    }
    
    // Traceback to find alignment
    int i = max_i, j = max_j;
    int align_len = 0;
    uint8_t mut_count = 0;
    
    while (i > 0 && j > 0 && H[i][j] > 0 && align_len < MAX_ALIGN_LEN-1) {
        if (i > 0 && j > 0 && H[i][j] == H[i-1][j-1] + get_blosum_score(query[i-1], ref[j-1])) {
            // Match or mismatch
            if (query[i-1] != ref[j-1] && mut_count < 10) {
                mutations[mut_count] = j-1; // Reference position
                ref_mutations[mut_count] = ref[j-1];
                query_mutations[mut_count] = query[i-1];
                blosum_mutations[mut_count] = get_blosum_score(query[i-1], ref[j-1]);
                mut_count++;
            }
            i--; j--;
        } else if (i > 0 && H[i][j] == H[i-1][j] + GAP_OPEN) {
            // Deletion in reference
            i--;
        } else if (j > 0 && H[i][j] == H[i][j-1] + GAP_OPEN) {
            // Insertion in reference  
            j--;
        } else {
            break;
        }
        align_len++;
    }
    
    *best_query_start = i;
    *best_ref_start = j;
    *best_length = align_len;
    *num_mutations = mut_count;
    
    return max_score;
}

// 6-frame translation kernel (unchanged)
__global__ void six_frame_translate_kernel(
    const char* reads,
    const int* read_lengths,
    const int* read_offsets,
    const int num_reads,
    TranslatedFrame* translated_frames,
    uint32_t* frame_counts
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_reads) return;
    
    const char* read = reads + read_offsets[tid];
    const int read_len = read_lengths[tid];
    
    TranslatedFrame* read_frames = &translated_frames[tid * NUM_FRAMES];
    int valid_frames = 0;
    
    // Forward frames (+1, +2, +3)
    for (int frame = 0; frame < 3; frame++) {
        TranslatedFrame& tf = read_frames[valid_frames];
        tf.frame = frame + 1;
        tf.start_pos = frame;
        tf.length = 0;
        
        for (int pos = frame; pos + 2 < read_len; pos += 3) {
            if (tf.length >= MAX_PROTEIN_LENGTH - 1) break;
            
            char aa = translate_codon(&read[pos]);
            if (aa == '*') {
                if (tf.length >= MIN_PEPTIDE_LENGTH) {
                    tf.sequence[tf.length] = '\0';
                    valid_frames++;
                    
                    if (valid_frames < NUM_FRAMES) {
                        tf = read_frames[valid_frames];
                        tf.frame = frame + 1;
                        tf.start_pos = pos + 3;
                        tf.length = 0;
                    }
                } else {
                    tf.length = 0;
                    tf.start_pos = pos + 3;
                }
            } else {
                tf.sequence[tf.length++] = aa;
            }
        }
        
        if (tf.length >= MIN_PEPTIDE_LENGTH) {
            tf.sequence[tf.length] = '\0';
            valid_frames++;
        }
    }
    
    // Reverse complement frames (-1, -2, -3)
    for (int frame = 0; frame < 3 && valid_frames < NUM_FRAMES; frame++) {
        TranslatedFrame& tf = read_frames[valid_frames];
        tf.frame = -(frame + 1);
        tf.start_pos = read_len - frame - 1;
        tf.length = 0;
        
        for (int pos = read_len - frame - 1; pos >= 2; pos -= 3) {
            if (tf.length >= MAX_PROTEIN_LENGTH - 1) break;
            
            char rc_codon[3];
            for (int i = 0; i < 3; i++) {
                char base = read[pos - i];
                switch(base) {
                    case 'A': case 'a': rc_codon[i] = 'T'; break;
                    case 'T': case 't': rc_codon[i] = 'A'; break;
                    case 'G': case 'g': rc_codon[i] = 'C'; break;
                    case 'C': case 'c': rc_codon[i] = 'G'; break;
                    default: rc_codon[i] = 'N';
                }
            }
            
            char aa = translate_codon(rc_codon);
            if (aa == '*') {
                if (tf.length >= MIN_PEPTIDE_LENGTH) {
                    tf.sequence[tf.length] = '\0';
                    valid_frames++;
                    
                    if (valid_frames < NUM_FRAMES) {
                        tf = read_frames[valid_frames];
                        tf.frame = -(frame + 1);
                        tf.start_pos = pos - 3;
                        tf.length = 0;
                    }
                } else {
                    tf.length = 0;
                    tf.start_pos = pos - 3;
                }
            } else {
                tf.sequence[tf.length++] = aa;
            }
        }
        
        if (tf.length >= MIN_PEPTIDE_LENGTH) {
            tf.sequence[tf.length] = '\0';
            valid_frames++;
        }
    }
    
    frame_counts[tid] = valid_frames;
    
    if (tid == 0 && DEBUG_TRANS) {
        DEBUG_PRINT("Read 0: %d valid frames from %d bp", valid_frames, read_len);
    }
}

// Enhanced protein k-mer matching kernel with 5-mer seeding and extension
__global__ void enhanced_protein_kmer_match_kernel(
    const TranslatedFrame* translated_frames,
    const uint32_t* frame_counts,
    const int num_reads,
    const ProteinDatabase* protein_db,
    ProteinMatch* matches,
    uint32_t* match_counts,
    const uint32_t max_matches_per_read,
    const bool enable_smith_waterman = false
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num_reads) return;
    
    uint32_t num_frames = frame_counts[tid];
    if (num_frames == 0) {
        match_counts[tid] = 0;
        return;
    }
    
    const TranslatedFrame* read_frames = &translated_frames[tid * NUM_FRAMES];
    ProteinMatch* read_matches = &matches[tid * max_matches_per_read];
    uint32_t match_count = 0;
    
    // For each translated frame
    for (uint32_t frame_idx = 0; frame_idx < num_frames; frame_idx++) {
        const TranslatedFrame& frame = read_frames[frame_idx];
        
        if (frame.length < PROTEIN_KMER_SIZE) continue;
        
        // Collect 5-mer seed hits
        SeedHit seeds[100];
        int num_seeds = 0;
        
        // Find 5-mer seed matches
        for (int pos = 0; pos + PROTEIN_KMER_SIZE <= frame.length && num_seeds < 100; pos++) {
            uint64_t kmer_hash = hash_protein_5mer(&frame.sequence[pos]);
            
            int kmer_idx = binary_search_protein_kmer(protein_db, kmer_hash);
            if (kmer_idx >= 0) {
                uint32_t start_idx = protein_db->kmer_start_indices[kmer_idx];
                uint32_t count = protein_db->kmer_counts[kmer_idx];
                
                // Add hits for this k-mer (limit to avoid overflow)
                for (uint32_t i = 0; i < count && i < 5 && num_seeds < 100; i++) {
                    uint32_t encoded = protein_db->position_data[start_idx + i];
                    uint32_t protein_id = encoded >> 16;
                    uint32_t ref_pos = encoded & 0xFFFF;
                    
                    seeds[num_seeds] = {protein_id, (uint32_t)pos, ref_pos, 10.0f};
                    num_seeds++;
                }
            }
        }
        
        // Cluster seeds by protein and extend
        for (int s = 0; s < num_seeds && match_count < max_matches_per_read; s++) {
            uint32_t protein_id = seeds[s].protein_id;
            if (protein_id == UINT32_MAX) continue; // Already processed
            
            // Collect all seeds for this protein
            SeedHit protein_seeds[20];
            int seed_count = 0;
            
            for (int t = s; t < num_seeds && seed_count < 20; t++) {
                if (seeds[t].protein_id == protein_id) {
                    protein_seeds[seed_count++] = seeds[t];
                    seeds[t].protein_id = UINT32_MAX; // Mark as used
                }
            }
            
            if (seed_count < MIN_SEED_HITS) continue;
            
            // Safe extension from first seed - extend while maintaining alignment
            uint32_t seed_query_pos = protein_seeds[0].query_pos;
            uint32_t seed_ref_pos = protein_seeds[0].ref_pos;
            
            // Get reference sequence for this protein
            const char* ref_seq = &protein_db->sequences[protein_db->seq_offsets[protein_id]];
            uint16_t ref_len = protein_db->seq_lengths[protein_id];
            
            // Extend left from seed (allow some mismatches)
            int left_extend = 0;
            int left_mismatches = 0;
            const int max_mismatches = 2;  // Allow up to 2 mismatches during extension
            
            // Arrays to store mutation information
            uint8_t mutation_positions[10];
            char ref_aas[10];
            char query_aas[10];
            int num_mutations = 0;
            
            while (seed_query_pos - left_extend > 0 && 
                   seed_ref_pos - left_extend > 0 &&
                   left_extend < 20 &&
                   left_mismatches <= max_mismatches) {
                char query_aa = frame.sequence[seed_query_pos - left_extend - 1];
                char ref_aa = ref_seq[seed_ref_pos - left_extend - 1];
                
                if (query_aa == ref_aa) {
                    left_extend++;
                } else {
                    // Record mutation details
                    if (num_mutations < 10) {
                        mutation_positions[num_mutations] = left_extend - 1;
                        ref_aas[num_mutations] = ref_aa;
                        query_aas[num_mutations] = query_aa;
                        num_mutations++;
                    }
                    left_mismatches++;
                    left_extend++;  // Continue extending despite mismatch
                }
            }
            
            // Extend right from seed (allow some mismatches)
            int right_extend = PROTEIN_KMER_SIZE;  // Start after the k-mer
            int right_mismatches = 0;
            
            while (seed_query_pos + right_extend < frame.length && 
                   seed_ref_pos + right_extend < ref_len &&
                   right_extend < 50 &&
                   right_mismatches <= max_mismatches) {
                char query_aa = frame.sequence[seed_query_pos + right_extend];
                char ref_aa = ref_seq[seed_ref_pos + right_extend];
                
                if (query_aa == ref_aa) {
                    right_extend++;
                } else {
                    // Record mutation details
                    if (num_mutations < 10) {
                        mutation_positions[num_mutations] = seed_query_pos - (seed_query_pos - left_extend) + right_extend;
                        ref_aas[num_mutations] = ref_aa;
                        query_aas[num_mutations] = query_aa;
                        num_mutations++;
                    }
                    right_mismatches++;
                    right_extend++;  // Continue extending despite mismatch
                }
            }
            
            uint32_t min_query = seed_query_pos - left_extend;
            uint32_t min_ref = seed_ref_pos - left_extend;
            uint32_t query_span = left_extend + right_extend;
            uint32_t ref_span = query_span;  // Same span due to exact extension
            
            // Always proceed if we have enough seed hits
            if (seed_count >= MIN_SEED_HITS) {
                
                // Create temporary match to check identity
                ProteinMatch temp_match;
                temp_match.read_id = tid;
                temp_match.frame = frame.frame;
                temp_match.protein_id = protein_id;
                temp_match.gene_id = protein_db->gene_ids[protein_id];
                temp_match.species_id = protein_db->species_ids[protein_id];
                temp_match.query_start = min_query;
                temp_match.ref_start = min_ref;
                temp_match.match_length = query_span;  // Use the extended length
                
                // Calculate identity based on mismatches found during extension
                int total_mismatches = left_mismatches + right_mismatches;
                temp_match.identity = 1.0f - (float)total_mismatches / query_span;
                temp_match.alignment_score = query_span * 2.0f - total_mismatches * 4.0f;  // Penalize mismatches
                temp_match.num_mutations = num_mutations;
                temp_match.used_smith_waterman = false;
                
                // Copy mutation details
                for (int i = 0; i < num_mutations && i < 10; i++) {
                    temp_match.mutation_positions[i] = mutation_positions[i];
                    temp_match.ref_aas[i] = ref_aas[i];
                    temp_match.query_aas[i] = query_aas[i];
                }
                
                // Initialize new fields
                temp_match.is_qrdr_alignment = false;  // Will be set on host side
                // Extract peptide sequence from translated frame (extended region)
                int peptide_len = min(query_span, 50);
                for (int k = 0; k < peptide_len && k < 50; k++) {
                    if (min_query + k < frame.length) {
                        temp_match.query_peptide[k] = frame.sequence[min_query + k];
                    } else {
                        temp_match.query_peptide[k] = 'X';
                    }
                }
                temp_match.query_peptide[peptide_len] = '\0';
                
                // Debug: Print scoring info for first few matches
                if (tid == 0 && match_count < 3 && DEBUG_TRANS) {
                    DEBUG_PRINT("Match %d: seed_count=%d, score=%.1f, threshold=%.1f, SW enabled=%s", 
                               match_count, seed_count, temp_match.alignment_score, SW_SCORE_THRESHOLD, enable_smith_waterman ? "YES" : "NO");
                }
                
                // Apply Smith-Waterman to extend alignments and find more mutations
                // Run SW for all matches to get better extension (not just high-scoring ones)
                if (enable_smith_waterman && seed_count >= MIN_SEED_HITS) {
                    if (tid == 0 && match_count < 3 && DEBUG_TRANS) {
                        DEBUG_PRINT("Triggering Smith-Waterman: protein_id=%d, score=%.1f", protein_id, temp_match.alignment_score);
                    }
                    
                    // Get reference sequence
                    const char* ref_seq = &protein_db->sequences[protein_db->seq_offsets[protein_id]];
                    uint16_t ref_len = protein_db->seq_lengths[protein_id];
                    
                    if (temp_match.ref_start < ref_len) {
                        // Use the already extended region for Smith-Waterman
                        // Allow SW to see the full remaining sequence (up to 60 AA)
                        uint16_t available_ref = ref_len - temp_match.ref_start;
                        uint16_t available_query = frame.length - temp_match.query_start;
                        uint16_t sw_ref_len = min(available_ref, (uint16_t)60);
                        uint16_t sw_query_len = min(available_query, (uint16_t)60);
                        
                        uint16_t sw_query_start, sw_ref_start, sw_length;
                        uint8_t sw_num_mutations;
                        
                        // Debug: print sequences being aligned (first few chars)
                        if (tid == 0 && DEBUG_TRANS) {
                            DEBUG_PRINT("SW input: query_len=%d, ref_len=%d, protein_id=%d", 
                                    sw_query_len, sw_ref_len, protein_id);
                            DEBUG_PRINT("Query offset=%d: %.15s", temp_match.query_start, &frame.sequence[temp_match.query_start]);
                            DEBUG_PRINT("Ref offset=%d: %.15s", temp_match.ref_start, &ref_seq[temp_match.ref_start]);
                            DEBUG_PRINT("Ref full seq (first 20): %.20s", ref_seq);
                        }
                        
                        float sw_score = smith_waterman_align(
                            &frame.sequence[temp_match.query_start], sw_query_len,
                            &ref_seq[temp_match.ref_start], sw_ref_len,
                            &sw_query_start, &sw_ref_start, &sw_length,
                            temp_match.mutation_positions,
                            temp_match.ref_aas,
                            temp_match.query_aas,
                            temp_match.blosum_scores,
                            &sw_num_mutations
                        );
                        
                        // ADD DETAILED DEBUG OUTPUT HERE FOR MUTATION DETECTION
                        if (sw_score > temp_match.alignment_score) {
                            temp_match.alignment_score = sw_score;
                            temp_match.query_start += sw_query_start;
                            temp_match.ref_start += sw_ref_start;
                            temp_match.match_length = sw_length;
                            temp_match.num_mutations = sw_num_mutations;
                            temp_match.identity = (float)(sw_length - sw_num_mutations) / sw_length;
                            temp_match.used_smith_waterman = true;
                            
                            // DEBUG: Add detailed mutation analysis
                            if (tid < 5 && sw_num_mutations > 0) {  // First 5 reads with mutations
                                printf("\n[MUTATION DEBUG] Read %d, Frame %d, Gene %d:\n", 
                                    tid, frame.frame, temp_match.gene_id);
                                printf("  Alignment: Query pos %d-%d → Ref pos %d-%d\n",
                                    temp_match.query_start, temp_match.query_start + sw_length,
                                    temp_match.ref_start, temp_match.ref_start + sw_length);
                                printf("  Identity: %.1f%% (%d mutations in %d AA)\n",
                                    temp_match.identity * 100, sw_num_mutations, sw_length);
                                
                                // Check for QRDR positions
                                // gyrA: positions 83, 87
                                // parC: positions 80, 84
                                bool has_qrdr_mutation = false;
                                
                                for (int i = 0; i < sw_num_mutations; i++) {
                                    int mutation_pos_in_protein = temp_match.ref_start + temp_match.mutation_positions[i];
                                    
                                    printf("  Mutation %d: Position %d, %c → %c (BLOSUM: %.1f)\n",
                                        i + 1, mutation_pos_in_protein,
                                        temp_match.ref_aas[i], temp_match.query_aas[i],
                                        temp_match.blosum_scores[i]);
                                    
                                    // Check if this is a QRDR position
                                    if (temp_match.gene_id == 1) {  // gyrA (corrected gene_id)
                                        if (mutation_pos_in_protein == 82) {  // 0-based, so 83-1
                                            printf("    *** QRDR MUTATION: gyrA S83%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        } else if (mutation_pos_in_protein == 86) {  // 87-1
                                            printf("    *** QRDR MUTATION: gyrA D87%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        }
                                    } else if (temp_match.gene_id == 2) {  // parC (corrected gene_id)
                                        if (mutation_pos_in_protein == 79) {  // 80-1
                                            printf("    *** QRDR MUTATION: parC S80%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        } else if (mutation_pos_in_protein == 83) {  // 84-1
                                            printf("    *** QRDR MUTATION: parC E84%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        }
                                    } else if (temp_match.gene_id == 0) {  // grlA (S. aureus)
                                        if (mutation_pos_in_protein == 83) {  // S84 (0-based)
                                            printf("    *** QRDR MUTATION: grlA S84%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        } else if (mutation_pos_in_protein == 87) {  // E88
                                            printf("    *** QRDR MUTATION: grlA E88%c ***\n", temp_match.query_aas[i]);
                                            has_qrdr_mutation = true;
                                        }
                                    }
                                }
                                
                                if (!has_qrdr_mutation && sw_num_mutations > 0) {
                                    printf("  Note: Mutations found but NOT in QRDR positions\n");
                                }
                                
                                // Show the actual alignment region
                                printf("  Query peptide: ");
                                for (int j = 0; j < min(sw_length, 30); j++) {
                                    printf("%c", frame.sequence[temp_match.query_start + j]);
                                }
                                printf("\n");
                                
                                printf("  Ref peptide:   ");
                                for (int j = 0; j < min(sw_length, 30); j++) {
                                    printf("%c", ref_seq[temp_match.ref_start + j]);
                                }
                                printf("\n\n");
                            }
                        }
                    }
                }
                                
                // Only accept matches with sufficient identity
                if (temp_match.identity >= MIN_IDENTITY_THRESHOLD) {
                    read_matches[match_count] = temp_match;
                    match_count++;
                }
            }
        }
    }
    
    // Keep only the best scoring match (unless there are ties)
    if (match_count > 1) {
        float max_score = read_matches[0].alignment_score;
        int best_idx = 0;
        
        // Find the maximum score
        for (uint32_t i = 1; i < match_count; i++) {
            if (read_matches[i].alignment_score > max_score) {
                max_score = read_matches[i].alignment_score;
                best_idx = i;
            }
        }
        
        // Count how many matches have the same max score
        int ties = 0;
        for (uint32_t i = 0; i < match_count; i++) {
            if (read_matches[i].alignment_score == max_score) {
                ties++;
            }
        }
        
        // If there's a single best match, keep only that one
        if (ties == 1) {
            if (best_idx != 0) {
                read_matches[0] = read_matches[best_idx];
            }
            match_count = 1;
        } else {
            // Keep all matches with the max score
            uint32_t kept = 0;
            for (uint32_t i = 0; i < match_count; i++) {
                if (read_matches[i].alignment_score == max_score) {
                    if (kept != i) {
                        read_matches[kept] = read_matches[i];
                    }
                    kept++;
                }
            }
            match_count = kept;
        }
    }
    
    match_counts[tid] = match_count;
    
    if (tid == 0 && match_count > 0 && DEBUG_TRANS) {
        DEBUG_PRINT("Read 0: %d protein matches found (SW enabled: %s)", 
                   match_count, enable_smith_waterman ? "YES" : "NO");
    }
    
    // Add summary statistics at the end of the kernel
    if (tid == 0) {
        // Count how many reads had QRDR coverage
        __shared__ int qrdr_coverage_count;
        __shared__ int qrdr_mutation_count;
        
        if (threadIdx.x == 0) {
            qrdr_coverage_count = 0;
            qrdr_mutation_count = 0;
        }
        __syncthreads();
        
        // Each thread checks its matches
        for (uint32_t i = 0; i < match_count; i++) {
            if (read_matches[i].used_smith_waterman) {
                if (covers_qrdr_region(read_matches[i].gene_id, 
                                      read_matches[i].ref_start, 
                                      read_matches[i].match_length)) {
                    atomicAdd(&qrdr_coverage_count, 1);
                    
                    // Check if any mutations are in QRDR positions
                    for (int j = 0; j < read_matches[i].num_mutations; j++) {
                        int pos = read_matches[i].ref_start + read_matches[i].mutation_positions[j];
                        // Check for mutations at key QRDR positions (works for any gene)
                        // gyrA: S83 (pos 82) and D87 (pos 86)
                        // parC: S80 (pos 79) and E84 (pos 83)
                        if (pos == 82 || pos == 86 || pos == 79 || pos == 83) {
                            atomicAdd(&qrdr_mutation_count, 1);
                            break;
                        }
                    }
                }
            }
        }
        
        __syncthreads();
        if (threadIdx.x == 0) {
            printf("[SUMMARY] QRDR coverage: %d alignments, QRDR mutations: %d\n", 
                   qrdr_coverage_count, qrdr_mutation_count);
        }
    }
}

// Host wrapper class
class TranslatedSearchEngine {
private:
    ProteinDatabase* d_protein_db;
    TranslatedFrame* d_translated_frames;
    uint32_t* d_frame_counts;
    ProteinMatch* d_matches;
    uint32_t* d_match_counts;
    
    int max_batch_size;
    bool smith_waterman_enabled;
    
public:
    TranslatedSearchEngine(int batch_size = 10000, bool enable_sw = false) 
        : max_batch_size(batch_size), smith_waterman_enabled(enable_sw) {
        
        DEBUG_PRINT("Initializing TranslatedSearchEngine (batch=%d, SW=%s)", 
                   batch_size, enable_sw ? "enabled" : "disabled");
        
        hipError_t err;
        err = hipMalloc(&d_translated_frames, batch_size * NUM_FRAMES * sizeof(TranslatedFrame));
        if (err != hipSuccess) {
            DEBUG_PRINT("Failed to allocate d_translated_frames: %s", hipGetErrorString(err));
        }
        err = hipMalloc(&d_frame_counts, batch_size * sizeof(uint32_t));
        if (err != hipSuccess) {
            DEBUG_PRINT("Failed to allocate d_frame_counts: %s", hipGetErrorString(err));
        }
        err = hipMalloc(&d_matches, batch_size * 32 * sizeof(ProteinMatch));
        if (err != hipSuccess) {
            DEBUG_PRINT("Failed to allocate d_matches: %s", hipGetErrorString(err));
        }
        err = hipMalloc(&d_match_counts, batch_size * sizeof(uint32_t));
        if (err != hipSuccess) {
            DEBUG_PRINT("Failed to allocate d_match_counts: %s", hipGetErrorString(err));
        }
        
        d_protein_db = nullptr;
    }
    
    ~TranslatedSearchEngine() {
        if (d_translated_frames) hipFree(d_translated_frames);
        if (d_frame_counts) hipFree(d_frame_counts);
        if (d_matches) hipFree(d_matches);
        if (d_match_counts) hipFree(d_match_counts);
        
        if (d_protein_db) {
            ProteinDatabase h_db;
            hipMemcpy(&h_db, d_protein_db, sizeof(ProteinDatabase), hipMemcpyDeviceToHost);
            
            if (h_db.sorted_kmer_hashes) hipFree(h_db.sorted_kmer_hashes);
            if (h_db.kmer_start_indices) hipFree(h_db.kmer_start_indices);
            if (h_db.kmer_counts) hipFree(h_db.kmer_counts);
            if (h_db.position_data) hipFree(h_db.position_data);
            if (h_db.protein_ids) hipFree(h_db.protein_ids);
            if (h_db.gene_ids) hipFree(h_db.gene_ids);
            if (h_db.species_ids) hipFree(h_db.species_ids);
            if (h_db.seq_lengths) hipFree(h_db.seq_lengths);
            if (h_db.sequences) hipFree(h_db.sequences);
            if (h_db.seq_offsets) hipFree(h_db.seq_offsets);
            
            hipFree(d_protein_db);
        }
    }
    
    bool loadProteinDatabase(const std::string& db_path) {
        DEBUG_PRINT("Loading enhanced protein database from %s", db_path.c_str());
        
        if (d_protein_db) {
            DEBUG_PRINT("Protein database already loaded");
            return true;
        }
        
        // Load k-mer index
        std::string kmer_path = db_path + "/protein_kmers.bin";
        std::ifstream kmer_file(kmer_path, std::ios::binary);
        if (!kmer_file.good()) {
            DEBUG_PRINT("ERROR: Cannot read k-mer file: %s", kmer_path.c_str());
            return false;
        }
        
        uint32_t kmer_length, num_kmers;
        kmer_file.read(reinterpret_cast<char*>(&kmer_length), sizeof(uint32_t));
        kmer_file.read(reinterpret_cast<char*>(&num_kmers), sizeof(uint32_t));
        
        if (kmer_length != PROTEIN_KMER_SIZE) {
            DEBUG_PRINT("ERROR: K-mer size mismatch: expected %d, got %d", PROTEIN_KMER_SIZE, kmer_length);
            return false;
        }
        
        DEBUG_PRINT("Loading %d protein 5-mers", num_kmers);
        
        // Load and sort k-mers
        std::map<uint64_t, std::vector<uint32_t>> kmer_map;
        
        for (uint32_t i = 0; i < num_kmers; i++) {
            char kmer_seq[6] = {0};
            kmer_file.read(kmer_seq, kmer_length);
            
            uint64_t hash = 0;
            const uint64_t prime = 31;
            for (int j = 0; j < kmer_length; j++) {
                hash = hash * prime + aa_to_index(kmer_seq[j]);
            }
            
            uint32_t num_positions;
            kmer_file.read(reinterpret_cast<char*>(&num_positions), sizeof(uint32_t));
            
            for (uint32_t j = 0; j < num_positions; j++) {
                uint32_t protein_idx, position;
                kmer_file.read(reinterpret_cast<char*>(&protein_idx), sizeof(uint32_t));
                kmer_file.read(reinterpret_cast<char*>(&position), sizeof(uint32_t));
                
                uint32_t encoded = (protein_idx << 16) | (position & 0xFFFF);
                kmer_map[hash].push_back(encoded);
            }
        }
        kmer_file.close();
        
        // Create sorted arrays
        std::vector<uint64_t> sorted_hashes;
        std::vector<uint32_t> start_indices;
        std::vector<uint32_t> kmer_counts;
        std::vector<uint32_t> position_data;
        
        for (const auto& pair : kmer_map) {
            sorted_hashes.push_back(pair.first);
            start_indices.push_back(position_data.size());
            kmer_counts.push_back(pair.second.size());
            
            for (uint32_t pos : pair.second) {
                position_data.push_back(pos);
            }
        }
        
        // Load protein sequences for Smith-Waterman
        std::string protein_path = db_path + "/proteins.bin";
        std::ifstream protein_file(protein_path, std::ios::binary);
        if (!protein_file.good()) {
            DEBUG_PRINT("ERROR: Cannot read protein file: %s", protein_path.c_str());
            return false;
        }
        
        uint32_t num_proteins;
        protein_file.read(reinterpret_cast<char*>(&num_proteins), sizeof(uint32_t));
        
        DEBUG_PRINT("Reading %d proteins from database", num_proteins);
        
        // Read all remaining data as one big sequence block
        protein_file.seekg(0, std::ios::end);
        size_t file_size = protein_file.tellg();
        protein_file.seekg(sizeof(uint32_t), std::ios::beg); // Skip num_proteins
        
        size_t remaining_size = file_size - sizeof(uint32_t);
        std::vector<char> all_sequences(remaining_size + 1);
        protein_file.read(all_sequences.data(), remaining_size);
        all_sequences[remaining_size] = '\0';
        
        DEBUG_PRINT("Read %zu bytes of sequence data", remaining_size);
        printf("[DATABASE DEBUG] Loading protein database from: %s\n", db_path.c_str());
        printf("[DATABASE DEBUG] Number of proteins: %d\n", num_proteins);
        printf("[DATABASE DEBUG] Total sequence bytes: %zu\n", remaining_size);
        
        // Load protein metadata from protein_details.json
        std::vector<uint32_t> protein_ids(num_proteins);
        std::vector<uint32_t> gene_ids(num_proteins);
        std::vector<uint32_t> species_ids(num_proteins);
        std::vector<uint16_t> seq_lengths(num_proteins);
        std::vector<uint32_t> seq_offsets(num_proteins);
        
        DEBUG_PRINT("Loading protein metadata for %d proteins", num_proteins);
        
        // Try to load metadata from protein_details.json
        std::string metadata_path = db_path + "/protein_details.json";
        std::ifstream metadata_file(metadata_path);
        
        if (metadata_file.good()) {
            // Parse JSON metadata
            std::string json_content((std::istreambuf_iterator<char>(metadata_file)),
                                   std::istreambuf_iterator<char>());
            metadata_file.close();
            
            // Simple JSON parsing for the array of protein objects
            size_t pos = 0;
            size_t protein_idx = 0;
            size_t current_offset = 0;
            
            while ((pos = json_content.find("\"id\":", pos)) != std::string::npos && protein_idx < num_proteins) {
                // Parse protein ID
                pos += 5;
                size_t id_start = json_content.find_first_of("0123456789", pos);
                size_t id_end = json_content.find_first_not_of("0123456789", id_start);
                protein_ids[protein_idx] = std::stoi(json_content.substr(id_start, id_end - id_start));
                
                // Parse gene_id
                size_t gene_pos = json_content.find("\"gene_id\":", pos);
                if (gene_pos != std::string::npos && gene_pos < pos + 500) {
                    gene_pos += 10;
                    size_t gene_start = json_content.find_first_of("0123456789", gene_pos);
                    size_t gene_end = json_content.find_first_not_of("0123456789", gene_start);
                    gene_ids[protein_idx] = std::stoi(json_content.substr(gene_start, gene_end - gene_start));
                }
                
                // Parse species_id
                size_t species_pos = json_content.find("\"species_id\":", pos);
                if (species_pos != std::string::npos && species_pos < pos + 500) {
                    species_pos += 13;
                    size_t species_start = json_content.find_first_of("0123456789", species_pos);
                    size_t species_end = json_content.find_first_not_of("0123456789", species_start);
                    species_ids[protein_idx] = std::stoi(json_content.substr(species_start, species_end - species_start));
                }
                
                // Parse length
                size_t length_pos = json_content.find("\"length\":", pos);
                if (length_pos != std::string::npos && length_pos < pos + 500) {
                    length_pos += 9;
                    size_t length_start = json_content.find_first_of("0123456789", length_pos);
                    size_t length_end = json_content.find_first_not_of("0123456789", length_start);
                    seq_lengths[protein_idx] = std::stoi(json_content.substr(length_start, length_end - length_start));
                }
                
                // Set offset
                seq_offsets[protein_idx] = current_offset;
                current_offset += seq_lengths[protein_idx];
                
                protein_idx++;
                pos = id_end;
            }
            
            DEBUG_PRINT("Loaded metadata for %d proteins from JSON", protein_idx);
            
            // Print mappings for debugging
            for (uint32_t i = 0; i < std::min((uint32_t)5, num_proteins); i++) {
                printf("[GENE MAPPING] Protein %d: gene_id=%d, species_id=%d, length=%d, offset=%d\n",
                       i, gene_ids[i], species_ids[i], seq_lengths[i], seq_offsets[i]);
            }
        } else {
            // Fallback: estimate from sequence data
            printf("[WARNING] Could not load protein_details.json, using estimated mappings\n");
            size_t avg_protein_len = remaining_size / num_proteins;
            for (uint32_t i = 0; i < num_proteins; i++) {
                protein_ids[i] = i;
                gene_ids[i] = 1;  // Default to gyrA (gene_id 1)
                species_ids[i] = i % 6;  // Cycle through species
                seq_offsets[i] = i * avg_protein_len;
                seq_lengths[i] = (i == num_proteins - 1) ? 
                                (remaining_size - seq_offsets[i]) : avg_protein_len;
            }
        }
        
        // Debug: Print first few proteins
        if (DEBUG_TRANS) {
            for (int i = 0; i < 3 && i < num_proteins; i++) {
                const char* seq_start = all_sequences.data() + seq_offsets[i];
                DEBUG_PRINT("Protein %d: offset=%d, len=%d, seq=%.15s...", 
                           i, seq_offsets[i], seq_lengths[i], seq_start);
            }
        }
        
        // Use the all_sequences vector as our sequences
        std::vector<char> sequences = std::move(all_sequences);
        protein_file.close();
        
        DEBUG_PRINT("Loaded %d proteins, total sequence length: %zu", num_proteins, sequences.size());
        
        // Allocate and copy to GPU
        ProteinDatabase h_db;
        h_db.num_proteins = num_proteins;
        h_db.num_kmers = sorted_hashes.size();
        
        hipError_t err;
        
        // K-mer data
        err = hipMalloc(&h_db.sorted_kmer_hashes, sorted_hashes.size() * sizeof(uint64_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.kmer_start_indices, start_indices.size() * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.kmer_counts, kmer_counts.size() * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.position_data, position_data.size() * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        
        // Protein metadata
        err = hipMalloc(&h_db.protein_ids, num_proteins * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.gene_ids, num_proteins * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.species_ids, num_proteins * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.seq_lengths, num_proteins * sizeof(uint16_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.seq_offsets, num_proteins * sizeof(uint32_t));
        if (err != hipSuccess) return false;
        err = hipMalloc(&h_db.sequences, sequences.size() * sizeof(char));
        if (err != hipSuccess) return false;
        
        // Copy data
        hipMemcpy(h_db.sorted_kmer_hashes, sorted_hashes.data(), sorted_hashes.size() * sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.kmer_start_indices, start_indices.data(), start_indices.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.kmer_counts, kmer_counts.data(), kmer_counts.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.position_data, position_data.data(), position_data.size() * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.protein_ids, protein_ids.data(), num_proteins * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.gene_ids, gene_ids.data(), num_proteins * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.species_ids, species_ids.data(), num_proteins * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.seq_lengths, seq_lengths.data(), num_proteins * sizeof(uint16_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.seq_offsets, seq_offsets.data(), num_proteins * sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(h_db.sequences, sequences.data(), sequences.size() * sizeof(char), hipMemcpyHostToDevice);
        
        // Copy database structure
        err = hipMalloc(&d_protein_db, sizeof(ProteinDatabase));
        if (err != hipSuccess) return false;
        err = hipMemcpy(d_protein_db, &h_db, sizeof(ProteinDatabase), hipMemcpyHostToDevice);
        if (err != hipSuccess) return false;
        
        DEBUG_PRINT("Enhanced protein database loaded: %d proteins, %d unique 5-mers, SW=%s", 
                   num_proteins, (int)sorted_hashes.size(), smith_waterman_enabled ? "enabled" : "disabled");
        return true;
    }
    
    void setSmithWatermanEnabled(bool enabled) {
        smith_waterman_enabled = enabled;
        DEBUG_PRINT("Smith-Waterman alignment %s", enabled ? "ENABLED" : "DISABLED");
    }
    
    void searchTranslatedReads(
        const char* d_reads,
        const int* d_read_lengths,
        const int* d_read_offsets,
        const bool* d_reads_to_process,
        int num_reads,
        ProteinMatch* results,
        uint32_t* result_counts
    ) {
        if (num_reads > max_batch_size) {
            DEBUG_PRINT("ERROR: num_reads (%d) exceeds max_batch_size (%d)", num_reads, max_batch_size);
            return;
        }
        
        if (!d_protein_db) {
            DEBUG_PRINT("ERROR: Protein database not loaded");
            return;
        }
        
        hipError_t err;
        err = hipMemset(d_frame_counts, 0, num_reads * sizeof(uint32_t));
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Failed to reset frame counts: %s", hipGetErrorString(err));
            return;
        }
        err = hipMemset(d_match_counts, 0, num_reads * sizeof(uint32_t));
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Failed to reset match counts: %s", hipGetErrorString(err));
            return;
        }
        
        int block_size = 256;
        int grid_size = (num_reads + block_size - 1) / block_size;
        
        // Stage 1: 6-frame translation
        six_frame_translate_kernel<<<grid_size, block_size>>>(
            d_reads, d_read_lengths, d_read_offsets,
            num_reads, d_translated_frames, d_frame_counts
        );
        
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Translation kernel failed: %s", hipGetErrorString(err));
            return;
        }
        
        // Stage 2: Enhanced 5-mer protein matching with optional Smith-Waterman
        enhanced_protein_kmer_match_kernel<<<grid_size, block_size>>>(
            d_translated_frames, d_frame_counts,
            num_reads, d_protein_db,
            d_matches, d_match_counts, 32,
            smith_waterman_enabled
        );
        
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Enhanced protein matching kernel failed: %s", hipGetErrorString(err));
            return;
        }
        
        // Copy results
        err = hipMemcpy(results, d_matches, num_reads * 32 * sizeof(ProteinMatch), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Failed to copy protein matches: %s", hipGetErrorString(err));
            return;
        }
        err = hipMemcpy(result_counts, d_match_counts, num_reads * sizeof(uint32_t), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            DEBUG_PRINT("ERROR: Failed to copy match counts: %s", hipGetErrorString(err));
            return;
        }
    }
};

// C interface for integration
extern "C" {
    void* create_translated_search_engine(int batch_size) {
        TranslatedSearchEngine* engine = new TranslatedSearchEngine(batch_size, false); // SW disabled by default
        return engine;
    }
    
    void* create_translated_search_engine_with_sw(int batch_size, bool enable_sw) {
        TranslatedSearchEngine* engine = new TranslatedSearchEngine(batch_size, enable_sw);
        return engine;
    }
    
    void destroy_translated_search_engine(void* engine) {
        if (engine) {
            delete static_cast<TranslatedSearchEngine*>(engine);
        }
    }
    
    int load_protein_database(void* engine, const char* db_path) {
        if (!engine) return -1;
        TranslatedSearchEngine* te = static_cast<TranslatedSearchEngine*>(engine);
        return te->loadProteinDatabase(db_path) ? 0 : -1;
    }
    
    void set_smith_waterman_enabled(void* engine, bool enabled) {
        if (engine) {
            TranslatedSearchEngine* te = static_cast<TranslatedSearchEngine*>(engine);
            te->setSmithWatermanEnabled(enabled);
        }
    }
    
    int search_translated_reads(
        void* engine,
        const char* d_reads,
        const int* d_read_lengths,
        const int* d_read_offsets,
        const bool* d_reads_to_process,
        int num_reads,
        void* results,
        uint32_t* result_counts
    ) {
        if (!engine) return -1;
        TranslatedSearchEngine* te = static_cast<TranslatedSearchEngine*>(engine);
        te->searchTranslatedReads(
            d_reads, d_read_lengths, d_read_offsets,
            d_reads_to_process, num_reads,
            static_cast<ProteinMatch*>(results), result_counts
        );
        return 0;
    }
}

#endif // TRANSLATED_SEARCH_CU